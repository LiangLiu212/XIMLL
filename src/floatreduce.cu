#include "hip/hip_runtime.h"
#include "floatreduce.h"

#include <cmath>

#include "hip/hip_runtime.h"
#include ""

// Bandwidth: (((2^27) + 1) unsigned ints * 4 bytes/unsigned int)/(38.716 * 10^-3 s)
//  13.867 GB/s = 96.297% -> excellent memory bandwidth
// Reasonable point to stop working on this implementation's optimization
// Algorithm is not compute-intensive, so acheiving >75% of theoretical bandwidth is goal
// Main strategies used:
// - Process as much data as possible (in terms of algorithm correctness) in shared memory
// - Use sequential addressing to get rid of bank conflicts
__global__
void block_float_sum_reduce(double*  d_block_sums, 
	const double*  d_in,
	const unsigned int d_in_len)
{
	extern __shared__ double s_out[];

	unsigned int max_elems_per_block = blockDim.x;
	unsigned int glbl_tid = (blockDim.x) * blockIdx.x + threadIdx.x;
	unsigned int tid = threadIdx.x;
	
	// Zero out shared memory
	// Especially important when padding shmem for
	//  non-power of 2 sized input
//	s_out[threadIdx.x] = 0;
//	s_out[threadIdx.x + blockDim.x] = 0;

//	if (glbl_tid < d_in_len){
			s_out[tid] = d_in[glbl_tid];
//	}

	__syncthreads();

	// Actually do the reduction
	for (unsigned int s = 1; s < blockDim.x; s *= 2) {
			unsigned int index  = 2 *s *tid;
			if ( index < blockDim.x && index + s <  blockDim.x && index + s < d_in_len) {
					s_out[index] += s_out[index + s ];
			}
			__syncthreads();
	}

	// write result for this block to global mem
	if (tid == 0)
			d_block_sums[blockIdx.x] = s_out[0];
	__syncthreads();
}


double gpu_float_sum_reduce(double* d_in, unsigned int d_in_len)
{
		double total_sum = 0;

		// Set up number of threads and blocks
		// If input size is not power of two, the remainder will still need a whole block
		// Thus, number of blocks must be the least number of 2048-blocks greater than the input size
		unsigned int block_sz = MAX_BLOCK_SZ; // Halve the block size due to reduce3() and further 
		//  optimizations from there
		// our block_sum_reduce()
		unsigned int max_elems_per_block = block_sz ; // due to binary tree nature of algorithm
		// NVIDIA's reduceX()
		//unsigned int max_elems_per_block = block_sz;

		unsigned int grid_sz = 0;
		if (d_in_len <= max_elems_per_block)
		{
				grid_sz = (unsigned int)std::ceil(float(d_in_len) / float(max_elems_per_block));
		}
		else
		{
				grid_sz = (d_in_len ) / max_elems_per_block;
				if (d_in_len % max_elems_per_block != 0)
						grid_sz++;
		}

		// Allocate memory for array of total sums produced by each block
		// Array length must be the same as number of blocks / grid size
		double* d_block_sums;
		checkCudaErrors(hipMalloc(&d_block_sums, sizeof(double) * grid_sz));
		checkCudaErrors(hipMemset(d_block_sums, 0, sizeof(double) * grid_sz));

		// Sum data allocated for each block
		block_float_sum_reduce<<<grid_sz, block_sz, sizeof(double) * max_elems_per_block>>>(d_block_sums, d_in, d_in_len);

	//	std::cout << grid_sz << "	" << block_sz << "	" << max_elems_per_block << "  " << d_in_len << std::endl;
		//	reduce4<<<grid_sz, block_sz, sizeof(double) * block_sz>>>(d_block_sums, d_in, d_in_len);
		//print_d_array(d_block_sums, grid_sz);

		// Sum each block's total sums (to get global total sum)
		// Use basic implementation if number of total sums is <= 2048
		// Else, recurse on this same function
		if (grid_sz <= max_elems_per_block)
		{
				double* d_total_sum;
				checkCudaErrors(hipMalloc(&d_total_sum, sizeof(double)));
				checkCudaErrors(hipMemset(d_total_sum, 0, sizeof(double)));
				block_float_sum_reduce<<<1, block_sz, sizeof(double) * max_elems_per_block>>>(d_total_sum, d_block_sums, grid_sz);
		//		std::cout << 1 << "	" << block_sz << "	" << max_elems_per_block << "  " << grid_sz << std::endl;
				//	reduce4<<<1, block_sz, sizeof(double) * block_sz>>>(d_total_sum, d_block_sums, grid_sz);
				checkCudaErrors(hipMemcpy(&total_sum, d_total_sum, sizeof(double), hipMemcpyDeviceToHost));
				checkCudaErrors(hipFree(d_total_sum));
		}
		else
		{
				double* d_in_block_sums;
				checkCudaErrors(hipMalloc(&d_in_block_sums, sizeof(double) * grid_sz));
				checkCudaErrors(hipMemcpy(d_in_block_sums, d_block_sums, sizeof(double) * grid_sz, hipMemcpyDeviceToDevice));
				total_sum = gpu_float_sum_reduce(d_in_block_sums, grid_sz);
				checkCudaErrors(hipFree(d_in_block_sums));
		}

		checkCudaErrors(hipFree(d_block_sums));
		return total_sum;
}
