#include "hip/hip_runtime.h"
#include "TRandom.h"
#include "TH1F.h"
#include "TF1.h"
#include "TGraph.h"
#include "TCanvas.h"
#include "AngDisXiXi.hh"
#include "Amplitude.cuh"
#include "TFile.h"
#include "TTree.h"
#include "TStyle.h"
#include "TSystem.h"
#include "TVectorT.h"
#include "TStopwatch.h"
#include "TMath.h"
#include <TMinuit.h>
#include <vector>
#include <iostream>
#include <iomanip>
#include <sstream>
#include <time.h>
#include <fstream>
#include <string>
#include <cstdlib>
#include "readData.h"
#include "hip/hip_runtime.h"
#include ""
#include <cstdlib>
#include <getopt.h>
#include "rootfile.cuh"
#include <map>
#define _SLOOW
const Int_t NUM = 10000000;
#ifndef THREADS_PER_BLOCK
#define THREADS_PER_BLOCK 6
#define MATRIX_SIZE 80
#endif
int verbose_flag;
rootfile *rf;

void fcnMLLG(Int_t &npar, Double_t *gin, Double_t &f, Double_t *pp, Int_t iflag)
{
		double llf = rf->IOfcnmll(pp);
		if(llf == 0){ cout << "ERROR" << endl; return;}
		f =  llf;
}
//=====================================================================
// input [1] =  0; [2] =  type; [3] = step; [4] = output file
void XiXiMLL(int index, int MM){

		ofstream out;
		TString outfile_name = "out.txt";
		cout << outfile_name << endl;
		out.open(outfile_name, ios::out | ios::app);

		cout << "OK 11111111113" << endl;
		double Jpsi_alpha       = 0.586;  	  // alpha_J/Psi 
		double Jpsi_phi       =  1.121;		  //-TMath::Pi()/4.; // relative phase, Dphi_J/Psi
		double xi_alpha     = -0.3756;  		  // alpha (Sgm->p pi0)
		double xi_phi   = 0.012;   			  // alpha (Sgm->pbar pi0)
		double xib_alpha     = 0.3756;   
		double xib_phi   = -0.012;  
		double L1_alpha     = 0.692;   
		double L2_alpha     = -0.751;   
		double L3_alpha     = 0.751;   
		double L4_alpha     = -0.692;   
		cout << "OK" << endl;
		// cout << argv[1] << endl;
		// fit nr is used to tell which analysis cuts that are used
		TMinuit *minuit=new TMinuit(10);
		Int_t ierflag=0; 
		Double_t arglist[100];
		cout << "OK 11111111111" << endl;
		minuit->SetFCN(fcnMLLG);
		cout << "OK 11111111111" << endl;
		arglist[0]= 0;
		minuit->mnexcm("SET PRINT",arglist,1,ierflag);
		arglist[0]= 0.5;
		minuit->mnexcm("SET ERR",arglist,1,ierflag);
		minuit->mnparm(0, "alpha_jpsi" ,Jpsi_alpha, 0.001, -1., 1., ierflag);
		minuit->mnparm(1, "dphi_jpsi", Jpsi_phi, 0.001, -TMath::Pi(), TMath::Pi(), ierflag);
		cout << "OK 11111111111" << endl;
		minuit->mnparm(2, "xi_alpha" , xi_alpha, 0.001, -1., 0., ierflag);
		minuit->mnparm(3, "xi_phi" , xi_phi, 0.001,  -TMath::Pi(), TMath::Pi(), ierflag);
		minuit->mnparm(4, "xib_alpha" , xib_alpha, 0.001, 0, 1., ierflag);
		minuit->mnparm(5, "xib_phi" , xib_phi, 0.001,  -TMath::Pi(), TMath::Pi(), ierflag);
		minuit->mnparm(6, "L1_alpha" , L1_alpha, 0.001, 0., 1., ierflag);
		minuit->mnparm(7, "L2_alpha" , L2_alpha, 0.001, -1., 0., ierflag);
		minuit->mnparm(8, "L3_alpha" , L3_alpha, 0.001, 1., 0., ierflag);
		minuit->mnparm(9, "L4_alpha" , L4_alpha, 0.001, -1., 0., ierflag);
		cout << "OK 11111111113" << endl;
		//minuit->mnparm(3, "A_CP" , A_CP, 0.001, 0.,0., ierflag); 
		// 	 		minuit->FixParameter(7);
		//		 minuit->FixParameter(9);
		//		 minuit->FixParameter(12);
		//		 minuit->FixParameter(13);
		cout << "OK 11111111114" << endl;
		minuit->mnexcm("MINI",arglist,0,ierflag); //minimization using the migrag
		cout << "OK 11111111115" << endl;
		//limits both 0 implies no limit 
		minuit->mnexcm("MINOS",arglist,0,ierflag);
		cout << "OK 11111111115" << endl;
		minuit->mnmatu(1);
		cout << "OK 11111111115" << endl;
		Double_t fmin, fedm, errdef;
		Int_t   npari, nparx, istat; 
		minuit->mnstat(fmin, fedm, errdef, npari, nparx, istat);
		double res[10], err_res[10];
		for(int p = 0; p < 10; p++)
				minuit->GetParameter(p, res[p], err_res[p]);
		out << fmin << "," << istat << ","; 
		out << res[0]<< "," << err_res[0] << "," << res[1]<< "," << err_res[1]<< ","; 
		out << res[2]<< "," << err_res[2] << "," << res[3]<< "," << err_res[3]<< ",";
		out << res[4]<< "," << err_res[4] << "," << res[5]<< "," << err_res[5]<< ",";
		out << res[6]<< "," << err_res[6] << "," << res[7]<< "," << err_res[7]<< ",";
		out << res[8]<< "," << err_res[8] << "," << res[9]<< "," << err_res[9]<< endl;
		out.close();
		//	return 0;
}

int main(int argc, char **argv){

		int c;
		int m_command;
		vector<TString> m_year;
		vector<TString> m_namesample; 		// 
		TString m_version;


		while (1)
		{
				static struct option long_options[] =
				{
						/* These options set a flag. */
						{"data", no_argument,       0, 'd'},

						{"iocheck",   no_argument,       0, 0},

						{"inclusive",   no_argument,       0, 3},
						/* These options don’t set a flag.
						   We distinguish them by their indices. */
						{"bkg1",     no_argument,       0, 1},
						{"bkg2",  no_argument,       0, 2},

						{"version",  required_argument, 0, 'v'},

						{"mix",  no_argument, 0, 'm'},

						{"year",    required_argument, 0, 'y'},
						{"type",    required_argument, 0, 't'},
						{0, 0, 0, 0}
				};
				/* getopt_long stores the option index here. */
				int option_index = 0;

				c = getopt_long (argc, argv, "dv:my:t:",
								long_options, &option_index);


				/* Detect the end of the options. */
				if (c == -1)
						break;

				switch (c)
				{
						case 'd':
								m_command = 0;
								m_namesample.push_back("data");
								m_namesample.push_back("mdiy");
								m_namesample.push_back("phsp");
								break;
						case 0:
								m_command = 1;
								m_namesample.push_back("mdiy");
								m_namesample.push_back("phsp");
								puts ("option -a\n");
								break;

						case 3: 
								m_namesample.push_back("inclusive");
								break;
						case 1:
								m_namesample.push_back("bkg1");
								puts ("option -b\n");
								break;

						case 2:
								m_namesample.push_back("bkg2");
								break;

						case 'v':
								m_version = optarg;
								printf ("option -d with value `%s'\n", optarg);
								break;
						case 'y':
								m_year.push_back(optarg);

								printf ("option -i with value `%s'\n", optarg);
								while (optind < argc && argv[optind][0] != '-'){
										m_year.push_back(argv[optind]);
										optind++;
								}

								printf ("option -d with value `%s'\n", optarg);
								break;

						case 'm':
								break;

						case '?':

								/* getopt_long already printed an error message. */
								break;

						default:
								abort ();
				}
		}

		/* Instead of reporting ‘--verbose’
		   and ‘--brief’ as they are encountered,
		   we report the final status resulting from them. */
		if (verbose_flag)
				puts ("verbose flag is set");

		/* Print any remaining command line arguments (not options). */
		if (optind < argc)
		{
			
				while (optind < argc)
						printf ("%s ", argv[optind++]);
				putchar ('\n');
		}



		rf = new rootfile();
		rf->SetNyear(m_year.size());

		TString path = "/data/liul/workarea/XIXI/fit/boost";
		switch (m_command){
				case 0: {
								TString infile;
								for(int i = 0; i < m_year.size(); i++){
										for(int j = 0; j < m_namesample.size(); j++){
												if(!m_namesample[j].CompareTo("data")){
														infile = path + "/" + m_year[i] + "/" + "xixipm" + "/" + m_version + "/" + "data/data.root";
												}
												else if(!m_namesample[j].CompareTo("mdiy")){
														infile = path + "/" + m_year[i] + "/" + "xixipm" + "/" + m_version + "/" + "mdiy/mdiy30x.root";
												}
												else if(!m_namesample[j].CompareTo("phsp")){
														infile = path + "/" + m_year[i] + "/" + "xixipm" + "/" + m_version + "/" + "phsp/phsp30x.root";
												}
												else if(!m_namesample[j].CompareTo("bkg1")){
														infile = path + "/" + m_year[i] + "/" + "xixipm" + "/" + m_version + "/" + "bkg1/bkg30x.root";
												}
												else if(!m_namesample[j].CompareTo("bkg2")){
														infile = path + "/" + m_year[i] + "/" + "xixipm" + "/" + m_version + "/" + "bkg2/bkg30x.root";
												}
												else if(!m_namesample[j].CompareTo("inclusive")){
														infile = path + "/" + m_year[i] + "/" + "xixipm" + "/" + m_version + "/" + "inclusive/inclusive.root";
												}
												rf->Setfile(infile);
												rf->Setyear(m_year[i]);
												rf->Setsample(m_namesample[j]);
												rf->Settype(Form("xixipm"));
												rf->Setversion(m_version);
										}

										for(int j = 0; j < m_namesample.size(); j++){
												if(!m_namesample[j].CompareTo("data")){
														infile = path + "/" + m_year[i] + "/" + "xixipp" + "/" + m_version + "/" + "data/data.root";
												}
												else if(!m_namesample[j].CompareTo("mdiy")){
														infile = path + "/" + m_year[i] + "/" + "xixipp" + "/" + m_version + "/" + "mdiy/mdiy30x.root";
												}
												else if(!m_namesample[j].CompareTo("phsp")){
														infile = path + "/" + m_year[i] + "/" + "xixipp" + "/" + m_version + "/" + "phsp/phsp30x.root";
												}
												else if(!m_namesample[j].CompareTo("bkg1")){
														infile = path + "/" + m_year[i] + "/" + "xixipp" + "/" + m_version + "/" + "bkg1/bkg30x.root";
												}
												else if(!m_namesample[j].CompareTo("bkg2")){
														infile = path + "/" + m_year[i] + "/" + "xixipp" + "/" + m_version + "/" + "bkg2/bkg30x.root";
												}
												else if(!m_namesample[j].CompareTo("inclusive")){
														infile = path + "/" + m_year[i] + "/" + "xixipp" + "/" + m_version + "/" + "inclusive/inclusive.root";
												}
												rf->Setfile(infile);
												rf->Setyear(m_year[i]);
												rf->Setsample(m_namesample[j]);
												rf->Settype(Form("xixipp"));
												rf->Setversion(m_version);
										}
								}
								break;



						}
				case 1: {
								TString infile;
								for(int i = 0; i < m_year.size(); i++){
										for(int j = 0; j < m_namesample.size(); j++){
												if(!m_namesample[j].CompareTo("data")){
														infile = path + "/" + m_year[i] + "/" + "xixipm" + "/" + m_version + "/" + "data/data.root";
												}
												else if(!m_namesample[j].CompareTo("mdiy")){
														infile = path + "/" + m_year[i] + "/" + "xixipm" + "/" + m_version + "/" + "mdiy/mdiy30x.root";
												}
												else if(!m_namesample[j].CompareTo("phsp")){
														infile = path + "/" + m_year[i] + "/" + "xixipm" + "/" + m_version + "/" + "phsp/phsp30x.root";
												}
												else if(!m_namesample[j].CompareTo("bkg1")){
														infile = path + "/" + m_year[i] + "/" + "xixipm" + "/" + m_version + "/" + "bkg1/bkg30x.root";
												}
												else if(!m_namesample[j].CompareTo("bkg2")){
														infile = path + "/" + m_year[i] + "/" + "xixipm" + "/" + m_version + "/" + "bkg2/bkg30x.root";
												}
												else if(!m_namesample[j].CompareTo("inclusive")){
														infile = path + "/" + m_year[i] + "/" + "xixipm" + "/" + m_version + "/" + "inclusive/inclusive.root";
												}
												rf->Setfile(infile);
												rf->Setyear(m_year[i]);
												rf->Setsample(m_namesample[j]);
												rf->Settype(Form("xixipm"));
												rf->Setversion(m_version);
										}

										for(int j = 0; j < m_namesample.size(); j++){
												if(!m_namesample[j].CompareTo("data")){
														infile = path + "/" + m_year[i] + "/" + "xixipp" + "/" + m_version + "/" + "data/data.root";
												}
												else if(!m_namesample[j].CompareTo("mdiy")){
														infile = path + "/" + m_year[i] + "/" + "xixipp" + "/" + m_version + "/" + "mdiy/mdiy30x.root";
												}
												else if(!m_namesample[j].CompareTo("phsp")){
														infile = path + "/" + m_year[i] + "/" + "xixipp" + "/" + m_version + "/" + "phsp/phsp30x.root";
												}
												else if(!m_namesample[j].CompareTo("bkg1")){
														infile = path + "/" + m_year[i] + "/" + "xixipp" + "/" + m_version + "/" + "bkg1/bkg30x.root";
												}
												else if(!m_namesample[j].CompareTo("bkg2")){
														infile = path + "/" + m_year[i] + "/" + "xixipp" + "/" + m_version + "/" + "bkg2/bkg30x.root";
												}
												else if(!m_namesample[j].CompareTo("inclusive")){
														infile = path + "/" + m_year[i] + "/" + "xixipp" + "/" + m_version + "/" + "inclusive/inclusive.root";
												}
												rf->Setfile(infile);
												rf->Setyear(m_year[i]);
												rf->Setsample(m_namesample[j]);
												rf->Settype(Form("xixipp"));
												rf->Setversion(m_version);
										}
								}

								for(int i  = 0; i < rf->size(); i++){
										cout << rf->file(i) << " => " << rf->year(i) << '\n';
								}
								cout << endl;
								rf->InitialMemory();
								for(int i  = 0; i < 30; i++){
										rf->IOReadData(i, 30);
									//	rf->MassFit();
										XiXiMLL(i, 30);
										rf->FreeMemory();
								}
								break;
						}
		}
/*
		for(int i  = 0; i < rf->size(); i++){
				cout << rf->file(i) << " => " << rf->year(i) << '\n';
		}
		cout << endl;
		rf->InitialMemory();
		for(int i  = 0; i < 30; i++){
				rf->IOReadData(i, 30);
				rf->MassFit();
				//		XiXiMLL(i, 30);
				rf->FreeMemory();
		}
*/

		return 0;
}

