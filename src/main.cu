#include "hip/hip_runtime.h"
#include "TRandom.h"
#include "TH1F.h"
#include "TF1.h"
#include "TGraph.h"
#include "TCanvas.h"
#include "AngDisXiXi.hh"
#include "Amplitude.cuh"
#include "TFile.h"
#include "TTree.h"
#include "TStyle.h"
#include "TSystem.h"
#include "TVectorT.h"
#include "TStopwatch.h"
#include "TMath.h"
#include <TMinuit.h>
#include <vector>
#include <iostream>
#include <iomanip>
#include <sstream>
#include <time.h>
#include <fstream>
#include <string>
#include <cstdlib>
#include "readData.h"
#include "gpu_AngDisXiXi.hh"
#include "hip/hip_runtime.h"
#include ""
#include <cstdlib>
#include <getopt.h>
#include "rootfile.h"
#include <map>
#define _SLOOW
const Int_t NUM = 10000000;
#define THREADS_PER_BLOCK 6
#define MATRIX_SIZE 80
Int_t NN[4][12];
AngDisXiXi *angdis[4][2];
double **angdata[4][12];
double **gpu_angdata[4][12];
double *gpu_Matrix[4][12];
double *gpu_amp[4][12];
double *out_amp[4][12];
static int years;
static int fit_flag = 0;
static int fit_step = 0;
static int nsample = 0;
static int readdata_index = 0;

int verbose_flag;
std::vector<int> i_year;
std::vector<int> idx;


void InitialMemory(rootfile *rf){

		nsample = rf->size()/years;

		for(int i = 0; i < years; i++){
				for(int j = 0; j < nsample; j ++){
						angdata[i][j] = new double * [9];
						for(int l = 0; l < 9; l++){
								*(angdata[i][j] + l) =  new double [NUM];
						}
				}
		}

		for(int i = 0; i < years; i++)
				for(int j = 0; j < 2; j ++){
								if(angdis[i][j]){
						}
						else{
								angdis[i][j] = new AngDisXiXi();
						}
				}
}

//=====================================================================
void ReadData(rootfile *rf, const int index, const int MM)
{
		//	int years = 2;
		for(int i = 0; i < years; i++){ 		// read data
				for(int j = 0; j < nsample; j++){
						int l = -1;
						int n = i*nsample + j;
						if(!rf->type(n).CompareTo("xixipm")){
								l = 0;
						}
						else if(!rf->type(n).CompareTo("xixipp")){
								l = 1;
						}
						NN[i][j] =  readData(rf->file(n), angdis[i][l], angdata[i][j], rf->year(n), rf->sample(n), index, MM);
						if(rf->sample(n).CompareTo("phsp")){
								idx.push_back(n);
						}
				}
		}
		for(int i = 0; i < years; i++){
				for(int j = 0; j < nsample; j ++){
						cout << "N[" << i << "][" << j << "] : " << NN[i][j] << endl;
				}
		}

		double **temp_angdata[years][12]; // define a temporary array 
		for(int i = 0; i < years; i++){
				for(int j = 0; j < nsample; j ++){
						temp_angdata[i][j] = new double * [9];
						for(int l = 0; l < 9; l++){
								*(temp_angdata[i][j] + l) =  new double [NN[i][j] + THREADS_PER_BLOCK];
								for(int k = 0; k< NN[i][j]; k ++){
										*(*(temp_angdata[i][j] + l) + k) = *(*(angdata[i][j] + l) + k);
								}
						}
				}
		}
		if(readdata_index == 0){

				for(int i = 0; i < years; i++){    // copy data from cpu to gpu
						for(int j = 0; j < nsample; j ++){
								int size1 = (NN[i][j] + THREADS_PER_BLOCK) *sizeof(double);
								gpu_angdata[i][j] = new double * [9];
								for(int l = 0; l < 9; l++){
										hipMalloc( (void **) &(*(gpu_angdata[i][j] + l)), size1 );
										hipMemcpy( *(gpu_angdata[i][j] + l), *(temp_angdata[i][j] + l), size1, hipMemcpyHostToDevice );
										delete [] *(temp_angdata[i][j] + l);
								}
								hipMalloc( (void **) &gpu_Matrix[i][j], size1 * MATRIX_SIZE  );
								hipMalloc( (void **) &gpu_amp[i][j], size1 * MATRIX_SIZE  );
								out_amp[i][j] = new double [NN[i][j] + THREADS_PER_BLOCK];
						}
				}
		}
		else{
				for(int i = 0; i < years; i++){    // copy data from cpu to gpu
						for(int j = 0; j < nsample; j ++){
								int size1 = (NN[i][j] + THREADS_PER_BLOCK) *sizeof(double);
								for(int l = 0; l < 9; l++){
										hipMemcpy( *(gpu_angdata[i][j] + l), *(temp_angdata[i][j] + l), size1, hipMemcpyHostToDevice );
										delete [] *(temp_angdata[i][j] + l);
								}
						}
				}
		}
		readdata_index++;
}


void fcnMLLG(Int_t &npar, Double_t *gin, Double_t &f, Double_t *pp, Int_t iflag)
{ 
		double pp1[8], pp2[8];
		for(int i = 0; i < 6; i++){
				pp1[i] = pp[i];
				pp2[i] = pp[i];
		}
		pp1[6] = pp[6]; pp1[7] = pp[7];
		pp2[6] = pp[8]; pp2[7] = pp[9];

		AA_parameter aa_para;
		aa_para.alpha_jpsi = pp[0];
		aa_para.phi_jpsi = pp[1];
		aa_para.alpha_xi = pp[2];
		aa_para.phi_xi = pp[3];
		aa_para.alpha_xibar = pp[4];
		aa_para.phi_xibar = pp[5];
		aa_para.alpha1_lambda = pp[6];
		aa_para.alpha1_lambdabar = pp[7];
		aa_para.alpha2_lambda = pp[8];
		aa_para.alpha2_lambdabar = pp[9];


		hipError_t cudaStatus;
		clock_t start,end;
		double loglike[4][12];
		for(int i = 0; i < years; i ++){
				angdis[i][0]->SetParameter(pp1);
				angdis[i][1]->SetParameter(pp2);
				for (int j = 0; j < (nsample - 2); j++){
						int flag = j / ((nsample - 2) / 2);
						start = clock();
						gpu_Amp <<< (NN[i][idx[j]] * MATRIX_SIZE + MATRIX_SIZE * THREADS_PER_BLOCK ) / (MATRIX_SIZE * THREADS_PER_BLOCK), MATRIX_SIZE * THREADS_PER_BLOCK >>> ( 
										*(gpu_angdata[i][idx[j]] + 0), 
										*(gpu_angdata[i][idx[j]] + 1), 
										*(gpu_angdata[i][idx[j]] + 2), 
										*(gpu_angdata[i][idx[j]] + 3), 
										*(gpu_angdata[i][idx[j]] + 4), 
										*(gpu_angdata[i][idx[j]] + 5), 
										*(gpu_angdata[i][idx[j]] + 6), 
										*(gpu_angdata[i][idx[j]] + 7), 
										*(gpu_angdata[i][idx[j]] + 8),
										gpu_amp[i][idx[j]],
										(NN[i][idx[j]] + THREADS_PER_BLOCK)*80, 
										aa_para, flag, gpu_Matrix[i][idx[j]]);
						hipDeviceSynchronize(); // wait until prior kernel is finished
						end = clock();
					//	double time3 = ((double)(end-start))/CLOCKS_PER_SEC;
					//	cout << "GPU 3: running kernel " << time3 << " seconds" << endl;
						cudaStatus = hipGetLastError();
						if (cudaStatus != hipSuccess){
								cerr << "failure to call cuda kernel 004!" << endl;
								exit(1);
						}

						int mat_size = (NN[i][idx[j]] + THREADS_PER_BLOCK) *sizeof(double);
						hipMemcpy( out_amp[i][idx[j]], gpu_amp[i][idx[j]], mat_size, hipMemcpyDeviceToHost );
						cudaStatus = hipGetLastError();
						if (cudaStatus != hipSuccess){
								cerr << "failure to call cuda kernel 002!" << endl;
								exit(1);
						}
						loglike[i][j] = 0;
						for(int evt = 0; evt < NN[i][idx[j]]; evt++){
								//	cout << "host C munu 0: " << 	*(host_eval + evt) << endl;
								if(*(out_amp[i][idx[j]] + evt) <= 0){ f=0; cout << "data : " << *(out_amp[i][idx[j]] + evt) << endl;  return; }
								loglike[i][j] += TMath::Log(*(out_amp[i][idx[j]] + evt));
						}
				}
		}
		//	exit(1);
		double norm[4][2];
		for (int i = 0; i < years; i++){
				for (int j = 0; j < 2; j++){
						norm[i][j] = 0;
						norm[i][j] = angdis[i][j]->CalcToIntegral();
				}
		}

		double N_BKG[4][2] = {{827.43333, 653.96667}, {4443, 3498}, {8827.38, 9803.34}, {8234.57, 9115.42}};
		int idx_year = 0;

		double llk = 0;
		double l1 = 0;
		double l2 = 0;

		for (int i = 0; i < years; i++){
				if(i_year[i] == 2009) idx_year = 0;
				if(i_year[i] == 2012) idx_year = 1;
				if(i_year[i] == 2018) idx_year = 2;
				if(i_year[i] == 2019) idx_year = 3;
				//	cout <<  i_year[i] <<  "  Background  : "  << N_BKG[idx_year][0] << " " << N_BKG[idx_year][1] << endl;
				if(nsample == 6){
				//		cout << loglike[i][0] << "  " << loglike[i][1] << "  " << loglike[i][2] << "  " << loglike[i][3] << endl;
						l1 = - loglike[i][0] + N_BKG[idx_year][0]*loglike[i][1]/Double_t(NN[i][1]) + (Double_t(NN[i][0]) - N_BKG[idx_year][0])*TMath::Log(norm[i][0]/Double_t(NN[i][2]));
						l2 = - loglike[i][2] + N_BKG[idx_year][1]*loglike[i][3]/Double_t(NN[i][4]) + (Double_t(NN[i][3]) - N_BKG[idx_year][1])*TMath::Log(norm[i][1]/Double_t(NN[i][5]));
				}
				else if(nsample == 4){
						l1 = - loglike[i][0] + (Double_t(NN[i][0]))*TMath::Log(norm[i][0]/NN[i][1]);
						l2 = - loglike[i][2] + (Double_t(NN[i][3]))*TMath::Log(norm[i][1]/NN[i][3]);
				}
				else if(nsample == 8){
						l1 = - loglike[i][0] - loglike[i][1] + N_BKG[idx_year][0]*loglike[i][2]/Double_t(NN[i][2]) + (Double_t(NN[i][0]) + NN[i][1] - N_BKG[idx_year][0])*TMath::Log(norm[i][0]/Double_t(NN[i][3]));
						l2 = - loglike[i][3] - loglike[i][4] + N_BKG[idx_year][1]*loglike[i][5]/Double_t(NN[i][6]) + (Double_t(NN[i][4]) + NN[i][5] - N_BKG[idx_year][1])*TMath::Log(norm[i][1]/Double_t(NN[i][7]));
				}
				else{
						cerr << "error fit_flag!" << endl;
				}
				llk += (l1 + l2);
		}

		f = llk;
		if(fit_step%100 == 0){
				std::cout << "Loglike: " << f << std::endl; 
				for( int i = 0; i<10 ; i++ ) cout<<pp[i]<<" ";
				cout << endl;
		}
		fit_step++;
}
//=====================================================================
// input [1] =  0; [2] =  type; [3] = step; [4] = output file
void XiXiMLL(rootfile *rf, int index, int MM){


		ofstream out;
		TString outfile_name = "out.txt";
		cout << outfile_name << endl;
		out.open(outfile_name, ios::out | ios::app);

		ReadData(rf, index, MM);



		cout << "OK 11111111113" << endl;
		double Jpsi_alpha       = 0.586;  	  // alpha_J/Psi 
		double Jpsi_phi       =  1.121;		  //-TMath::Pi()/4.; // relative phase, Dphi_J/Psi
		double xi_alpha     = -0.3756;  		  // alpha (Sgm->p pi0)
		double xi_phi   = 0.012;   			  // alpha (Sgm->pbar pi0)
		double xib_alpha     = 0.3756;   
		double xib_phi   = -0.012;  
		double L1_alpha     = 0.692;   
		double L2_alpha     = -0.751;   
		double L3_alpha     = 0.751;   
		double L4_alpha     = -0.692;   
		cout << "OK" << endl;
		// cout << argv[1] << endl;
		// fit nr is used to tell which analysis cuts that are used
		TMinuit *minuit=new TMinuit(10);
		Int_t ierflag=0; 
		Double_t arglist[100];
		cout << "OK 11111111111" << endl;
		minuit->SetFCN(fcnMLLG);
		cout << "OK 11111111111" << endl;
		arglist[0]= 0;
		minuit->mnexcm("SET PRINT",arglist,1,ierflag);
		arglist[0]= 0.5;
		minuit->mnexcm("SET ERR",arglist,1,ierflag);
		minuit->mnparm(0, "alpha_jpsi" ,Jpsi_alpha, 0.001, -1., 1., ierflag);
		minuit->mnparm(1, "dphi_jpsi", Jpsi_phi, 0.001, -TMath::Pi(), TMath::Pi(), ierflag);
		cout << "OK 11111111111" << endl;
		minuit->mnparm(2, "xi_alpha" , xi_alpha, 0.001, -1., 0., ierflag);
		minuit->mnparm(3, "xi_phi" , xi_phi, 0.001,  -TMath::Pi(), TMath::Pi(), ierflag);
		minuit->mnparm(4, "xib_alpha" , xib_alpha, 0.001, 0, 1., ierflag);
		minuit->mnparm(5, "xib_phi" , xib_phi, 0.001,  -TMath::Pi(), TMath::Pi(), ierflag);
		minuit->mnparm(6, "L1_alpha" , L1_alpha, 0.001, 0., 1., ierflag);
		minuit->mnparm(7, "L2_alpha" , L2_alpha, 0.001, -1., 0., ierflag);
		minuit->mnparm(8, "L3_alpha" , L3_alpha, 0.001, 1., 0., ierflag);
		minuit->mnparm(9, "L4_alpha" , L4_alpha, 0.001, -1., 0., ierflag);
		cout << "OK 11111111113" << endl;
		//minuit->mnparm(3, "A_CP" , A_CP, 0.001, 0.,0., ierflag); 
		// 	 		minuit->FixParameter(7);
		//		 minuit->FixParameter(9);
		//		 minuit->FixParameter(12);
		//		 minuit->FixParameter(13);
		cout << "OK 11111111114" << endl;
		minuit->mnexcm("MINI",arglist,0,ierflag); //minimization using the migrag
		cout << "OK 11111111115" << endl;
		//limits both 0 implies no limit 
		minuit->mnexcm("MINOS",arglist,0,ierflag);
		cout << "OK 11111111115" << endl;
		minuit->mnmatu(1);
		cout << "OK 11111111115" << endl;
		Double_t fmin, fedm, errdef;
		Int_t   npari, nparx, istat; 
		minuit->mnstat(fmin, fedm, errdef, npari, nparx, istat);
		double res[10], err_res[10];
		for(int p = 0; p < 10; p++)
				minuit->GetParameter(p, res[p], err_res[p]);
		out << fmin << "," << istat << "," << NN[0][0] << ","<< NN[0][1] << ","; 
		out << res[0]<< "," << err_res[0] << "," << res[1]<< "," << err_res[1]<< ","; 
		out << res[2]<< "," << err_res[2] << "," << res[3]<< "," << err_res[3]<< ",";
		out << res[4]<< "," << err_res[4] << "," << res[5]<< "," << err_res[5]<< ",";
		out << res[6]<< "," << err_res[6] << "," << res[7]<< "," << err_res[7]<< ",";
		out << res[8]<< "," << err_res[8] << "," << res[9]<< "," << err_res[9]<< endl;
		out.close();
		//	return 0;
}

int main(int argc, char **argv){

		int c;
		int m_command;
		vector<TString> m_year;
		TString m_bkg;
		TString m_inclusive;
		TString m_version;

		fit_flag = 2;

		while (1)
		{
				static struct option long_options[] =
				{
						/* These options set a flag. */
						{"data", no_argument,       0, 'd'},
						{"iocheck",   no_argument,       0, 0},
						{"inclusive",   no_argument,       0, 3},
						/* These options don’t set a flag.
						   We distinguish them by their indices. */
						{"bkg1",     no_argument,       0, 1},
						{"bkg2",  no_argument,       0, 2},
						{"version",  required_argument, 0, 'v'},
						{"mix",  no_argument, 0, 'm'},
						{"year",    required_argument, 0, 'y'},
						{0, 0, 0, 0}
				};
				/* getopt_long stores the option index here. */
				int option_index = 0;

				c = getopt_long (argc, argv, "dv:my:",
								long_options, &option_index);

				/* Detect the end of the options. */
				if (c == -1)
						break;

				switch (c)
				{
						case 'd':
								m_command = 0;
								break;
						case 0:
								m_command = 1;
								puts ("option -a\n");
								break;

						case 3: 
								m_inclusive = "inclusive";
								break;
						case 1:
								m_bkg = "bkg1";
								fit_flag = 1;
								puts ("option -b\n");
								break;

						case 2:
								m_bkg = "bkg2";
								fit_flag = 1;
								break;

						case 'v':
								m_version = optarg;
								printf ("option -d with value `%s'\n", optarg);
								break;
						case 'y':
								m_year.push_back(optarg);
								i_year.push_back(atoi(optarg));

								printf ("option -i with value `%s'\n", optarg);
								while (optind < argc && argv[optind][0] != '-'){
										m_year.push_back(argv[optind]);
										i_year.push_back(atoi(argv[optind]));
										optind++;
								}

								printf ("option -d with value `%s'\n", optarg);
								break;

						case 'm':
								break;

						case '?':

								/* getopt_long already printed an error message. */
								break;

						default:
								abort ();
				}
		}

		/* Instead of reporting ‘--verbose’
		   and ‘--brief’ as they are encountered,
		   we report the final status resulting from them. */
		if (verbose_flag)
				puts ("verbose flag is set");

		/* Print any remaining command line arguments (not options). */
		if (optind < argc)
		{
				printf ("non-option ARGV-elements: ");
				while (optind < argc)
						printf ("%s ", argv[optind++]);
				putchar ('\n');
		}



		map<TString, TString> files;
		rootfile *rf = new rootfile();
		switch (m_command){
				case 0: {
						}
				case 1: {
								TString path = "/data/liul/workarea/XIXI/fit/boost";
								TString infile;
								TString type;
								years = m_year.size();
								for(int i = 0; i < m_year.size(); i++){
										infile = path + "/" + m_year[i] + "/" + "xixipm" + "/" + m_version + "/" + "mdiy/mdiy30x.root";
										type = "mdiy";
										rf->Setfile(infile);
										rf->Setyear(m_year[i]);
										rf->Setsample(type);
										rf->Settype(Form("xixipm"));
										files.insert(make_pair(infile, type));
										if(!m_inclusive.CompareTo("inclusive")){
												infile = path + "/" + m_year[i] + "/" + "xixipm" + "/" + m_version + "/" + "inclusive/inclusive.root";
												type = "inclusive";
												files.insert(make_pair(infile, type));
												rf->Setfile(infile);
												rf->Setyear(m_year[i]);
												rf->Setsample(type);
												rf->Settype(Form("xixipm"));
										}
										if(!m_bkg.CompareTo("bkg1")){
												infile = path + "/" + m_year[i] + "/" + "xixipm" + "/" + m_version + "/" + "bkg1/bkg30x.root";
												type = "bkg1";
												files.insert(make_pair(infile, type));
												rf->Setfile(infile);
												rf->Setyear(m_year[i]);
												rf->Setsample(type);
												rf->Settype(Form("xixipm"));
										}
										else if(!m_bkg.CompareTo("bkg2")){
												infile = path + "/" + m_year[i] + "/" + "xixipm" + "/" + m_version + "/" + "bkg2/bkg30x.root";
												type = "bkg2";
												files.insert(make_pair(infile, type));
												rf->Setfile(infile);
												rf->Setyear(m_year[i]);
												rf->Setsample(type);
												rf->Settype(Form("xixipm"));
										}
										infile = path + "/" + m_year[i] + "/" + "xixipm" + "/" + m_version + "/" + "phsp/phsp30x.root";
										type = "phsp";
										files.insert(make_pair(infile, type));
										rf->Setfile(infile);
										rf->Setyear(m_year[i]);
										rf->Setsample(type);
										rf->Settype(Form("xixipm"));

										infile = path + "/" + m_year[i] + "/" + "xixipp" + "/" + m_version + "/" + "mdiy/mdiy30x.root";
										type = "mdiy";
										files.insert(make_pair(infile, type));
										rf->Setfile(infile);
										rf->Setyear(m_year[i]);
										rf->Setsample(type);
										rf->Settype(Form("xixipp"));
										if(!m_inclusive.CompareTo("inclusive")){
												infile = path + "/" + m_year[i] + "/" + "xixipp" + "/" + m_version + "/" + "inclusive/inclusive.root";
												type = "inclusive";
												files.insert(make_pair(infile, type));
												rf->Setfile(infile);
												rf->Setyear(m_year[i]);
												rf->Setsample(type);
												rf->Settype(Form("xixipm"));
										}
										if(!m_bkg.CompareTo("bkg1")){
												infile = path + "/" + m_year[i] + "/" + "xixipp" + "/" + m_version + "/" + m_bkg +  "/bkg30x.root";
												type = "bkg1";
												files.insert(make_pair(infile, type));
												rf->Setfile(infile);
												rf->Setyear(m_year[i]);
												rf->Setsample(type);
												rf->Settype(Form("xixipp"));
										}
										else if(!m_bkg.CompareTo("bkg2")){
												infile = path + "/" + m_year[i] + "/" + "xixipp" + "/" + m_version + "/" + m_bkg+ "/bkg30x.root";
												type = "bkg2";
												files.insert(make_pair(infile, type));
												rf->Setfile(infile);
												rf->Setyear(m_year[i]);
												rf->Setsample(type);
												rf->Settype(Form("xixipp"));
										}
										infile = path + "/" + m_year[i] + "/" + "xixipp" + "/" + m_version + "/" + "phsp/phsp30x.root";
										type = "phsp";
										files.insert(make_pair(infile, type));
										rf->Setfile(infile);
										rf->Setyear(m_year[i]);
										rf->Setsample(type);
										rf->Settype(Form("xixipp"));
								}

						}
		}
		for(int i  = 0; i < rf->size(); i++){
				cout << rf->file(i) << " => " << rf->year(i) << '\n';
		}
		cout << endl;
		InitialMemory(rf);

		for(int i  = 0; i < 30; i++){
				XiXiMLL(rf, i, 30);
		}
		return 0;
}

