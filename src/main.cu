#include "hip/hip_runtime.h"
#include "TRandom.h"
#include "TH1F.h"
#include "TF1.h"
#include "TGraph.h"
#include "TCanvas.h"
#include "AngDisXiXi.hh"
#include "TFile.h"
#include "TTree.h"
#include "TStyle.h"
#include "TSystem.h"
#include "TVectorT.h"
#include "TStopwatch.h"
#include "TMath.h"
#include <TMinuit.h>
#include <vector>
#include <iostream>
#include <iomanip>
#include <sstream>
#include <fstream>
#include <string>
#include <cstdlib>
#include "readData.h"
#include "gpu_AngDisXiXi.hh"
#include "hip/hip_runtime.h"
#include ""
#define _SLOOW
const Int_t NUM = 10000000;
#define THREADS_PER_BLOCK 1024
Int_t NN[4][6];
AngDisXiXi *angdis[4][2];
double **angdata[4][6];
double **gpu_angdata[4][6];
static int years;
int fit_flag = 0;
int fit_step = 0;

std::vector<int> i_year;


struct AA_parameter{
	double alpha_jpsi;
	double phi_jpsi;
	double alpha_xi;
	double phi_xi;
	double alpha_xibar;
	double phi_xibar;
	double alpha1_lambda;
	double alpha1_lambdabar;
	double alpha2_lambda;
	double alpha2_lambdabar;
};



__global__ void gpu_aa(double *g_xithe,
					   double *g_lthe,
					   double *g_lphi,
					   double *g_lbthe,
					   double *g_lbphi,
					   double *g_pthe,
					   double *g_pphi,
					   double *g_apthe,
					   double *g_apphi,
					   const int g_NN,
					   const AA_parameter g_para,
					   AA_Matrix *g_munu,
					   const int g_flag, double *g_eval){
		int index = blockIdx.x * blockDim.x + threadIdx.x;
		if(index < g_NN){
				__shared__ double pp1[8], pp2[8];
				pp1[0] = g_para.alpha_jpsi;
				pp1[1] = g_para.phi_jpsi;
				pp1[2] = g_para.alpha_xi;
				pp1[3] = g_para.phi_xi;
				pp1[4] = g_para.alpha_xibar;
				pp1[5] = g_para.phi_xibar;
				pp1[6] = g_para.alpha1_lambda;
				pp1[7] = g_para.alpha1_lambdabar;
				pp2[0] = g_para.alpha_jpsi;
				pp2[1] = g_para.phi_jpsi;
				pp2[2] = g_para.alpha_xi;
				pp2[3] = g_para.phi_xi;
				pp2[4] = g_para.alpha_xibar;
				pp2[5] = g_para.phi_xibar;
				pp2[6] = g_para.alpha2_lambda;
				pp2[7] = g_para.alpha2_lambdabar;
				__syncthreads();
				if(g_flag < 2){
						Amp(g_xithe[index], g_lthe[index], g_lphi[index], g_lbthe[index], 
										g_lbphi[index], g_pthe[index], g_pphi[index], g_apthe[index], g_apphi[index], 
										pp1, &g_munu[index]);
				}
				else {
						Amp(g_xithe[index], g_lthe[index], g_lphi[index], g_lbthe[index], 
										g_lbphi[index], g_pthe[index], g_pphi[index], g_apthe[index], g_apphi[index], 
										pp2, &g_munu[index]);

				}
		//		double tep = 0;
				g_eval[index] = 0;
				for(int mu=0; mu<4;mu++){// Xi loop
						for(int nu=0;nu<4;nu++){// Xibar loop
								for(int k=0;k<4;k++){
										for(int j=0;j<4;j++){
												g_eval[index] +=  g_munu[index].thC[mu][nu]*
														g_munu[index].tHa[mu][k]*g_munu[index].tHb[nu][j]*
														g_munu[index].tHc[k][0]*g_munu[index].tHd[j][0];
										}
								}
						}
				}
				
	//			g_eval[index] = 1.0;


				//	printf("C numu  0 0 : %f\n", g_munu[index].thC[0][0]);
				//	printf("C numu  0 1 : %f\n", g_munu[index].thC[0][1]);
				//	printf("C numu  0 2 : %f\n", g_munu[index].thC[0][2]);
				//	printf("C numu  0 3 : %f\n", g_munu[index].thC[0][3]);


				//		#include "amplitude.cxx"

				//	printf("from gpu :  %f \n", g_eval[index]);
		}
}

//=====================================================================
void ReadData(int flag[4], const int index, const int MM)
{
		//	int years = 2;
		const char *type[6] = {"DATA", "BKG", "PHSP", "DATA", "BKG", "PHSP"};
		//	int flag[4] = {2009, 2012, 2018, 2019};
		const char *file[6] = {
				"/data/liul/workarea/XIXI/Rec3/mdiyRecpm2012.root", 
				"/data/liul/workarea/XIXI/Rec3/mdiyRecpm2012.root",
				"/data/liul/workarea/XIXI/Rec3/phspRecpm2012.root",
				"/data/liul/workarea/XIXI/Rec3/mdiyRecpp2012.root",
				"/data/liul/workarea/XIXI/Rec3/mdiyRecpp2012.root", 
				"/data/liul/workarea/XIXI/Rec3/phspRecpp2012.root" };

		for(int i = 0; i < years; i++){
				for(int j = 0; j < 6; j ++){
						angdata[i][j] = new double * [9];
						for(int l = 0; l < 9; l++){
								*(angdata[i][j] + l) =  new double [NUM];
						}
				}
		}

		for(int i = 0; i < years; i++)
				for(int j = 0; j < 2; j ++){
								if(angdis[i][j]){
						}
						else{
								angdis[i][j] = new AngDisXiXi();
						}
				}

		for(int i = 0; i < years; i++){ 		// read data
				for(int j = 0; j < 6; j ++){
						int l = j / 3;
						NN[i][j] =  readData(file[j], angdis[i][l], angdata[i][j], flag[i], type[j], index, MM);
				}
		}
		for(int i = 0; i < years; i++){
				for(int j = 0; j < 6; j ++){
						cout << "N[" << i << "][" << j << "] : " << NN[i][j] << endl;
				}
		}
		double **temp_angdata[years][6]; // define a temporary array 
		for(int i = 0; i < years; i++){
				for(int j = 0; j < 6; j ++){
						temp_angdata[i][j] = new double * [9];
						for(int l = 0; l < 9; l++){
								*(temp_angdata[i][j] + l) =  new double [NN[i][j]];
								for(int k = 0; k< NN[i][j]; k ++){
										*(*(temp_angdata[i][j] + l) + k) = *(*(angdata[i][j] + l) + k);
								}
						}
				}
		}



		for(int i = 0; i < years; i++){    // copy data from cpu to gpu
				for(int j = 0; j < 6; j ++){
						int size1 = NN[i][j] *sizeof(double);
						gpu_angdata[i][j] = new double * [9];
						for(int l = 0; l < 9; l++){
								hipMalloc( (void **) &(*(gpu_angdata[i][j] + l)), size1 );
								hipMemcpy( *(gpu_angdata[i][j] + l), *(temp_angdata[i][j] + l), size1, hipMemcpyHostToDevice );
								//	for ( int k = 0; k < NN[i][j]; k++ ){
								//			cout << "read file : " << *(*((temp_angdata[i][j]) + 0) + k) << endl;
								//	}
						}
				}
		}
}


void fcnMLLG(Int_t &npar, Double_t *gin, Double_t &f, Double_t *pp, Int_t iflag)
{ 
		double pp1[8], pp2[8];
		for(int i = 0; i < 6; i++){
				pp1[i] = pp[i];
				pp2[i] = pp[i];
		}
		pp1[6] = pp[6]; pp1[7] = pp[7];
		pp2[6] = pp[8]; pp2[7] = pp[9];

		AA_parameter aa_para;
		aa_para.alpha_jpsi = pp[0];
		aa_para.phi_jpsi = pp[1];
		aa_para.alpha_xi = pp[2];
		aa_para.phi_xi = pp[3];
		aa_para.alpha_xibar = pp[4];
		aa_para.phi_xibar = pp[5];
		aa_para.alpha1_lambda = pp[6];
		aa_para.alpha1_lambdabar = pp[7];
		aa_para.alpha2_lambda = pp[8];
		aa_para.alpha2_lambdabar = pp[9];

		double *host_eval;
		double *gpu_eval;
		AA_Matrix *gpu_munu;
		AA_Matrix *host_munu;

		hipError_t cudaStatus;
		double loglike[4][4];
		int idx[4] = {0, 1, 3, 4};  // data1 bgk1 data2 bkg2
		for(int i = 0; i < years; i ++){
				angdis[i][0]->SetParameter(pp1);
				angdis[i][1]->SetParameter(pp2);
				for (int j = 0; j < 4; j++){

						host_munu = new AA_Matrix [NN[i][idx[j]]];
						host_eval = new double [NN[i][idx[j]]];
						int mat_size = NN[i][idx[j]]*sizeof(*gpu_munu);
						hipMalloc( (void **) &gpu_munu,  mat_size);
						int size = NN[i][idx[j]] * sizeof(*gpu_eval);
						hipMalloc( (void **) &gpu_eval, size);
						cudaStatus = hipGetLastError();
						if (cudaStatus != hipSuccess){
								cerr << "failure to call cuda kernel 003!" << endl;
								exit(1);
						}


						gpu_aa <<< (NN[i][idx[j]] + (THREADS_PER_BLOCK-1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>> ( 
										*(gpu_angdata[i][idx[j]] + 0), 
										*(gpu_angdata[i][idx[j]] + 1), 
										*(gpu_angdata[i][idx[j]] + 2), 
										*(gpu_angdata[i][idx[j]] + 3), 
										*(gpu_angdata[i][idx[j]] + 4), 
										*(gpu_angdata[i][idx[j]] + 5), 
										*(gpu_angdata[i][idx[j]] + 6), 
										*(gpu_angdata[i][idx[j]] + 7), 
										*(gpu_angdata[i][idx[j]] + 8), 
										NN[i][idx[j]], 
										aa_para, gpu_munu, j, gpu_eval);
						cudaStatus = hipGetLastError();
						if (cudaStatus != hipSuccess){
								cerr << "failure to call cuda kernel 004!" << endl;
								exit(1);
						}

						hipDeviceSynchronize();
						cudaStatus = hipGetLastError();
						if (cudaStatus != hipSuccess){
								cerr << "failure to call cuda kernel 001!" << endl;
								exit(1);
						}
						hipMemcpy( host_munu, gpu_munu, mat_size, hipMemcpyDeviceToHost );
						hipMemcpy( host_eval, gpu_eval, size, hipMemcpyDeviceToHost );
						hipFree( gpu_munu );
						hipFree( gpu_eval );
						cudaStatus = hipGetLastError();
						if (cudaStatus != hipSuccess){
								cerr << "failure to call cuda kernel 002!" << endl;
								exit(1);
						}
						loglike[i][j] = 0;
						for(int evt = 0; evt < NN[i][idx[j]]; evt++){
								//	cout << "host C munu 0: " << 	*(host_eval + evt) << endl;
								if(*(host_eval + evt) <= 0){ f=0; cout << "data : " << *(host_eval + evt) << endl;  return; }
								loglike[i][j] += TMath::Log(*(host_eval + evt));
								//	cout << "host C munu 1: " << 	(host_munu + evt)->tHa[0][1] << endl;
								//	cout << "host C munu 2: " << 	(host_munu + evt)->tHa[0][2] << endl;
								//	cout << "host C munu 3: " << 	(host_munu + evt)->tHa[0][3] << endl;
						}
						delete [] host_munu;
						host_munu = NULL;
						delete [] host_eval;
						host_eval = NULL;
				}
		}
	//	exit(1);

		double norm[4][2];
		for (int i = 0; i < years; i++){
				for (int j = 0; j < 2; j++){
						norm[i][j] = 0;
						norm[i][j] = angdis[i][j]->CalcToIntegral();
						norm[i][j]/=Double_t(NN[i][3*j + 2]);
				}
		}

		double N_BKG[4][2] = {{477.15, 448.882}, {2293.86, 2427.39}, {8827.38, 9803.34}, {8234.57, 9115.42}};
		int idx_year = 0;

		double llk = 0;
		double l1 = 0;
		double l2 = 0;

		for (int i = 0; i < years; i++){
				if(i_year[i] == 2009) idx_year = 0;
				if(i_year[i] == 2012) idx_year = 1;
				if(i_year[i] == 2018) idx_year = 2;
				if(i_year[i] == 2019) idx_year = 3;
			//	cout <<  i_year[i] <<  "  Background  : "  << N_BKG[idx_year][0] << " " << N_BKG[idx_year][1] << endl;
				if(fit_flag == 1){
						l1 = - loglike[i][0] + N_BKG[idx_year][0]*loglike[i][1]/Double_t(NN[i][1]) + (Double_t(NN[i][0]) - N_BKG[idx_year][0])*TMath::Log(norm[i][0]);
						l2 = - loglike[i][2] + N_BKG[idx_year][1]*loglike[i][3]/Double_t(NN[i][4]) + (Double_t(NN[i][3]) - N_BKG[idx_year][1])*TMath::Log(norm[i][1]);
				}
				else if(fit_flag == 2){
						l1 = - loglike[i][0] + (Double_t(NN[i][0]))*TMath::Log(norm[i][0]);
						l2 = - loglike[i][2] + (Double_t(NN[i][3]))*TMath::Log(norm[i][1]);
				}
				else{
						cerr << "error fit_flag!" << endl;
				}
				llk += (l1 + l2);
		}

		f = llk;
		if(fit_step%100 == 0){
		std::cout << "Loglike: " << f << std::endl; 
		for( int i = 0; i<10 ; i++ ) cout<<pp[i]<<" ";
		cout << endl;
		}
		fit_step++;
}
//=====================================================================
// input [1] =  0; [2] =  type; [3] = step; [4] = output file
void XiXiMLL(int argc, char** argv, const int index, const int MM){


		ofstream out;
		TString outfile_name = argv[1];
		cout << outfile_name << endl;
		out.open(outfile_name, ios::out | ios::app);
		int year[4];
		fit_flag = atoi(argv[2]);
		if(argc < 4 || argc > 7){
				cerr << "wrong arguments!" << endl;
				exit(1);
		}
		cout << "OK 11111111111" << endl;
		i_year.clear();
		if(argc >= 4 && argc <= 7){
				years = argc - 3;
				for(int i = 0; i < years; i++){
						year[i] = atoi(argv[i+3]);
						if(year[i] != 2009 &&  year[i] != 2012 && year[i] != 2018 && year[i] != 2019 ){
								cerr << "wrong data sets : " << year[i] << endl;
								exit(1);
						}
						i_year.push_back(year[i]);
				}
		}
		ReadData(year, index, MM);
//		for(int i = 0; i < years; i++){
//				for(int j = 0; j < 2; j++){
//						angdis[i][j]->PrintInt();
//				}
//		}
		cout << "OK 11111111113" << endl;
		// instantiating the values to be measured 
		//
		// initial values for fit
		double Jpsi_alpha       = 0.586;  	  // alpha_J/Psi 
		double Jpsi_phi       =  1.121;		  //-TMath::Pi()/4.; // relative phase, Dphi_J/Psi
		double xi_alpha     = -0.3756;  		  // alpha (Sgm->p pi0)
		double xi_phi   = 0.012;   			  // alpha (Sgm->pbar pi0)
		double xib_alpha     = 0.3756;   
		double xib_phi   = -0.012;  
		double L1_alpha     = 0.692;   
		double L2_alpha     = -0.751;   
		double L3_alpha     = 0.751;   
		double L4_alpha     = -0.692;   
		cout << "OK" << endl;
		// cout << argv[1] << endl;
		// fit nr is used to tell which analysis cuts that are used
		TMinuit *minuit=new TMinuit(10);
		Int_t ierflag=0; 
		Double_t arglist[100];
		cout << "OK 11111111111" << endl;
		minuit->SetFCN(fcnMLLG);
		cout << "OK 11111111111" << endl;
		arglist[0]= 0;
		minuit->mnexcm("SET PRINT",arglist,1,ierflag);
		arglist[0]= 0.5;
		minuit->mnexcm("SET ERR",arglist,1,ierflag);
		minuit->mnparm(0, "alpha_jpsi" ,Jpsi_alpha, 0.001, -1., 1., ierflag);
		minuit->mnparm(1, "dphi_jpsi", Jpsi_phi, 0.001, -TMath::Pi(), TMath::Pi(), ierflag);
		cout << "OK 11111111111" << endl;
		minuit->mnparm(2, "xi_alpha" , xi_alpha, 0.001, -1., 0., ierflag);
		minuit->mnparm(3, "xi_phi" , xi_phi, 0.001,  -TMath::Pi(), TMath::Pi(), ierflag);
		minuit->mnparm(4, "xib_alpha" , xib_alpha, 0.001, 0, 1., ierflag);
		minuit->mnparm(5, "xib_phi" , xib_phi, 0.001,  -TMath::Pi(), TMath::Pi(), ierflag);
		minuit->mnparm(6, "L1_alpha" , L1_alpha, 0.001, 0., 1., ierflag);
		minuit->mnparm(7, "L2_alpha" , L2_alpha, 0.001, -1., 0., ierflag);
		minuit->mnparm(8, "L3_alpha" , L3_alpha, 0.001, 1., 0., ierflag);
		minuit->mnparm(9, "L4_alpha" , L4_alpha, 0.001, -1., 0., ierflag);
		cout << "OK 11111111113" << endl;
		//minuit->mnparm(3, "A_CP" , A_CP, 0.001, 0.,0., ierflag); 
		// 	 		minuit->FixParameter(7);
		//		 minuit->FixParameter(9);
		//		 minuit->FixParameter(12);
		//		 minuit->FixParameter(13);
		cout << "OK 11111111114" << endl;
		minuit->mnexcm("MINI",arglist,0,ierflag); //minimization using the migrag
		cout << "OK 11111111115" << endl;
		//limits both 0 implies no limit 
		minuit->mnexcm("MINOS",arglist,0,ierflag);
		cout << "OK 11111111115" << endl;
		minuit->mnmatu(1);
		cout << "OK 11111111115" << endl;
		Double_t fmin, fedm, errdef;
		Int_t   npari, nparx, istat; 
		minuit->mnstat(fmin, fedm, errdef, npari, nparx, istat);
		double res[10], err_res[10];
		for(int p = 0; p < 10; p++)
				minuit->GetParameter(p, res[p], err_res[p]);
		out << fmin << "," << istat << "," << NN[0][0] << ","<< NN[0][1] << ","; 
		out << res[0]<< "," << err_res[0] << "," << res[1]<< "," << err_res[1]<< ","; 
		out << res[2]<< "," << err_res[2] << "," << res[3]<< "," << err_res[3]<< ",";
		out << res[4]<< "," << err_res[4] << "," << res[5]<< "," << err_res[5]<< ",";
		out << res[6]<< "," << err_res[6] << "," << res[7]<< "," << err_res[7]<< ",";
		out << res[8]<< "," << err_res[8] << "," << res[9]<< "," << err_res[9]<< endl;
		out.close();
	//	return 0;
}

int main(int argc, char **argv){
	for(int i  = 0; i < 30; i++){
		XiXiMLL(argc, argv, i, 30);
	}
}

