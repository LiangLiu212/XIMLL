#include "hip/hip_runtime.h"
#include "rootfile.cuh"
#include "TROOT.h"
#include "TMath.h"
#include ""
#include "hip/hip_runtime.h"
#include "floatreduce.h"
using namespace ROOT;

rootfile::~rootfile(){
		delete r1;
}

void rootfile::ReadData(){
		for(int i = 0; i < (N_Sample +1); i++){
				D_Sample[i].datamassn.clear();
				D_Sample[i].mdiymassn.clear();
				D_Sample[i].bkg1massn.clear();
				D_Sample[i].NN.clear();
				for(int j = 0; j < D_Sample[i].m_sample.size(); j++){
						if(m_isIO){
								readDataIO(i, j);
						}
						else {
						readData(i, j);
						}
				}
		}
		cout << "Hello 1" << endl;
        double **temp_angdata[4][2][20]; // define a temporary array
		for(int i = 0; i < (N_Sample +1); i++){
				for(int j = 0; j < D_Sample[i].m_sample.size(); j++){
						int iyear = IndexYear(D_Sample[i].m_year);
						int ich = IndexChannel(D_Sample[i].m_channel);
						int isample = IndexSample(D_Sample[i].m_sample[j]);
						temp_angdata[iyear][ich][isample] = new double * [9];
						for(int l = 0; l < 9; l++){
								*(temp_angdata[iyear][ich][isample] + l) = new double [D_Sample[i].NN[j] + THREADS_PER_BLOCK];
								for(int k = 0; k< D_Sample[i].NN[j]; k ++){
										*(*(temp_angdata[iyear][ich][isample] + l)+k) = *(*(angdata[iyear][ich][isample] + l) + k);
								}
						}
				}
		}

		cout << "Hello 2" << endl;
        for(int i = 0; i < (N_Sample +1); i++){    // copy data from cpu to gpu
                for(int j = 0; j < D_Sample[i].m_sample.size(); j ++){
						int iyear = IndexYear(D_Sample[i].m_year);
						int ich = IndexChannel(D_Sample[i].m_channel);
						int isample = IndexSample(D_Sample[i].m_sample[j]);

                        if(!D_Sample[i].m_sample[j].CompareTo("phsp")) continue;
                        if(!D_Sample[i].m_sample[j].CompareTo("mdiy")) continue;
                        int size1 = (D_Sample[i].NN[j] + THREADS_PER_BLOCK) *sizeof(double);
                        gpu_angdata[iyear][ich][isample] = new double * [9];
                        for(int l = 0; l < 9; l++){
                                hipMalloc( (void **) &(*(gpu_angdata[iyear][ich][isample] + l)), size1 );
                                hipMemcpy( *(gpu_angdata[iyear][ich][isample] + l), *(temp_angdata[iyear][ich][isample] + l), size1, hipMemcpyHostToDevice );
                                delete [] *(temp_angdata[iyear][ich][isample] + l);
                        }
                        hipMalloc( (void **) &gpu_Matrix[iyear][ich][isample], size1 * MATRIX_SIZE  );
                        hipMalloc( (void **) &gpu_amp[iyear][ich][isample], size1 * MATRIX_SIZE  );
                        out_amp[iyear][ich][isample] = new double [D_Sample[i].NN[j] + THREADS_PER_BLOCK];
                }
        }
		hipMalloc((void **)&t_sumAmp, sizeof(double));
		setBKG();

}


void rootfile::FreeMemory(){
		for(int i = 0; i < (N_Sample +1); i++){
				for(int j = 0; j < D_Sample[i].m_sample.size(); j ++){
						int iyear = IndexYear(D_Sample[i].m_year);
						int ich = IndexChannel(D_Sample[i].m_channel);
						int isample = IndexSample(D_Sample[i].m_sample[j]);
                        if(!D_Sample[i].m_sample[j].CompareTo("phsp")) continue;
                        if(!D_Sample[i].m_sample[j].CompareTo("mdiy")) continue;
                        for(int l = 0; l < 9; l++){
                                hipFree((*(gpu_angdata[iyear][ich][isample] + l)));
                        }
                        hipFree(gpu_Matrix[iyear][ich][isample]);
                        hipFree(gpu_amp[iyear][ich][isample]);
                        delete [] out_amp[iyear][ich][isample];
				}
		}
}

void rootfile::readData(int index, int jndex){
		TString path = "/data/liul/workarea/XIXI/fit/boost";
		TString year = D_Sample[index].m_year;
		TString channel = D_Sample[index].m_channel;
		TString version = D_Sample[index].m_version;
		TString sample = D_Sample[index].m_sample[jndex];
		TString insample = sample;
		if(!sample.CompareTo("phsp")){
				if(!m_norm.CompareTo("mdiy")){
				insample = "mdiy";
				}
				else if(!m_norm.CompareTo("phsp")){
						insample = "phsp";
				}
		}
		if(!sample.CompareTo("bkg1")){
				insample = "mdiy";
		}
		if(!sample.CompareTo("bkg2")){
				insample = "data";
		}
		if(!sample.CompareTo("sideband")){
				insample = "data";
		}
		TString infile  = path + "/" + year + "/" + channel + "/" + version + "/" + insample + "/boost.root";
		cout << infile << " ==> " << sample << endl;
//#define read_data
#ifndef read_data
		Int_t NN1;
		std::vector<double> gD1Xithe;
		std::vector<double> gD1Lthe;
		std::vector<double> gD1Lphi;
		std::vector<double> gD1Lbthe;
		std::vector<double> gD1Lbphi;
		std::vector<double> gD1pthe;
		std::vector<double> gD1pphi;
		std::vector<double> gD1apthe;
		std::vector<double> gD1apphi;

		gD1Xithe.clear();
		gD1Lthe.clear();
		gD1Lphi.clear();
		gD1Lbthe.clear();
		gD1Lbphi.clear();
		gD1pthe.clear();
		gD1pphi.clear();
		gD1apthe.clear();
		gD1apphi.clear();



		TFile *f1 = new TFile(infile, "read");
		TTree *t1 = (TTree*)f1->Get("xixi");
		t1->SetBranchAddress("the", &the);
		t1->SetBranchAddress("Lthe", &Lthe);
		t1->SetBranchAddress("Lphi", &Lphi);
		t1->SetBranchAddress("Lbthe", &Lbthe);
		t1->SetBranchAddress("Lbphi", &Lbphi);
		t1->SetBranchAddress("pthe", &pthe);
		t1->SetBranchAddress("pphi", &pphi);
		t1->SetBranchAddress("apthe", &apthe);
		t1->SetBranchAddress("apphi", &apphi);
		t1->SetBranchAddress("runNo", &m_runNo);
		t1->SetBranchAddress("LmdDL", &m_LmdDL);
		t1->SetBranchAddress("XiDL", &m_XiDL);
		t1->SetBranchAddress("XiCosTheta", &m_XiCosTheta);
		t1->SetBranchAddress("mXi2", &m_mXi2);
		t1->SetBranchAddress("mXi1", &m_mXi1);
		t1->SetBranchAddress("mLmd1", &m_mLmd1);
		t1->SetBranchAddress("mn", &m_mn);
		t1->SetBranchAddress("chi2kmf", &m_chi2kmf);
		t1->SetBranchAddress("chi2Xi", &m_chi2Xi);
		t1->SetBranchAddress("chi2Lmd", &m_chi2Lmd);
		t1->SetBranchAddress("angle_gam1", &m_angle_gam1);
		t1->SetBranchAddress("angle_gam2", &m_angle_gam2);
		t1->SetBranchAddress("lmd_p", &m_lmd_p);
		t1->SetBranchAddress("lmd_cos", &m_lmd_cos);
		t1->SetBranchAddress("pion1_1_cos", &m_pion1_1_cos);
		t1->SetBranchAddress("pion1_1_pt", &m_pion1_1_pt);
		t1->SetBranchAddress("pion1_2_cos", &m_pion1_2_cos);
		t1->SetBranchAddress("pion1_2_pt", &m_pion1_2_pt);
		t1->SetBranchAddress("pion2_1_cos", &m_pion2_1_cos);
		t1->SetBranchAddress("pion2_1_pt", &m_pion2_1_pt);
		t1->SetBranchAddress("pion0_cos", &m_pion0_cos);
		t1->SetBranchAddress("pion0_rho", &m_pion0_rho);

		int nn = 0;
		int NEvt = t1->GetEntries();
		int low = 0;
		int high = NEvt;
		r1->SetSeed(3251);

		for(int i = low; i <  high; i++){
				t1->GetEntry(i);
				if(!Selection(year, channel, sample, index)) continue;
				if(!sample.CompareTo("phsp")){
						int factor = CorrFactor(year, channel, sample);
						for(int iCorr = 0; iCorr < factor; iCorr++){
								gD1Xithe.push_back(the);
								gD1Lthe.push_back(Lthe);
								gD1Lphi.push_back(Lphi);
								gD1Lbthe.push_back(Lbthe);
								gD1Lbphi.push_back(Lbphi);
								gD1pthe.push_back(pthe);
								gD1pphi.push_back(pphi);
								gD1apthe.push_back(apthe);
								gD1apphi.push_back(apphi);
								if(!m_norm.CompareTo("phsp")){
										angdis[IndexYear(year)][IndexChannel(channel)]->AddToIntegral(the, Lthe, Lphi, Lbthe, Lbphi, pthe, pphi, apthe, apphi);
								}
								else if(!m_norm.CompareTo("mdiy")){
										angdis[IndexYear(year)][IndexChannel(channel)]->AddToIntegralmDIY(the, Lthe, Lphi, Lbthe, Lbphi, pthe, pphi, apthe, apphi);
								}
								nn++;
						}
				}
				else if(!sample.CompareTo("bkg1")){
						int factor = CorrFactor(year, channel, sample);
						for(int iCorr = 0; iCorr < factor; iCorr++){
								gD1Xithe.push_back(the);
								gD1Lthe.push_back(Lthe);
								gD1Lphi.push_back(Lphi);
								gD1Lbthe.push_back(Lbthe);
								gD1Lbphi.push_back(Lbphi);
								gD1pthe.push_back(pthe);
								gD1pphi.push_back(pphi);
								gD1apthe.push_back(apthe);
								gD1apphi.push_back(apphi);
								nn++;
						}
				}
				else{
						gD1Xithe.push_back(the);
						gD1Lthe.push_back(Lthe);
						gD1Lphi.push_back(Lphi);
						gD1Lbthe.push_back(Lbthe);
						gD1Lbphi.push_back(Lbphi);
						gD1pthe.push_back(pthe);
						gD1pphi.push_back(pphi);
						gD1apthe.push_back(apthe);
						gD1apphi.push_back(apphi);
						nn++;
				}
		}
		D_Sample[index].NN.push_back(nn);

		int iyear = IndexYear(year);
		int ich = IndexChannel(channel);
		int isample = IndexSample(sample);
		cout << "Hello " << iyear << ich << isample << endl;

		for(int i = 0; i < nn; i++){
				*(*(angdata[iyear][ich][isample]+0)+i) = gD1Xithe[i];
				*(*(angdata[iyear][ich][isample]+1)+i) = gD1Lthe[i];
				*(*(angdata[iyear][ich][isample]+2)+i) = gD1Lphi[i];
				*(*(angdata[iyear][ich][isample]+3)+i) = gD1Lbthe[i];
				*(*(angdata[iyear][ich][isample]+4)+i) = gD1Lbphi[i];
				*(*(angdata[iyear][ich][isample]+5)+i) = gD1pthe[i];
				*(*(angdata[iyear][ich][isample]+6)+i) = gD1pphi[i];
				*(*(angdata[iyear][ich][isample]+7)+i) = gD1apthe[i];
				*(*(angdata[iyear][ich][isample]+8)+i) = gD1apphi[i];
		}


		cout << nn << endl;
		f1->Close();
#endif
}

int rootfile::RunHigh(const TString year){
		if(!year.CompareTo("2009")){
				return 11000;
		}
		else if(!year.CompareTo("2012")){
				return 28400;
		}
		else if(!year.CompareTo("2018")){
				return 56646;
		}
		else if(!year.CompareTo("2019")){
				return 59115;
		}
}


int rootfile::RunLow(const TString year){
		if(!year.CompareTo("2009")){
				return 9800;
		}
		else if(!year.CompareTo("2012")){
				return 27100;
		}
		else if(!year.CompareTo("2018")){
				return 52840;
		}
		else if(!year.CompareTo("2019")){
				return 56778;
		}
}

bool rootfile::Selection(const TString year, const TString channel, const TString sample, const int index){
		if(!sample.CompareTo("sideband")){
				if(abs(m_runNo) > RunLow(year) && abs(m_runNo) < RunHigh(year) && m_LmdDL > cut_LmdDL && m_XiDL > cut_XiDL && fabs(m_XiCosTheta) < cut_XiCosTheta){
						if(fabs(m_mXi2 - 1.32171) < (3*cut_mXi + 0.005) && fabs(m_mXi2 - 1.32171) > (cut_mXi + 0.005) && fabs(m_mXi1 - 1.32171) < (3*cut_mXi + 0.005) && fabs(m_mXi1 - 1.32171) > (cut_mXi + 0.005)  && fabs(m_mLmd1 - 1.1157) < cut_mLmd1 ){
								if(m_chi2kmf < cut_chi2kmf && m_chi2Xi < cut_chi2Xi && m_chi2Lmd < cut_chi2Lmd){
										if(!channel.CompareTo("xixipm")){
												if(fabs(cos(pthe) - cut_ncos) < cut_deltancos && fabs(cos(apthe) - cut_pbarcos) < cut_deltapbarcos){
														D_Sample[index].bkgsideband.push_back(m_mn);
														if(m_mn > cut_mn1 && m_mn < cut_mn2){
																return true;
														}															
												}
										}
										else if(!channel.CompareTo("xixipp")){
												if(fabs(cos(pthe) - cut_pcos) < cut_deltapcos && fabs(cos(apthe) - cut_nbarcos) < cut_deltanbarcos){
														D_Sample[index].bkgsideband.push_back(m_mn);
														if(m_mn > cut_mn1 && m_mn < cut_mn2){
																return true;
														}															
												}
										}
								}
						}
				}
		}

		else if(abs(m_runNo) > RunLow(year) && abs(m_runNo) < RunHigh(year) && m_LmdDL > cut_LmdDL && m_XiDL > cut_XiDL && fabs(m_XiCosTheta) < cut_XiCosTheta){
				if(fabs(m_mXi2 - 1.32171) < cut_mXi && fabs(m_mXi1 - 1.32171) < cut_mXi && fabs(m_mLmd1 - 1.1157) < cut_mLmd1 ){
						if(m_chi2kmf < cut_chi2kmf && m_chi2Xi < cut_chi2Xi && m_chi2Lmd < cut_chi2Lmd){
								if(!channel.CompareTo("xixipm")){ 
										if(fabs(cos(pthe) - cut_ncos) < cut_deltancos && fabs(cos(apthe) - cut_pbarcos) < cut_deltapbarcos){
												if(!sample.CompareTo("data")){
														D_Sample[index].datamassn.push_back(m_mn);
														if(m_mn > cut_mn1 && m_mn < cut_mn2){
																return true;
														}
												}
												else if(!sample.CompareTo("mdiy")){
														if(m_angle_gam1 < 0.3 && m_angle_gam2 < 0.3){
																D_Sample[index].mdiymassn.push_back(m_mn);
																if(m_mn > cut_mn1 && m_mn < cut_mn2){
																		return true;
																}
														}
												}
												else if(!sample.CompareTo("phsp")){
														if(m_angle_gam1 < 0.3 && m_angle_gam2 < 0.3){
																if(m_mn > cut_mn1 && m_mn < cut_mn2){
																		return true;
																}
														}
												}
												else if(!sample.CompareTo("bkg1")){
														if(m_angle_gam1 > 0.3 || m_angle_gam2 > 0.3){
																D_Sample[index].bkg1massn.push_back(m_mn);
																if(m_mn > cut_mn1 && m_mn < cut_mn2){
																		return true;
																}
														}
												}
												else if(!sample.CompareTo("bkg2")){
														if((m_mn > 0.905 && m_mn < 0.920) || (m_mn > 0.960 && m_mn < 0.975)){
																return true;
														}
												}
												else if(!sample.CompareTo("etac")){
														D_Sample[index].bkgetac.push_back(m_mn);
														if(m_mn > cut_mn1 && m_mn < cut_mn2){
																return true;
														}
												}
												else if(!sample.CompareTo("charge")){
														D_Sample[index].bkgcharged.push_back(m_mn);
														if(m_mn > cut_mn1 && m_mn < cut_mn2){
																return true;
														}
												}
										}
								}
								else if(!channel.CompareTo("xixipp")){ 
										if(fabs(cos(pthe) - cut_pcos) < cut_deltapcos && fabs(cos(apthe) - cut_nbarcos) < cut_deltanbarcos){
												if(!sample.CompareTo("data")){
														D_Sample[index].datamassn.push_back(m_mn);
														if(m_mn > cut_mn1 && m_mn < cut_mn2){
																return true;
														}
												}
												else if(!sample.CompareTo("mdiy")){
														if(m_angle_gam1 < 0.3 && m_angle_gam2 < 0.3){
																D_Sample[index].mdiymassn.push_back(m_mn);
																if(m_mn > cut_mn1 && m_mn < cut_mn2){
																		return true;
																}
														}
												}
												else if(!sample.CompareTo("phsp")){
														if(m_angle_gam1 < 0.3 && m_angle_gam2 < 0.3){
																if(m_mn > cut_mn1 && m_mn < cut_mn2){
																		return true;
																}
														}
												}
												else if(!sample.CompareTo("bkg1")){
														if(m_angle_gam1 > 0.3 || m_angle_gam2 > 0.3){
																D_Sample[index].bkg1massn.push_back(m_mn);
																if(m_mn > cut_mn1 && m_mn < cut_mn2){
																		return true;
																}
														}
												}
												else if(!sample.CompareTo("bkg2")){
														if((m_mn > 0.905 && m_mn < 0.920) || (m_mn > 0.960 && m_mn < 0.975)){
																return true;
														}
												}
												else if(!sample.CompareTo("etac")){
														D_Sample[index].bkgetac.push_back(m_mn);
														if(m_mn > cut_mn1 && m_mn < cut_mn2){
																return true;
														}
												}
												else if(!sample.CompareTo("charge")){
														D_Sample[index].bkgcharged.push_back(m_mn);
														if(m_mn > cut_mn1 && m_mn < cut_mn2){
																return true;
														}
												}
										}
								}
						}
				}
		}
		else {
				return false;
		}
}

void rootfile::MassFit(){
		for(int i = 0; i < (N_Sample +1); i++){
				if(m_isIO){
						massFitIO(i);
				}
				else{
						massFit(i);
				}
		}
		for(int i = 0; i < (N_Sample +1); i++){
				cout << "sample " << i << " : " << D_Sample[i].n_bkg << endl;
		}
}


void rootfile::massFitIO(const int index){
		cout << index << endl;
		RooRealVar mn("mn", "M_{n} (GeV/c^{2})",0.90,0.985);
		RooDataSet signal("signal", "signal", mn);
		RooDataSet data("data", "data", mn);
		RooDataSet bkg("bkg", "bkg", mn);
		RooDataSet bkgetac("bkgetac", "bkgetac", mn);
		RooDataSet bkgcharged("bkgcharged", "bkgcharged", mn);
		RooDataSet bkgsideband("bkgsideband", "bkgsideband", mn);
		for(int i = 0; i < D_Sample[index].datamassn.size(); i++){
				mn = D_Sample[index].datamassn[i];
				data.add(mn);
		}

		for(int i = 0; i < D_Sample[index].mdiymassn.size(); i++){
				mn = D_Sample[index].mdiymassn[i];
				if(i > 10000) break;
				signal.add(mn);
		}

		for(int i = 0; i < D_Sample[index].bkg1massn.size(); i++){
				mn = D_Sample[index].bkg1massn[i];
				if(i > 10000) break;
				bkg.add(mn);
		}
		cout << "kernel estimation p.d.f of background......" << endl;
		//      RooKeysPdf keysbkg("keysbkg", "keysbkg", mn, bkg, RooKeysPdf::MirrorLeft, 1);

		//#define bkg
#ifndef bkg
		cout << "kernel estimation p.d.f of signal......" << endl;
		RooKeysPdf keysshape("keysshape", "keysshape", mn, signal, RooKeysPdf::MirrorLeft, 3);




		RooRealVar mean2("mean2", "mean2", 0.001, -0.05, 0.05);
		RooRealVar sigma2("sigma2", "sigma2",0.001, 0, 0.05);
		RooGaussian ga1("ga1", "ga1", mn, mean2, sigma2);
		mn.setBins(10000, "cache");
		RooFFTConvPdf shape("shape", "shape", mn, keysshape, ga1);


		RooRealVar p0("p0", "poly 0", 0.5, -50., 50.);
		RooRealVar p1("p1", "poly 1", 0.5, -50., 50.);
		RooRealVar p2("p2", "poly 2", -1.5, -10., 10.);
		RooRealVar p3("p3", "poly 3", 0.5, -50., 50.);
		RooRealVar p4("p4", "poly 4", 0, -900000., 900000.);
		RooRealVar m01("m01", "m01", 0.981);
		RooRealVar k1("k1", "k1", -5, -50, -0.0);
		RooRealVar pp1("pp1", "pp1", 0.5, 0, 1);
		RooArgusBG argus1("argus1", "argus1", mn, m01, k1);
		RooRealVar mean3("mean3", "mean3", 0.935, 0, 2);
		RooRealVar sigma3("sigma3", "sigma3",0.2, 0, 50);
		RooGaussian ga3("ga3", "ga3", mn, mean3, sigma3);

		RooArgusPoly argus3("argus3", "argus3", mn, m01, k1, pp1, p1, p2, p3);

		RooRealVar fra1("fra1", "fra1", 1000000, 0, 20000000);
		RooRealVar fra2("fra2", "fra2", 160000, 0, 20000000);
		RooAddPdf sum("sum", "sig+bak", RooArgList(shape, argus3), RooArgList(fra1, fra2));

		RooFitResult *resbkg = argus3.fitTo(bkg, Minimizer("Minuit2", "Migrad"));
		k1.setConstant(kTRUE);
		pp1.setConstant(kTRUE);
		p0.setConstant(kTRUE);
		p1.setConstant(kTRUE);
		p2.setConstant(kTRUE);
		p3.setConstant(kTRUE);
		RooFitResult *res = sum.fitTo(data, Extended(), Save(1), Minimizer("Minuit2", "Migrad"));
		mn.setRange("bkgrange2", cut_mn1, cut_mn2);
		mn.setRange("bkgrange1", 0.9, 0.98);
		RooAbsReal *intpolyX10 = shape.createIntegral(mn,  NormSet(mn), Range("bkgrange2"));
		RooAbsReal *intpolyX11 = argus3.createIntegral(mn,  NormSet(mn), Range("bkgrange2"));
		D_Sample[index].n_signal = intpolyX10->getVal()*fra1.getVal();
		D_Sample[index].n_bkg = intpolyX11->getVal()*fra2.getVal();
		D_Sample[index].n_signalerr = fra1.getError();
		D_Sample[index].n_bkgerr = fra2.getError();
		/*

		   TCanvas *c1 = new TCanvas("c1", "c1", 800, 600);

		   RooPlot *xframe = mn.frame(Title("Mass Fit p.d.f."));

		   data.plotOn(xframe);

		   sum.plotOn(xframe, LineColor(4));

		   sum.plotOn(xframe, Components("keysshape"), LineColor(2), LineStyle(2));

		   sum.plotOn(xframe, Components("argus3"), LineColor(6), LineStyle(3));
		   sum.plotOn(xframe, Components("keysbkgetac"), LineColor(7), LineStyle(3));

		   sum.plotOn(xframe, Components("keysbkgcharged"), LineColor(8), LineStyle(3));
		   sum.plotOn(xframe, Components("keysbkgsideband"), LineColor(8), LineStyle(3));
		   xframe->Draw();
		   c1->SaveAs("massfit2018.eps");
		 */

#endif
}




void rootfile::massFit(const int index){
		cout << index << endl;
		double inisignal = 0;
		double inibkg = 0;
		RooRealVar mn("mn", "M_{n} (GeV/c^{2})",0.90,0.985);
		RooDataSet signal("signal", "signal", mn);
		RooDataSet data("data", "data", mn);
		RooDataSet bkg("bkg", "bkg", mn);
		RooDataSet bkgetac("bkgetac", "bkgetac", mn);
		RooDataSet bkgcharged("bkgcharged", "bkgcharged", mn);
		RooDataSet bkgsideband("bkgsideband", "bkgsideband", mn);
		inisignal = D_Sample[index].datamassn.size()*0.825 * 1.2;
		inibkg = D_Sample[index].datamassn.size()*0.125 ;
		for(int i = 0; i < D_Sample[index].datamassn.size(); i++){
				mn = D_Sample[index].datamassn[i];
				data.add(mn);
		}

		for(int i = 0; i < D_Sample[index].mdiymassn.size(); i++){
				mn = D_Sample[index].mdiymassn[i];
				if(i == 10000) break;
				signal.add(mn);
		}

		for(int i = 0; i < D_Sample[index].bkg1massn.size(); i++){
				mn = D_Sample[index].bkg1massn[i];
				if(i == 15000) break;
				bkg.add(mn);
		}
		cout << "kernel estimation p.d.f of background......" << endl;
		//      RooKeysPdf keysbkg("keysbkg", "keysbkg", mn, bkg, RooKeysPdf::MirrorLeft, 1);

		double f_etac = 0, f_charged = 0, f_sideband = 0;
		for(int i = 0; i < D_Sample[index].bkgetac.size(); i++){
				mn = D_Sample[index].bkgetac[i];
				if(D_Sample[index].bkgetac[i] < 0.90) continue;
				f_etac += 1.0;
				bkgetac.add(mn);
		}
		f_etac = setBKGMassFit(f_etac, D_Sample[index].m_year, D_Sample[index].m_channel, "etac");
		cout << "kernel estimation p.d.f of background......" << endl;
		RooKeysPdf keysbkgetac("keysbkgetac", "keysbkgetac", mn, bkgetac, RooKeysPdf::MirrorBoth, 1);
		for(int i = 0; i < D_Sample[index].bkgcharged.size(); i++){
				mn = D_Sample[index].bkgcharged[i];
				if(D_Sample[index].bkgcharged[i] < 0.90) continue;
				f_charged += 1.0;
				bkgcharged.add(mn);
		}
		f_charged = setBKGMassFit(f_charged, D_Sample[index].m_year, D_Sample[index].m_channel, "charge");

		cout << "kernel estimation p.d.f of background......" << endl;
		RooKeysPdf keysbkgcharged("keysbkgcharged", "keysbkgcharged", mn, bkgcharged, RooKeysPdf::MirrorBoth, 1);


		for(int i = 0; i < D_Sample[index].bkgsideband.size(); i++){
				mn = D_Sample[index].bkgsideband[i];
				if(D_Sample[index].bkgsideband[i] < 0.90) continue;
				f_sideband += 1.0;
				bkgsideband.add(mn);
		}
		f_sideband = f_sideband * 0.25;  // FIXME: Just a test 
		cout << "kernel estimation p.d.f of background......" << endl;
		RooKeysPdf keysbkgsideband("keysbkgsideband", "keysbkgsideband", mn, bkgsideband, RooKeysPdf::MirrorBoth, 1);

		//#define bkg
#ifndef bkg
		cout << "kernel estimation p.d.f of signal......" << endl;
		RooKeysPdf keysshape("keysshape", "keysshape", mn, signal, RooKeysPdf::MirrorLeft, 3);




		RooRealVar mean2("mean2", "mean2", 0.001, -0.05, 0.05);
		RooRealVar sigma2("sigma2", "sigma2",0.001, 0, 0.05);
		RooGaussian ga1("ga1", "ga1", mn, mean2, sigma2);
		//		mn.setBins(10000, "cache");
		RooFFTConvPdf shape("shape", "shape", mn, keysshape, ga1);


		RooRealVar p0("p0", "poly 0", 0.5, -5000., 5000.);
		RooRealVar p1("p1", "poly 1", 0.5, -5000., 5000.);
		RooRealVar p2("p2", "poly 2", 0.5, -5000., 5000.);
		RooRealVar p3("p3", "poly 3", 0.5, -5000., 5000.);
		RooRealVar p4("p4", "poly 4", 0, -900000., 900000.);
		RooRealVar m01("m01", "m01", 0.981);
		RooRealVar k1("k1", "k1", -5, -10, -0.0);
		RooRealVar pp1("pp1", "pp1", 0.5, 0.0, 1.0);
		RooArgusBG argus1("argus1", "argus1", mn, m01, k1);
		RooRealVar mean3("mean3", "mean3", 0.935, 0, 2);
		RooRealVar sigma3("sigma3", "sigma3",0.2, 0, 50);
		RooGaussian ga3("ga3", "ga3", mn, mean3, sigma3);

		RooArgusPoly argus3("argus3", "argus3", mn, m01, k1, pp1, p1, p2, p3);

		RooRealVar fra1("fra1", "fra1", inisignal, 0, 4000000);
		RooRealVar fra2("fra2", "fra2", inibkg, 0, 400000);
		RooRealVar fra3("fra3", "fra3", f_etac);
		RooRealVar fra4("fra4", "fra4", f_charged);
		RooRealVar fra5("fra5", "fra5", f_sideband);
		RooAddPdf sum("sum", "sig+bak", RooArgList(shape, argus3, keysbkgetac, keysbkgcharged, keysbkgsideband), RooArgList(fra1, fra2, fra3, fra4, fra5));
		RooAddPdf sum2("sum2", "sig+bak", RooArgList(shape, argus3, keysbkgetac, keysbkgcharged), RooArgList(fra1, fra2, fra3, fra4));

		RooFitResult *resbkg = argus3.fitTo(bkg);
		//	resbkg->Print("v");
		cout << "Finish model background" << endl;

		k1.setConstant(kTRUE);
		pp1.setConstant(kTRUE);
		p0.setConstant(kTRUE);
		p1.setConstant(kTRUE);
		p2.setConstant(kTRUE);
		p3.setConstant(kTRUE);

		cout << "Number of background ==> " << f_etac << "  " << f_charged << "  " << f_sideband << endl;
		if(IndexSample("sideband") != -1) {
				RooFitResult *res = sum.fitTo(data,  Extended(), Save(1), Minimizer("Minuit2", "Migrad"));
				int fitstatus = res->status();
				int fit_iter = 0;  // the count of fit if status > 1
				while(fitstatus > 1){
					//	fra1.setVal(fra1.getVal());
					//	fra2.setVal(fra2.getVal());
						res = sum.fitTo(data,  Extended(), Save(1), Minimizer("Minuit2", "Migrad"));
						fitstatus = res->status();
						cout << "Status : " << fitstatus << endl;
						fit_iter++;
						if(fit_iter > 5) break;
				}
				//	res->Print("v");
				cout << "Finish mass background  " << D_Sample[index].m_year << "  " << D_Sample[index].m_channel << "  " << res->status() << endl;
		}
		else{
				RooFitResult *res = sum2.fitTo(data,  Extended(), Save(1), Minimizer("Minuit2", "Migrad"));
		}
		mn.setRange("bkgrange2", cut_mn1, cut_mn2);
		mn.setRange("bkgrange1", 0.9, 0.98);
		RooAbsReal *intpolyX10 = shape.createIntegral(mn,  NormSet(mn), Range("bkgrange2"));
		RooAbsReal *intpolyX11 = argus3.createIntegral(mn,  NormSet(mn), Range("bkgrange2"));
		D_Sample[index].n_signal = intpolyX10->getVal()*fra1.getVal();
		D_Sample[index].n_bkg = intpolyX11->getVal()*fra2.getVal();
		D_Sample[index].n_signalerr = fra1.getError();
		D_Sample[index].n_bkgerr = fra2.getError();

		/*
		   TCanvas *c1 = new TCanvas("c1", "c1", 800, 600);

		   RooPlot *xframe = mn.frame(Title("Mass Fit p.d.f."));

		   data.plotOn(xframe);

		   sum.plotOn(xframe, LineColor(4));

		   sum.plotOn(xframe, Components("keysshape"), LineColor(2), LineStyle(2));

		   sum.plotOn(xframe, Components("argus3"), LineColor(6), LineStyle(3));
		   sum.plotOn(xframe, Components("keysbkgetac"), LineColor(7), LineStyle(3));

		   sum.plotOn(xframe, Components("keysbkgcharged"), LineColor(8), LineStyle(3));
		   sum.plotOn(xframe, Components("keysbkgsideband"), LineColor(8), LineStyle(3));
		   xframe->Draw();
		   c1->SaveAs("/home/liul/workarea/XiXi/MLL/XIXIRUN/RUNDATA/v9/massfit2018.eps");
		 */


#endif
}

// ==================  GPU MEMORY =============

void rootfile::InitialMemory(){
		for(int i = 0; i < 4; i++){
				for(int ch = 0; ch < 2; ch++){
						for(int j = 0; j < 10; j ++){
								angdata[i][ch][j] = new double * [10];
								for(int l = 0; l < 10; l++){
										*(angdata[i][ch][j] + l) =  new double [NUM];
								}
						}
				}
		}
		TString s_corr = "/data/liul/workarea/XIXI/fit/boost/correctionv8.root";
		fbkgcorr= new TFile(s_corr, "read");
		fpicorr= new TFile("/data/liul/workarea/XIXI/fit/boost/picorrv8.root", "read");
		fpi0corr= new TFile("/data/liul/workarea/XIXI/fit/boost/pion0corrv8.root", "read");



		TString iniyear[4] = {"2009", "2012", "2018", "2019"};
		TString inich[2] = {"xixipm", "xixipp"};
		r1 = new TRandom();

		for(int i = 0; i < 4; i++){
				for(int j = 0; j < 2; j ++){
						angdis[i][j] = new AngDisXiXi();
						angdis[i][j]->InitialInt();
						angdis[i][j]->InitialIntmDIY();
						double pp[2][8];
						pp[0][0] = 0.611;   pp[1][0] = 0.611;
						pp[0][1] = 1.2665;   pp[1][1] = 1.2665;
						pp[0][2] = -0.3722;  pp[1][2] = -0.3722;
						pp[0][3] = -0.0154;    pp[1][3] = -0.0154;
						pp[0][4] = 0.3722;   pp[1][4] = 0.3722;
						pp[0][5] = 0.0154;    pp[1][5] = 0.0154;
						pp[0][6] = 0.6727;   pp[1][6] = 0.7703;
						pp[0][7] = -0.7703;  pp[1][7] = -0.6727;

						angdis[i][j]->SetParameter(pp[j]);
						angdis[i][j]->InitialInt();
						angdis[i][j]->InitialIntmDIY();

						hcorrbkg[i][j] = (TH2D*)fbkgcorr->Get(inich[j] + iniyear[i] + "bkg");
						hpicorr[i][j]  = (TH2D*)fpicorr->Get("picorr" + iniyear[i] + inich[j]);
						hpi0corr[i][j] = (TH2D*)fpi0corr->Get("pion0" + iniyear[i] + inich[j]);

						if(m_seed/1000 == 2){  // for lamdba 2xxx
								r1->SetSeed(m_seed);
								for(int m = 0; m < 10; m++){
										for(int n = 0; n <7; n++){
												double mean = hpicorr[i][j]->GetBinContent(m+1, n+1);
												double meanerr = hpicorr[i][j]->GetBinError(m+1, n+1);
												hpicorr[i][j]->SetBinContent(m+1, n+1, r1->Gaus(mean,meanerr));
										}
								}
						}

						if(m_seed/1000 == 3){  // for lamdba 2xxx
								r1->SetSeed(m_seed);
								for(int m = 0; m < 10; m++){
										for(int n = 0; n <7; n++){
												double mean = hpi0corr[i][j]->GetBinContent(m+1, n+1);
												double meanerr = hpi0corr[i][j]->GetBinError(m+1, n+1);
												hpi0corr[i][j]->SetBinContent(m+1, n+1, r1->Gaus(mean,meanerr));
										}
								}
						}

				}
		}	
}



double rootfile::cpufcnmll(double *pp){
		double pp1[8], pp2[8];
		for(int i = 0; i < 6; i++){
				pp1[i] = pp[i];
				pp2[i] = pp[i];
		}
		pp1[6] = pp[6]; pp1[7] = pp[7];
		pp2[6] = pp[8]; pp2[7] = pp[9];

		AA_parameter aa_para;
		aa_para.alpha_jpsi = pp[0];
		aa_para.phi_jpsi = pp[1];
		aa_para.alpha_xi = pp[2];
		aa_para.phi_xi = pp[3];
		aa_para.alpha_xibar = pp[4];
		aa_para.phi_xibar = pp[5];
		aa_para.alpha1_lambda = pp[6];
		aa_para.alpha1_lambdabar = pp[7];
		aa_para.alpha2_lambda = pp[8];
		aa_para.alpha2_lambdabar = pp[9];
		hipError_t cudaStatus;
		clock_t start,end;
		double loglike[4][2][6];
		for(int i = 0; i < 4; i++){
				angdis[i][0]->SetParameter(pp1);
				angdis[i][1]->SetParameter(pp2);
		}


		double norm[4][2];
		for(int i = 0; i < (N_Sample +1); i++){
				int iyear = IndexYear(D_Sample[i].m_year);
				int ich = IndexChannel(D_Sample[i].m_channel);
				norm[iyear][ich] = 0;
				if(!m_norm.CompareTo("phsp")){
						norm[iyear][ich] = angdis[iyear][ich]->CalcToIntegral()/Double_t(D_Sample[i].NN[IndexSample("phsp")]);
				}
				else if(!m_norm.CompareTo("mdiy")){
						norm[iyear][ich] = angdis[iyear][ich]->CalcToIntegralmDIY()/Double_t(D_Sample[i].NN[IndexSample("phsp")]);
				}
		}



		start = clock();

		for(int i = 0; i < (N_Sample +1); i++){
				for(int j = 0; j < D_Sample[i].m_sample.size(); j++){
						int iyear = IndexYear(D_Sample[i].m_year);
						int ich = IndexChannel(D_Sample[i].m_channel);
						int isample = IndexSample(D_Sample[i].m_sample[j]);
						if(!D_Sample[i].m_sample[j].CompareTo("phsp")) continue;
						if(!D_Sample[i].m_sample[j].CompareTo("mdiy")) continue;
						loglike[iyear][ich][isample] = 0;
						for(int evt = 0; evt < D_Sample[i].NN[j]; evt++){
								*(out_amp[iyear][ich][isample] + evt) = angdis[iyear][ich]->Amp(*(*(angdata[iyear][ich][isample]+0)+evt),
												*(*(angdata[iyear][ich][isample]+1)+evt),
												*(*(angdata[iyear][ich][isample]+2)+evt),
												*(*(angdata[iyear][ich][isample]+3)+evt),
												*(*(angdata[iyear][ich][isample]+4)+evt),
												*(*(angdata[iyear][ich][isample]+5)+evt),
												*(*(angdata[iyear][ich][isample]+6)+evt),
												*(*(angdata[iyear][ich][isample]+7)+evt),
												*(*(angdata[iyear][ich][isample]+8)+evt));
								if(*(out_amp[iyear][ich][isample] + evt) <= 0){ cout << "data : " << *(out_amp[iyear][ich][isample] + evt) << endl;  return 0; }
								loglike[iyear][ich][isample] += TMath::Log((*(out_amp[iyear][ich][isample] + evt))/norm[iyear][ich]);
						}
				}
		}
		end = clock();
		double time3 = ((double)(end-start))/CLOCKS_PER_SEC;
		cout << "CPU 3: running kernel " << time3 << " seconds" << endl;
	//	//	exit(1);
		double llk = 0;
		double l1 = 0;

		double scale_factor_1[4][2];
		double scale_factor_2[4][2];
		for(int i = 0; i < 4; i++){
				for(int j = 0; j < 2; j++){
						scale_factor_1[i][j] = 0;
						scale_factor_2[i][j] = 0;
				}
		}




		for(int i = 0; i < (N_Sample +1); i++){
				int iyear = IndexYear(D_Sample[i].m_year);
				int ich = IndexChannel(D_Sample[i].m_channel);
				for(int j = 0; j < D_Sample[i].m_sample.size(); j++){
						if(!D_Sample[i].m_sample[j].CompareTo("data")){
								scale_factor_1[iyear][ich] = Double_t(D_Sample[i].NN[IndexSample("data")]);
								scale_factor_2[iyear][ich] = Double_t(D_Sample[i].NN[IndexSample("data")]);
						}
						else if(!D_Sample[i].m_sample[j].CompareTo("bkg1")){
								if(0 == D_Sample[i].NN[IndexSample("bkg1")]){
										scale_factor_1[iyear][ich] -= 0;
										scale_factor_2[iyear][ich] += 0;
								}
								else{
										scale_factor_1[iyear][ich] -= (D_Sample[i].n_bkg);
										scale_factor_2[iyear][ich] += (D_Sample[i].n_bkg / Double_t(D_Sample[i].NN[IndexSample("bkg1")])) * (D_Sample[i].n_bkg / Double_t(D_Sample[i].NN[IndexSample("bkg1")])) * Double_t(D_Sample[i].NN[IndexSample("bkg1")]);
								}
						}
						else if(!D_Sample[i].m_sample[j].CompareTo("sideband")){
								if(0 == D_Sample[i].NN[IndexSample("sideband")]){
										scale_factor_1[iyear][ich] -= 0;
										scale_factor_2[iyear][ich] += 0;
								}
								else{
										scale_factor_1[iyear][ich]-= 0.25 * Double_t(D_Sample[i].NN[IndexSample("sideband")]);   //  		FIXME: It is just a mistake;  Zhipeng uses a tight region of mXi1
										scale_factor_2[iyear][ich]+= 0.25*0.25 * Double_t(D_Sample[i].NN[IndexSample("sideband")]) ;   //
								}
						}
						else if(!D_Sample[i].m_sample[j].CompareTo("etac")){
								if(0 == D_Sample[i].NN[IndexSample("etac")]){
										scale_factor_1[iyear][ich] -= 0;
										scale_factor_2[iyear][ich] += 0;
								}
								else{
										scale_factor_1[iyear][ich] -= D_Sample[i].n_etac;
										scale_factor_2[iyear][ich] += (D_Sample[i].n_etac / Double_t(D_Sample[i].NN[IndexSample("etac")])) * (D_Sample[i].n_etac / Double_t(D_Sample[i].NN[IndexSample("etac")])) * Double_t(D_Sample[i].NN[IndexSample("etac")]);
								}
						}
						else if(!D_Sample[i].m_sample[j].CompareTo("charge")){
								if(0 == D_Sample[i].NN[IndexSample("charge")]){
										scale_factor_1[iyear][ich] -= 0;
										scale_factor_2[iyear][ich] += 0;
								}
								else{
										scale_factor_1[iyear][ich] -= D_Sample[i].n_charge;
										scale_factor_2[iyear][ich] += (D_Sample[i].n_charge / Double_t(D_Sample[i].NN[IndexSample("charge")])) * (D_Sample[i].n_charge / Double_t(D_Sample[i].NN[IndexSample("charge")])) * Double_t(D_Sample[i].NN[IndexSample("charge")]);
								}
						}
				}
		}

		// TODO:with or without the global factor
		/*
		   for(int i = 0; i < 4; i++){
		   for(int j = 0; j < 2; j++){
		   scale_factor_1[i][j] = 1.0;
		   scale_factor_2[i][j] = 1.0;
		   }
		   }
		 */


		for(int i = 0; i < (N_Sample +1); i++){
				int iyear = IndexYear(D_Sample[i].m_year);
				int ich = IndexChannel(D_Sample[i].m_channel);
				for(int j = 0; j < D_Sample[i].m_sample.size(); j++){
						if(!D_Sample[i].m_sample[j].CompareTo("data")){
								l1 += ( scale_factor_1[iyear][ich] / scale_factor_2[iyear][ich]) * (-loglike[iyear][ich][j]) ;
						}
						else if(!D_Sample[i].m_sample[j].CompareTo("bkg1")){
								if(0 == D_Sample[i].NN[IndexSample("bkg1")]){
										l1 += 0.;
								}
								else{
										l1 += ( scale_factor_1[iyear][ich] / scale_factor_2[iyear][ich]) * D_Sample[i].n_bkg * (loglike[iyear][ich][j]/Double_t(D_Sample[i].NN[IndexSample("bkg1")]));
								}
						}
						else if(!D_Sample[i].m_sample[j].CompareTo("sideband")){
								if(0 == D_Sample[i].NN[IndexSample("sideband")]){
										l1 += 0.;
								}
								else{
										l1 += ( scale_factor_1[iyear][ich] / scale_factor_2[iyear][ich]) * 0.25 * ( loglike[iyear][ich][j] );		// FIXME  the scale of the sideband  Zhipeng uses a tight region of mXi1
								}
						}
						else if(!D_Sample[i].m_sample[j].CompareTo("etac")){
								if(0 == D_Sample[i].NN[IndexSample("etac")]){
										l1 += 0.;
								}
								else{
										l1 += ( scale_factor_1[iyear][ich] / scale_factor_2[iyear][ich]) * D_Sample[i].n_etac * ( loglike[iyear][ich][j] / Double_t(D_Sample[i].NN[IndexSample("etac")]));
								}
						}
						else if(!D_Sample[i].m_sample[j].CompareTo("charge")){
								if(0 == D_Sample[i].NN[IndexSample("charge")]){
										l1 += 0.;
								}
								else{
										l1 += ( scale_factor_1[iyear][ich] / scale_factor_2[iyear][ich]) * D_Sample[i].n_charge * ( loglike[iyear][ich][j] / Double_t(D_Sample[i].NN[IndexSample("charge")]));
								}
						}
				}
		}

		llk =  l1;		

		if(fit_step%100 == 0){
				cout << "likelihood : " << llk << "  ";
				for(int i =0; i < 10; i++){
						cout <<  pp[i] << "  ";
				}
				cout << endl;
		}

		fit_step++;
		return llk;

}



double rootfile::fcnmll(double *pp){

		double pp1[8], pp2[8];
		for(int i = 0; i < 6; i++){
				pp1[i] = pp[i];
				pp2[i] = pp[i];
		}
		pp1[6] = pp[6]; pp1[7] = pp[7];
		pp2[6] = pp[8]; pp2[7] = pp[9];

		AA_parameter aa_para;
		aa_para.alpha_jpsi = pp[0];
		aa_para.phi_jpsi = pp[1];
		aa_para.alpha_xi = pp[2];
		aa_para.phi_xi = pp[3];
		aa_para.alpha_xibar = pp[4];
		aa_para.phi_xibar = pp[5];
		aa_para.alpha1_lambda = pp[6];
		aa_para.alpha1_lambdabar = pp[7];
		aa_para.alpha2_lambda = pp[8];
		aa_para.alpha2_lambdabar = pp[9];
		hipError_t cudaStatus;
		clock_t start,end;
		double loglike[4][2][6];
		for(int i = 0; i < 4; i++){
				angdis[i][0]->SetParameter(pp1);
				angdis[i][1]->SetParameter(pp2);
		}


		double norm[4][2];
		for(int i = 0; i < (N_Sample +1); i++){
				int iyear = IndexYear(D_Sample[i].m_year);
				int ich = IndexChannel(D_Sample[i].m_channel);
				norm[iyear][ich] = 0;
				if(!m_norm.CompareTo("phsp")){
						norm[iyear][ich] = angdis[iyear][ich]->CalcToIntegral()/Double_t(D_Sample[i].NN[IndexSample("phsp")]);
				}
				else if(!m_norm.CompareTo("mdiy")){
						norm[iyear][ich] = angdis[iyear][ich]->CalcToIntegralmDIY()/Double_t(D_Sample[i].NN[IndexSample("phsp")]);
				}
		}



		start = clock();
		clock_t start1,end1;
		clock_t start2,end2;
	//	clock_t start3,end3;

		for(int i = 0; i < (N_Sample +1); i++){
				for(int j = 0; j < D_Sample[i].m_sample.size(); j++){
						int iyear = IndexYear(D_Sample[i].m_year);
						int ich = IndexChannel(D_Sample[i].m_channel);
						int isample = IndexSample(D_Sample[i].m_sample[j]);
						if(!D_Sample[i].m_sample[j].CompareTo("phsp")) continue;
						if(!D_Sample[i].m_sample[j].CompareTo("mdiy")) continue;
						start1 = clock();
						gpu_Amp <<< (D_Sample[i].NN[j] * MATRIX_SIZE + MATRIX_SIZE * THREADS_PER_BLOCK ) / (MATRIX_SIZE * THREADS_PER_BLOCK), MATRIX_SIZE * THREADS_PER_BLOCK >>> ( 
										*(gpu_angdata[iyear][ich][isample] + 0), 
										*(gpu_angdata[iyear][ich][isample] + 1), 
										*(gpu_angdata[iyear][ich][isample] + 2), 
										*(gpu_angdata[iyear][ich][isample] + 3), 
										*(gpu_angdata[iyear][ich][isample] + 4), 
										*(gpu_angdata[iyear][ich][isample] + 5), 
										*(gpu_angdata[iyear][ich][isample] + 6), 
										*(gpu_angdata[iyear][ich][isample] + 7), 
										*(gpu_angdata[iyear][ich][isample] + 8),
										gpu_amp[iyear][ich][isample],
										(D_Sample[i].NN[j] + THREADS_PER_BLOCK)*80, 
										aa_para, ich, gpu_Matrix[iyear][ich][isample], norm[iyear][ich]);
						end1 = clock();
						hipDeviceSynchronize(); // wait until prior kernel is finished
						cudaStatus = hipGetLastError();
						if (cudaStatus != hipSuccess){
								cerr << "failure to call cuda kernel 004!" << endl;
								exit(1);
						}
						start2 = clock();
						loglike[iyear][ich][isample] = gpu_float_sum_reduce(gpu_amp[iyear][ich][isample], D_Sample[i].NN[j]);
						end2 = clock();
	//					double time13 = ((double)(end1-start1))/CLOCKS_PER_SEC;
	//					cout << "GPU 13: running kernel " << time13 << " seconds" << endl;
	//					double time23 = ((double)(end2-start2))/CLOCKS_PER_SEC;
	//					cout << "GPU 23: running kernel " << time23 << " seconds" << endl;
/*
						int mat_size = (D_Sample[i].NN[j] + THREADS_PER_BLOCK) *sizeof(double);
						hipMemcpy( out_amp[iyear][ich][isample], gpu_amp[iyear][ich][isample], mat_size, hipMemcpyDeviceToHost );
						cudaStatus = hipGetLastError();
						if (cudaStatus != hipSuccess){
								cerr << "failure to call cuda kernel 002!" << endl;
								exit(1);
						}
						loglike[iyear][ich][isample] = 0;
						if(!D_Sample[i].m_sample[j].CompareTo("phsp")){
								for(int evt = 0; evt < D_Sample[i].NN[j]; evt++){
										if(*(out_amp[iyear][ich][isample] + evt) <= 0){ cout << "data : " << *(out_amp[iyear][ich][isample] + evt) << endl;  return 0; }
										loglike[iyear][ich][isample] += *(out_amp[iyear][ich][isample] + evt);
								}

						}
						else{
						start3 = clock();
								for(int evt = 0; evt < D_Sample[i].NN[j]; evt++){
										//	cout << "host C munu 0: " << 	*(host_eval + evt) << endl;
										if(*(out_amp[iyear][ich][isample] + evt) <= 0){ cout << "data : " << *(out_amp[iyear][ich][isample] + evt) << endl;  return 0; }
										loglike[iyear][ich][isample] += TMath::Log((*(out_amp[iyear][ich][isample] + evt))/norm[iyear][ich]);
								}

						end3 = clock();
						}
						double time13 = ((double)(end1-start1))/CLOCKS_PER_SEC;
						cout << "GPU 13: running kernel " << time13 << " seconds" << endl;
						double time23 = ((double)(end2-start2))/CLOCKS_PER_SEC;
						cout << "GPU 23: running kernel " << time23 << " seconds" << endl;
						double time33 = ((double)(end3-start3))/CLOCKS_PER_SEC;
						cout << "GPU 33: running kernel " << time33 << " seconds" << endl;
						*/
				}
		}
		end = clock();
		double time3 = ((double)(end-start))/CLOCKS_PER_SEC;
		cout << "GPU 3: running kernel " << time3 << " seconds" << endl;
		//	exit(1);
		double llk = 0;
		double l1 = 0;

		double scale_factor_1[4][2];
		double scale_factor_2[4][2];
		for(int i = 0; i < 4; i++){
				for(int j = 0; j < 2; j++){
						scale_factor_1[i][j] = 0;
						scale_factor_2[i][j] = 0;
				}
		}

		for(int i = 0; i < (N_Sample +1); i++){
				int iyear = IndexYear(D_Sample[i].m_year);
				int ich = IndexChannel(D_Sample[i].m_channel);
				for(int j = 0; j < D_Sample[i].m_sample.size(); j++){
						if(!D_Sample[i].m_sample[j].CompareTo("data")){
								scale_factor_1[iyear][ich] = Double_t(D_Sample[i].NN[IndexSample("data")]);
								scale_factor_2[iyear][ich] = Double_t(D_Sample[i].NN[IndexSample("data")]);
						}
						else if(!D_Sample[i].m_sample[j].CompareTo("bkg1")){
								if(0 == D_Sample[i].NN[IndexSample("bkg1")]){
										scale_factor_1[iyear][ich] -= 0;
										scale_factor_2[iyear][ich] += 0;
								}
								else{
										scale_factor_1[iyear][ich] -= (D_Sample[i].n_bkg);
										scale_factor_2[iyear][ich] += (D_Sample[i].n_bkg / Double_t(D_Sample[i].NN[IndexSample("bkg1")])) * (D_Sample[i].n_bkg / Double_t(D_Sample[i].NN[IndexSample("bkg1")])) * Double_t(D_Sample[i].NN[IndexSample("bkg1")]);
								}
						}
						else if(!D_Sample[i].m_sample[j].CompareTo("sideband")){
								if(0 == D_Sample[i].NN[IndexSample("sideband")]){
										scale_factor_1[iyear][ich] -= 0;
										scale_factor_2[iyear][ich] += 0;
								}
								else{
										scale_factor_1[iyear][ich]-= 0.25 * Double_t(D_Sample[i].NN[IndexSample("sideband")]);   //  		FIXME: It is just a mistake;  Zhipeng uses a tight region of mXi1
										scale_factor_2[iyear][ich]+= 0.25*0.25 * Double_t(D_Sample[i].NN[IndexSample("sideband")]) ;   //
								}
						}
						else if(!D_Sample[i].m_sample[j].CompareTo("etac")){
								if(0 == D_Sample[i].NN[IndexSample("etac")]){
										scale_factor_1[iyear][ich] -= 0;
										scale_factor_2[iyear][ich] += 0;
								}
								else{
										scale_factor_1[iyear][ich] -= D_Sample[i].n_etac;
										scale_factor_2[iyear][ich] += (D_Sample[i].n_etac / Double_t(D_Sample[i].NN[IndexSample("etac")])) * (D_Sample[i].n_etac / Double_t(D_Sample[i].NN[IndexSample("etac")])) * Double_t(D_Sample[i].NN[IndexSample("etac")]);
								}
						}
						else if(!D_Sample[i].m_sample[j].CompareTo("charge")){
								if(0 == D_Sample[i].NN[IndexSample("charge")]){
										scale_factor_1[iyear][ich] -= 0;
										scale_factor_2[iyear][ich] += 0;
								}
								else{
										scale_factor_1[iyear][ich] -= D_Sample[i].n_charge;
										scale_factor_2[iyear][ich] += (D_Sample[i].n_charge / Double_t(D_Sample[i].NN[IndexSample("charge")])) * (D_Sample[i].n_charge / Double_t(D_Sample[i].NN[IndexSample("charge")])) * Double_t(D_Sample[i].NN[IndexSample("charge")]);
								}
						}
				}
		}

		// TODO:with or without the global factor
		/*
		   for(int i = 0; i < 4; i++){
		   for(int j = 0; j < 2; j++){
		   scale_factor_1[i][j] = 1.0;
		   scale_factor_2[i][j] = 1.0;
		   }
		   }
		 */


		for(int i = 0; i < (N_Sample +1); i++){
				int iyear = IndexYear(D_Sample[i].m_year);
				int ich = IndexChannel(D_Sample[i].m_channel);
				for(int j = 0; j < D_Sample[i].m_sample.size(); j++){
						if(!D_Sample[i].m_sample[j].CompareTo("data")){
								l1 += ( scale_factor_1[iyear][ich] / scale_factor_2[iyear][ich]) * (-loglike[iyear][ich][j]) ;
						}
						else if(!D_Sample[i].m_sample[j].CompareTo("bkg1")){
								if(0 == D_Sample[i].NN[IndexSample("bkg1")]){
										l1 += 0.;
								}
								else{
										l1 += ( scale_factor_1[iyear][ich] / scale_factor_2[iyear][ich]) * D_Sample[i].n_bkg * (loglike[iyear][ich][j]/Double_t(D_Sample[i].NN[IndexSample("bkg1")]));
								}
						}
						else if(!D_Sample[i].m_sample[j].CompareTo("sideband")){
								if(0 == D_Sample[i].NN[IndexSample("sideband")]){
										l1 += 0.;
								}
								else{
										l1 += ( scale_factor_1[iyear][ich] / scale_factor_2[iyear][ich]) * 0.25 * ( loglike[iyear][ich][j] );		// FIXME  the scale of the sideband  Zhipeng uses a tight region of mXi1
								}
						}
						else if(!D_Sample[i].m_sample[j].CompareTo("etac")){
								if(0 == D_Sample[i].NN[IndexSample("etac")]){
										l1 += 0.;
								}
								else{
										l1 += ( scale_factor_1[iyear][ich] / scale_factor_2[iyear][ich]) * D_Sample[i].n_etac * ( loglike[iyear][ich][j] / Double_t(D_Sample[i].NN[IndexSample("etac")]));
								}
						}
						else if(!D_Sample[i].m_sample[j].CompareTo("charge")){
								if(0 == D_Sample[i].NN[IndexSample("charge")]){
										l1 += 0.;
								}
								else{
										l1 += ( scale_factor_1[iyear][ich] / scale_factor_2[iyear][ich]) * D_Sample[i].n_charge * ( loglike[iyear][ich][j] / Double_t(D_Sample[i].NN[IndexSample("charge")]));
								}
						}
				}
		}

		llk =  l1;		

		if(fit_step%100 == 0){
				cout << "likelihood : " << llk << "  ";
				for(int i =0; i < 10; i++){
						cout <<  pp[i] << "  ";
				}
				cout << endl;
		}

		fit_step++;
		return llk;
}


double rootfile::setBKGMassFit( double nn, TString year, TString channel, TString sample){
		double Njpis[4] = {224000000., 1087000000, 4387000000, 4387000000};
		double NMCetac[4] = {45000, 195000, 750000, 750000};
		double NMCcharge[4] = {3000000, 18000000, 55500000, 60000000};
		int iyear = IndexYear(year);
		int ich = IndexChannel(channel);
		if(!sample.CompareTo("etac")){
				return Njpis[iyear] * 0.017*0.0009*0.99524*0.99524*0.639*0.358* nn / NMCetac[iyear];
		}
		else if(!sample.CompareTo("charge")){
				return Njpis[iyear] * 0.0011*0.99524*0.99524*0.639*0.638* nn / NMCcharge[iyear];
		}
}



void rootfile::setBKG(){
		double Njpis[4] = {224000000., 1087000000, 4387000000, 4387000000};
		double NMCetac[4] = {45000, 195000, 750000, 750000};
		double NMCcharge[4] = {3000000, 18000000, 55500000, 60000000};
		for(int i = 0; i < (N_Sample +1); i++){
				int iyear = IndexYear(D_Sample[i].m_year);
				int ich = IndexChannel(D_Sample[i].m_channel);
				for(int j = 0; j < D_Sample[i].m_sample.size(); j++){
						if(!D_Sample[i].m_sample[j].CompareTo("etac")){
								D_Sample[i].n_etac = Njpis[iyear] * 0.017*0.0009*0.99524*0.99524*0.639*0.358* D_Sample[i].NN[j] / NMCetac[iyear];
								D_Sample[i].n_etacerr = D_Sample[i].n_etac * sqrt((0.4/1.7)*(0.4/1.7) 
												+ (2.6/9.0)*(2.6/9.0) + (0.5/63.9)*(0.5/63.9) + 
												(0.5/35.8)*(0.5/35.8));
						}
						else if(!D_Sample[i].m_sample[j].CompareTo("charge")){
								D_Sample[i].n_charge = Njpis[iyear] * 0.0011*0.99524*0.99524*0.639*0.638* D_Sample[i].NN[j] / NMCcharge[iyear];
								D_Sample[i].n_chargeerr = D_Sample[i].n_charge * sqrt(
												(0.8/9.7)*(0.8/9.7) + (0.5/63.9)*(0.5/63.9) + 
												(0.5/63.9)*(0.5/63.9));

								cout << "background charge : " << D_Sample[i].n_charge << "	" << D_Sample[i].n_chargeerr << endl;
						}
				}
		}
}

Int_t rootfile::CorrFactor(TString year, TString channel, TString sample){
		Double_t factor = 0;
		int iyear = IndexYear(year);
		int ich = IndexChannel(channel);
		if(!sample.CompareTo("phsp")){
				int i_pion1_1 = floor((m_pion1_1_cos + 1) / 0.2);
				int j_pion1_1 = floor(m_pion1_1_pt / 0.05);
				int i_pion2_1 = floor((m_pion2_1_cos + 1) / 0.2);
				int j_pion2_1 = floor(m_pion2_1_pt / 0.05);
				int i_pion0 = floor((m_pion0_cos + 1) / 0.2);
				int j_pion0 = floor(m_pion0_rho / 0.05);
				if(j_pion0 <= 2) j_pion0 = 2;
				if(j_pion0 >= 4) j_pion0 = 4;
				if(j_pion2_1 >=5) j_pion2_1 =5;
				if(j_pion1_1 >=5) j_pion1_1 =5;
				int ich2 = ich ? 0 : 1;
				factor = hpi0corr[iyear][ich]->GetBinContent(i_pion0+1, j_pion0+1) * hpicorr[iyear][ich2]->GetBinContent(i_pion1_1+1, j_pion1_1+1) * hpicorr[iyear][ich]->GetBinContent(i_pion2_1+1, j_pion2_1+1);

		}
		else if(!sample.CompareTo("bkg1")){
				int i = floor((m_lmd_cos + 1) / 0.2);
				int j = floor((m_lmd_p - 0.5)/ 0.05);

				int i_pion1_1 = floor((m_pion1_1_cos + 1) / 0.2);
				int j_pion1_1 = floor(m_pion1_1_pt / 0.05);
				int i_pion2_1 = floor((m_pion2_1_cos + 1) / 0.2);
				int j_pion2_1 = floor(m_pion2_1_pt / 0.05);
				if(j_pion2_1 >=5) j_pion2_1 =5;
				if(j_pion1_1 >=5) j_pion1_1 =5;

				int ich2 = ich ? 0 : 1;

				if(j >=0 && j < 7){
						factor = (hcorrbkg[iyear][ich]->GetBinContent(i+1, j+1)) * hpicorr[iyear][ich2]->GetBinContent(i_pion1_1+1, j_pion1_1+1) * hpicorr[iyear][ich]->GetBinContent(i_pion2_1+1, j_pion2_1+1);
				}
				else{
						factor  = 1.0;
				}
		}

		double rdm = r1->Rndm();
		if(factor < 1.0 && rdm > factor){
				return 0;
		}
		else if(factor < 1.0 && rdm < factor){
				return 1;
		}
		else if(factor > 1.0 && rdm < factor - 1.0 && factor < 2.0){
				return 2;
		}
		else if(factor > 1.0 && rdm > factor - 1.0 && factor < 2.0){
				return 1;
		}
		else if(factor > 2.0 && rdm < factor - 2.0 && factor < 3.0){
				return 3;
		}
		else if(factor > 2.0 && rdm > factor - 2.0 && factor < 3.0){
				return 2;
		}
		else if(factor > 3.0){  	// almost no event has a factor larger than 3.
				return 3;
		}
		else 
				return 1.0;
}
Double_t rootfile::calculate_int(Double_t par0, Double_t par1, Double_t intlow, Double_t intup){
		double cosup = intup;
		double coslow = intlow;
		double intfuncup = par0*(cosup+par1*pow(cosup,3)/3);
		double intfunclow  = par0*(coslow+par1*pow(coslow,3)/3);
		double integral = intfuncup - intfunclow;
		return integral;
}


void rootfile::readDataIO(int index, int jndex){
		TString path = "/data/liul/workarea/XIXI/fit/boost";
		TString year = D_Sample[index].m_year;
		TString channel = D_Sample[index].m_channel;
		TString version = D_Sample[index].m_version;
		TString sample = D_Sample[index].m_sample[jndex];
		TString insample = sample;
		if(!sample.CompareTo("data")){
				insample = "mdiy";
		}
		if(!sample.CompareTo("phsp")){
				if(!m_norm.CompareTo("mdiy")){
						insample = "mdiy";
				}
				else if(!m_norm.CompareTo("phsp")){
						insample = "phsp";
				}
		}
		if(!sample.CompareTo("bkg1")){
				insample = "mdiy";
		}
		if(!sample.CompareTo("sideband")){
				insample = "data";
		}
		TString infile  = path + "/" + year + "/" + channel + "/" + version + "/" + insample + "/boost.root";
		cout << infile << " ==> " << sample << endl;
		//#define read_data
#ifndef read_data
		Int_t NN1;
		std::vector<double> gD1Xithe;
		std::vector<double> gD1Lthe;
		std::vector<double> gD1Lphi;
		std::vector<double> gD1Lbthe;
		std::vector<double> gD1Lbphi;
		std::vector<double> gD1pthe;
		std::vector<double> gD1pphi;
		std::vector<double> gD1apthe;
		std::vector<double> gD1apphi;

		gD1Xithe.clear();
		gD1Lthe.clear();
		gD1Lphi.clear();
		gD1Lbthe.clear();
		gD1Lbphi.clear();
		gD1pthe.clear();
		gD1pphi.clear();
		gD1apthe.clear();
		gD1apphi.clear();



		TFile *f1 = new TFile(infile, "read");
		TTree *t1 = (TTree*)f1->Get("xixi");
		t1->SetBranchAddress("the", &the);
		t1->SetBranchAddress("Lthe", &Lthe);
		t1->SetBranchAddress("Lphi", &Lphi);
		t1->SetBranchAddress("Lbthe", &Lbthe);
		t1->SetBranchAddress("Lbphi", &Lbphi);
		t1->SetBranchAddress("pthe", &pthe);
		t1->SetBranchAddress("pphi", &pphi);
		t1->SetBranchAddress("apthe", &apthe);
		t1->SetBranchAddress("apphi", &apphi);
		t1->SetBranchAddress("runNo", &m_runNo);
		t1->SetBranchAddress("LmdDL", &m_LmdDL);
		t1->SetBranchAddress("XiDL", &m_XiDL);
		t1->SetBranchAddress("XiCosTheta", &m_XiCosTheta);
		t1->SetBranchAddress("mXi2", &m_mXi2);
		t1->SetBranchAddress("mXi1", &m_mXi1);
		t1->SetBranchAddress("mLmd1", &m_mLmd1);
		t1->SetBranchAddress("mn", &m_mn);
		t1->SetBranchAddress("chi2kmf", &m_chi2kmf);
		t1->SetBranchAddress("chi2Xi", &m_chi2Xi);
		t1->SetBranchAddress("chi2Lmd", &m_chi2Lmd);
		t1->SetBranchAddress("angle_gam1", &m_angle_gam1);
		t1->SetBranchAddress("angle_gam2", &m_angle_gam2);
		if(sample.CompareTo("etac")){
				t1->SetBranchAddress("lmd_p", &m_lmd_p);
				t1->SetBranchAddress("lmd_cos", &m_lmd_cos);
		}

		int nn = 0;
		int NEvt = t1->GetEntries();
		int low = 0;
		int high = NEvt;
		int LOW = -1;
		int HIGH = -1;
		r1->SetSeed(3251);
		if(!sample.CompareTo("data")){
				low = i_trial * NEvt/30;
				high = (i_trial + 1) * NEvt/30;
		}
		else if(!sample.CompareTo("phsp")){
				LOW =  i_trial * NEvt/30;
				HIGH = (i_trial + 1) * NEvt/30;
		}
		else if(!sample.CompareTo("bkg1")){
				LOW =  i_trial * NEvt/30;
				HIGH = (i_trial + 1) * NEvt/30;
		}

		for(int i = low; i <  high; i++){
				t1->GetEntry(i);
				if(i > LOW && i < HIGH) continue;
				if(!Selection(year, channel, sample, index)) continue;
				if(!sample.CompareTo("phsp")){
						gD1Xithe.push_back(the);
						gD1Lthe.push_back(Lthe);
						gD1Lphi.push_back(Lphi);
						gD1Lbthe.push_back(Lbthe);
						gD1Lbphi.push_back(Lbphi);
						gD1pthe.push_back(pthe);
						gD1pphi.push_back(pphi);
						gD1apthe.push_back(apthe);
						gD1apphi.push_back(apphi);
						nn++;
						if(!m_norm.CompareTo("phsp")){
								angdis[IndexYear(year)][IndexChannel(channel)]->AddToIntegral(the, Lthe, Lphi, Lbthe, Lbphi, pthe, pphi, apthe, apphi);
						}
						else if(!m_norm.CompareTo("mdiy")){
								angdis[IndexYear(year)][IndexChannel(channel)]->AddToIntegralmDIY(the, Lthe, Lphi, Lbthe, Lbphi, pthe, pphi, apthe, apphi);
						}

				}
				else{
						gD1Xithe.push_back(the);
						gD1Lthe.push_back(Lthe);
						gD1Lphi.push_back(Lphi);
						gD1Lbthe.push_back(Lbthe);
						gD1Lbphi.push_back(Lbphi);
						gD1pthe.push_back(pthe);
						gD1pphi.push_back(pphi);
						gD1apthe.push_back(apthe);
						gD1apphi.push_back(apphi);
						nn++;
				}
		}
		D_Sample[index].NN.push_back(nn);

		int iyear = IndexYear(year);
		int ich = IndexChannel(channel);
		int isample = IndexSample(sample);
		cout << "Hello " << iyear << ich << isample << endl;

		for(int i = 0; i < nn; i++){
				*(*(angdata[iyear][ich][isample]+0)+i) = gD1Xithe[i];
				*(*(angdata[iyear][ich][isample]+1)+i) = gD1Lthe[i];
				*(*(angdata[iyear][ich][isample]+2)+i) = gD1Lphi[i];
				*(*(angdata[iyear][ich][isample]+3)+i) = gD1Lbthe[i];
				*(*(angdata[iyear][ich][isample]+4)+i) = gD1Lbphi[i];
				*(*(angdata[iyear][ich][isample]+5)+i) = gD1pthe[i];
				*(*(angdata[iyear][ich][isample]+6)+i) = gD1pphi[i];
				*(*(angdata[iyear][ich][isample]+7)+i) = gD1apthe[i];
				*(*(angdata[iyear][ich][isample]+8)+i) = gD1apphi[i];
		}


		cout << nn << endl;
		f1->Close();
#endif
}

void rootfile::Print(){

		cout << "iyear	channel	signal		bkg		sideband	etac	charge" << endl;
		for(int i = 0; i < (N_Sample +1); i++){
				int iyear = IndexYear(D_Sample[i].m_year);
				int ich = IndexChannel(D_Sample[i].m_channel);
				cout << std::left << setw(4) << D_Sample[i].m_year << "	" << D_Sample[i].m_channel << "	";
				for(int j = 0; j < D_Sample[i].m_sample.size(); j++){
						if(!D_Sample[i].m_sample[j].CompareTo("data")){
								cout << D_Sample[i].n_signal << "±" << D_Sample[i].n_signalerr << "	";
						}
						else if(!D_Sample[i].m_sample[j].CompareTo("bkg1")){
								cout << D_Sample[i].n_bkg << "±" << D_Sample[i].n_bkgerr << "	";
						}
						else if(!D_Sample[i].m_sample[j].CompareTo("sideband")){
								cout << D_Sample[i].NN[IndexSample("sideband")]/4 << "±" << 
										sqrt(D_Sample[i].NN[IndexSample("sideband")]/4)<< "	";
						}
						else if(!D_Sample[i].m_sample[j].CompareTo("etac")){
								cout << D_Sample[i].n_etac << "±" << D_Sample[i].n_etacerr << "	";
						}
						else if(!D_Sample[i].m_sample[j].CompareTo("charge")){
								cout << D_Sample[i].n_charge << "±" << D_Sample[i].n_chargeerr << "	";
						}
				}
				cout << endl;
				cout << std::left << setw(4) << D_Sample[i].m_year << "	" << D_Sample[i].m_channel << "	";
				for(int j = 0; j < D_Sample[i].m_sample.size(); j++){
						if(!D_Sample[i].m_sample[j].CompareTo("data")){
								cout << D_Sample[i].NN[IndexSample("data")] << "	";
						}
						else if(!D_Sample[i].m_sample[j].CompareTo("bkg1")){
								cout << D_Sample[i].NN[IndexSample("bkg1")] << "	";
						}
						else if(!D_Sample[i].m_sample[j].CompareTo("sideband")){
								cout << D_Sample[i].NN[IndexSample("sideband")] << "	";
						}
						else if(!D_Sample[i].m_sample[j].CompareTo("etac")){
								cout << D_Sample[i].NN[IndexSample("etac")] << "	";
						}
						else if(!D_Sample[i].m_sample[j].CompareTo("charge")){
								cout << D_Sample[i].NN[IndexSample("charge")] << "	";
						}
				}
				cout << endl;
		}
}

void rootfile::SetBKGSysTest(vector<TString> vstr){
		for(int i = 0; i < vstr.size(); i++){
				if(!vstr[i].CompareTo("bkg1p1")){ // plus 1 sigma
						for(int j = 0; j < (N_Sample +1); j++){
								D_Sample[j].n_bkg +=  D_Sample[j].n_bkgerr;
						}
				}
				else if(!vstr[i].CompareTo("bkg1m1")){ // minus 1 sigma
						for(int j = 0; j < (N_Sample +1); j++){
								D_Sample[j].n_bkg -=  D_Sample[j].n_bkgerr;
						}
				}
				else if(!vstr[i].CompareTo("etac1p1")){ // plus 1 sigma
						for(int j = 0; j < (N_Sample +1); j++){
								D_Sample[j].n_etac +=  D_Sample[j].n_etacerr;
						}
				}
				else if(!vstr[i].CompareTo("etac1m1")){ // minus 1 sigma
						for(int j = 0; j < (N_Sample +1); j++){
								D_Sample[j].n_etac -=  D_Sample[j].n_etacerr;
						}
				}
				else if(!vstr[i].CompareTo("charge1p1")){ // plus 1 sigma
						for(int j = 0; j < (N_Sample +1); j++){
								D_Sample[j].n_charge +=  D_Sample[j].n_chargeerr;
								cout << "Backgroudn charge : " << D_Sample[j].n_charge << endl;
						}
				}
				else if(!vstr[i].CompareTo("charge1m1")){ // minus 1 sigma
						for(int j = 0; j < (N_Sample +1); j++){
								D_Sample[j].n_charge -=  D_Sample[j].n_chargeerr;
						}
				}
		}
}
