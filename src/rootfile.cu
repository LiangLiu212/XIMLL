#include "hip/hip_runtime.h"
#include "rootfile.cuh"

void rootfile::InitialMemory(){
		nsample = size()/Nyear();
		for(int i = 0; i < Nyear(); i++){
				for(int j = 0; j < nsample; j ++){
						angdata[i][j] = new double * [9];
						for(int l = 0; l < 9; l++){
								*(angdata[i][j] + l) =  new double [NUM];
						}
				}
		}
		for(int i = 0; i < Nyear(); i++)
				for(int j = 0; j < 2; j ++){
								angdis[i][j] = new AngDisXiXi();
								angdis[i][j]->InitialInt();
				}
}


//=====================================================================
void rootfile::ReadData(const int index, const int MM)
{
		//	int years = 2;
		for(int i = 0; i < m_nyear; i++){ 		// read data
				for(int j = 0; j < nsample; j++){
						int l = -1;
						int n = i*nsample + j;
						if(!type(n).CompareTo("xixipm")){
								l = 0;
						}
						else if(!type(n).CompareTo("xixipp")){
								l = 1;
						}
						cout << i << "   "  << l  << endl;
						NN[i][j] = readData(n, angdis[i][l], angdata[i][j], index, MM);
	//					if(sample(n).CompareTo("phsp")){
	//							idx.push_back(n);
	//					}
				}
		}
		for(int i = 0; i < m_nyear; i++){
				for(int j = 0; j < nsample; j ++){
						cout << "N[" << i << "][" << j << "] : " << NN[i][j] << endl;
				}
		}

		double **temp_angdata[m_nyear][20]; // define a temporary array 
		for(int i = 0; i < m_nyear; i++){
				for(int j = 0; j < nsample; j ++){
						temp_angdata[i][j] = new double * [9];
						for(int l = 0; l < 9; l++){
								*(temp_angdata[i][j] + l) =  new double [NN[i][j] + THREADS_PER_BLOCK];
								for(int k = 0; k< NN[i][j]; k ++){
										*(*(temp_angdata[i][j] + l) + k) = *(*(angdata[i][j] + l) + k);
								}
						}
				}
		}

		for(int i = 0; i < m_nyear; i++){    // copy data from cpu to gpu
				for(int j = 0; j < nsample; j ++){
						int size1 = (NN[i][j] + THREADS_PER_BLOCK) *sizeof(double);
						gpu_angdata[i][j] = new double * [9];
						for(int l = 0; l < 9; l++){
								hipMalloc( (void **) &(*(gpu_angdata[i][j] + l)), size1 );
								hipMemcpy( *(gpu_angdata[i][j] + l), *(temp_angdata[i][j] + l), size1, hipMemcpyHostToDevice );
								delete [] *(temp_angdata[i][j] + l);
						}
						hipMalloc( (void **) &gpu_Matrix[i][j], size1 * MATRIX_SIZE  );
						hipMalloc( (void **) &gpu_amp[i][j], size1 * MATRIX_SIZE  );
						out_amp[i][j] = new double [NN[i][j] + THREADS_PER_BLOCK];
				}
		}
}

void rootfile::FreeMemory(){
		for(int i = 0; i < m_nyear; i++){
				for(int j = 0; j < nsample; j ++){
						for(int l = 0; l < 9; l++){
								hipFree((*(gpu_angdata[i][j] + l)));
						}
						hipFree(gpu_Matrix[i][j]);
						hipFree(gpu_amp[i][j]);
						delete [] out_amp[i][j];
				}
		}
}


int rootfile::readData(const int n, AngDisXiXi *ang, double **para, const int  index, const int MM){
		cout << n << "   " << m_file[n] << " " << m_sample[n] << " " <<  m_year[n]<<  "  " << type(n) << endl;


		bool data = false;
		bool mdiy = false;
		bool phsp = false;
		bool bkg = false;
		bool inc = false;
		if(!m_sample[n].CompareTo("data")) data = true;
		if(!m_sample[n].CompareTo("mdiy")) mdiy = true;
		if(!m_sample[n].CompareTo("phsp")) phsp = true;
		if(!m_sample[n].CompareTo("bkg1")) bkg = true;
		if(!m_sample[n].CompareTo("inclusive")) inc = true;

		int runNo_low = 0;
		int runNo_high = 0;
		int iyear = -1;
		int itype = -1;
		if(!m_type[n].CompareTo("xixipm")){
				itype = 0;
		}
		else if(!m_type[n].CompareTo("xixipp")){
				itype = 1;
		}

		if(!m_year[n].CompareTo("2009")){
				iyear = 0;
				runNo_low = 9800;
				runNo_high =  11000;
		}
		else if(!m_year[n].CompareTo("2012")){
				iyear = 1;
				runNo_low = 27100;
				runNo_high = 28400;
		}
		else if(!m_year[n].CompareTo("2018")){
				iyear = 2;
				runNo_low = 52840;
				runNo_high = 56646;
		}
		else if(!m_year[n].CompareTo("2019")){
				iyear = 3;
				runNo_low = 56778 ;
				runNo_high = 59115;
		}
		Int_t NN1;
		std::vector<double> gD1Xithe;
		std::vector<double> gD1Lthe;
		std::vector<double> gD1Lphi;
		std::vector<double> gD1Lbthe;
		std::vector<double> gD1Lbphi;
		std::vector<double> gD1pthe;
		std::vector<double> gD1pphi;
		std::vector<double> gD1apthe;
		std::vector<double> gD1apphi;

		gD1Xithe.clear();
		gD1Lthe.clear();
		gD1Lphi.clear();
		gD1Lbthe.clear();
		gD1Lbphi.clear();
		gD1pthe.clear();
		gD1pphi.clear();
		gD1apthe.clear();
		gD1apphi.clear();

		Double_t the, Lthe, Lphi, Lbthe, Lbphi, pthe, pphi, apthe, apphi;
		Double_t m_LmdDL, m_XiDL, m_XiCosTheta, m_mXi2, m_mXi1, m_mLmd1, m_mn;
		Double_t m_chi2kmf, m_chi2Xi, m_chi2Lmd, m_angle_gam1, m_angle_gam2;

		int runNo;

		TFile *f1 = new TFile(m_file[n], "read");
		TTree *t1 = (TTree*)f1->Get("xixi");
		t1->SetBranchAddress("the", &the);
		t1->SetBranchAddress("Lthe", &Lthe);
		t1->SetBranchAddress("Lphi", &Lphi);
		t1->SetBranchAddress("Lbthe", &Lbthe);
		t1->SetBranchAddress("Lbphi", &Lbphi);
		t1->SetBranchAddress("pthe", &pthe);
		t1->SetBranchAddress("pphi", &pphi);
		t1->SetBranchAddress("apthe", &apthe);
		t1->SetBranchAddress("apphi", &apphi);
		t1->SetBranchAddress("runNo", &runNo);

		t1->SetBranchAddress("LmdDL", &m_LmdDL);
		t1->SetBranchAddress("XiDL", &m_XiDL);
		t1->SetBranchAddress("XiCosTheta", &m_XiCosTheta);
		t1->SetBranchAddress("mXi2", &m_mXi2);
		t1->SetBranchAddress("mXi1", &m_mXi1);
		t1->SetBranchAddress("mLmd1", &m_mLmd1);
		t1->SetBranchAddress("mn", &m_mn);
		t1->SetBranchAddress("chi2kmf", &m_chi2kmf);
		t1->SetBranchAddress("chi2Xi", &m_chi2Xi);
		t1->SetBranchAddress("chi2Lmd", &m_chi2Lmd);
		t1->SetBranchAddress("angle_gam1", &m_angle_gam1);
		t1->SetBranchAddress("angle_gam2", &m_angle_gam2);
		int nn = 0;
		int NEvt = t1->GetEntries();
		int low = 0;
		int high = NEvt;
		//		NEvt = 10000;

		int count = 0;
		for(int i = low; i <  high; i++){
				t1->GetEntry(i);
				if(abs(runNo) < runNo_low || abs(runNo) > runNo_high) continue;
				if(m_LmdDL < cut_LmdDL) continue;
				if(m_XiDL < cut_XiDL) continue;
				if(fabs(m_XiCosTheta) >  cut_XiCosTheta) continue;
				if(fabs(m_mXi2 - 1.32171) > cut_mXi) continue;
				if(fabs(m_mXi1 - 1.32171) > cut_mXi) continue;
				if(fabs(m_mLmd1 - 1.1157) > cut_mLmd1) continue;
				if(m_chi2kmf > cut_chi2kmf) continue;
				if(m_chi2Xi > cut_chi2Xi) continue;
				if(m_chi2Lmd > cut_chi2Lmd) continue;


				if(data){
						ang->setDataMass(count, m_mn);
						count++;
						if(m_mn < cut_mn1 || m_mn > cut_mn2) continue;
				}
				if(mdiy){
						if(m_angle_gam1 > 0.3) continue;
						if(m_angle_gam2 > 0.3) continue;
						ang->setMCMass(count, m_mn);
						count++;
						if(m_mn < cut_mn1 || m_mn > cut_mn2) continue;
				}
				if(phsp){
						if(m_angle_gam1 > 0.3) continue;
						if(m_angle_gam2 > 0.3) continue;
						if(m_mn < cut_mn1 || m_mn > cut_mn2) continue;
				}
				if(bkg){
						if(m_mn < cut_mn1 || m_mn > cut_mn2) continue;
				}
				if(inc){
						if(m_mn < cut_mn1 || m_mn > cut_mn2) continue;
				}

				gD1Xithe.push_back(the);
				gD1Lthe.push_back(Lthe);
				gD1Lphi.push_back(Lphi);
				gD1Lbthe.push_back(Lbthe);
				gD1Lbphi.push_back(Lbphi);
				gD1pthe.push_back(pthe);
				gD1pphi.push_back(pphi);
				gD1apthe.push_back(apthe);
				gD1apphi.push_back(apphi);
				nn++;
				if(phsp){
						ang->AddToIntegral(the, Lthe, Lphi, Lbthe, Lbphi, pthe, pphi, apthe, apphi);
				}
		}
		//	cout << infile << ", " << type << ", " << flag << ", number : " <<   nn << endl;

		NN1 = nn;

		for(int i = 0; i < NN1; i++){
				*(*(para+0)+i) = gD1Xithe[i];
				*(*(para+1)+i) = gD1Lthe[i];
				*(*(para+2)+i) = gD1Lphi[i];
				*(*(para+3)+i) = gD1Lbthe[i];
				*(*(para+4)+i) = gD1Lbphi[i];
				*(*(para+5)+i) = gD1pthe[i];
				*(*(para+6)+i) = gD1pphi[i];
				*(*(para+7)+i) = gD1apthe[i];
				*(*(para+8)+i) = gD1apphi[i];
		}
		f1->Close();
		cout << m_file[n] << endl;
		cout << "Finish" << endl;
		return NN1;
}

void rootfile::MassFit(){
		int iyear =-1;
		int itype = -1;
		cout << "Mass Fit" << endl;
		for(int i = 0; i < Nyear(); i++){
				for(int j =0; j < 2; j++){
				NBKG[i][j] = 0;
				NBKG[i][j]  = massFit(i, j);
				}
		}


}

double rootfile::massFit(const int iyear, const int itype){
		using namespace RooFit;
		RooRealVar mn("mn", "M(n#pi^{-}) (GeV/#font[52]{c}^{2})",0.90,0.98);
		RooDataSet signal("signal", "signal", mn);
		cout << angdis[iyear][itype]->getNmc() << endl;
		for(int i = 0; i < angdis[iyear][itype]->getNmc(); i++){
				if(i == 10000) break;
				mn = angdis[iyear][itype]->MCMass(i);
				signal.add(mn);
		}
		RooKeysPdf keysshape("keysshape", "keysshape", mn, signal, RooKeysPdf::MirrorBoth, 2);

		RooDataSet data("data", "data", mn);

		cout << angdis[iyear][itype]->getNdata() << endl;
		double dataNevt = angdis[iyear][itype]->getNdata();
		for(int i = 0; i < angdis[iyear][itype]->getNdata(); i++){
				mn = angdis[iyear][itype]->DataMass(i);
				data.add(mn);
		}


		RooRealVar mean2("mean2", "mean2", 0.005, -0.05, 0.05);	
		RooRealVar sigma2("sigma2", "sigma2",0.005, 0, 0.05);
		RooGaussian ga1("ga1", "ga1", mn, mean2, sigma2);
		mn.setBins(10000, "cache");
		RooFFTConvPdf shape("shape", "shape", mn, keysshape, ga1);


		RooRealVar m0("m0", "m0", 0.98, 0.96, 0.999);
		RooRealVar k("k", "k", -10, -50, -1.0);
		RooArgusBG argus("argus", "argus", mn, m0, k);

		RooRealVar fra1("fra1", "fra1", dataNevt*100.85, 0, 20000000);
		RooRealVar fra2("fra2", "fra2", dataNevt*10.4, 0, 20000000);
		RooAddPdf sum("sum", "sig+bak", RooArgList(shape, argus), RooArgList(fra1, fra2));

		RooFitResult *res = sum.fitTo(data, "r", Extended(), Save(1));
		mn.setRange("bkg1", 0.9, 0.98);
		mn.setRange("bkg2", cut_mn1, cut_mn2);
		RooAbsReal *intpolyX10 = argus.createIntegral(mn,  NormSet(mn), Range("bkg1"));
		RooAbsReal *intpolyX11 = argus.createIntegral(mn,  NormSet(mn), Range("bkg2"));
		double back10 = intpolyX10->getVal();
		double back11 = intpolyX11->getVal();
		cout << back10 << endl;
		cout << back11 << endl;
		cout << fra2.getVal() * back11 << endl;
		return fra2.getVal() * back11;


}

double rootfile::fcnmll(double *pp){
		double pp1[8], pp2[8];
		for(int i = 0; i < 6; i++){
				pp1[i] = pp[i];
				pp2[i] = pp[i];
		}
		pp1[6] = pp[6]; pp1[7] = pp[7];
		pp2[6] = pp[8]; pp2[7] = pp[9];

		AA_parameter aa_para;
		aa_para.alpha_jpsi = pp[0];
		aa_para.phi_jpsi = pp[1];
		aa_para.alpha_xi = pp[2];
		aa_para.phi_xi = pp[3];
		aa_para.alpha_xibar = pp[4];
		aa_para.phi_xibar = pp[5];
		aa_para.alpha1_lambda = pp[6];
		aa_para.alpha1_lambdabar = pp[7];
		aa_para.alpha2_lambda = pp[8];
		aa_para.alpha2_lambdabar = pp[9];


		hipError_t cudaStatus;
	//	clock_t start,end;
		double loglike[4][12];
		int years = Nyear();
		for(int i = 0; i < years; i ++){
				angdis[i][0]->SetParameter(pp1);
				angdis[i][1]->SetParameter(pp2);
				for (int j = 0; j < (nsample); j++){
						int flag = j / ((nsample) / 2);
						int n = i*nsample + j;
						if(!sample(n).CompareTo("phsp")) continue;
						if(!sample(n).CompareTo("mdiy")) continue;
					//	start = clock();
						gpu_Amp <<< (NN[i][j] * MATRIX_SIZE + MATRIX_SIZE * THREADS_PER_BLOCK ) / (MATRIX_SIZE * THREADS_PER_BLOCK), MATRIX_SIZE * THREADS_PER_BLOCK >>> ( 
										*(gpu_angdata[i][j] + 0), 
										*(gpu_angdata[i][j] + 1), 
										*(gpu_angdata[i][j] + 2), 
										*(gpu_angdata[i][j] + 3), 
										*(gpu_angdata[i][j] + 4), 
										*(gpu_angdata[i][j] + 5), 
										*(gpu_angdata[i][j] + 6), 
										*(gpu_angdata[i][j] + 7), 
										*(gpu_angdata[i][j] + 8),
										gpu_amp[i][j],
										(NN[i][j] + THREADS_PER_BLOCK)*80, 
										aa_para, flag, gpu_Matrix[i][j]);
						hipDeviceSynchronize(); // wait until prior kernel is finished
					//	end = clock();
					//	double time3 = ((double)(end-start))/CLOCKS_PER_SEC;
					//	cout << "GPU 3: running kernel " << time3 << " seconds" << endl;
						cudaStatus = hipGetLastError();
						if (cudaStatus != hipSuccess){
								cerr << "failure to call cuda kernel 004!" << endl;
								exit(1);
						}

						int mat_size = (NN[i][j] + THREADS_PER_BLOCK) *sizeof(double);
						hipMemcpy( out_amp[i][j], gpu_amp[i][j], mat_size, hipMemcpyDeviceToHost );
						cudaStatus = hipGetLastError();
						if (cudaStatus != hipSuccess){
								cerr << "failure to call cuda kernel 002!" << endl;
								exit(1);
						}
						loglike[i][j] = 0;
						if(!sample(n).CompareTo("phsp")){
								for(int evt = 0; evt < NN[i][j]; evt++){
										if(*(out_amp[i][j] + evt) <= 0){ cout << "data : " << *(out_amp[i][j] + evt) << endl;  return 0; }
										loglike[i][j] += *(out_amp[i][j] + evt);
								}

						}
						else{
								for(int evt = 0; evt < NN[i][j]; evt++){
										//	cout << "host C munu 0: " << 	*(host_eval + evt) << endl;
										if(*(out_amp[i][j] + evt) <= 0){ cout << "data : " << *(out_amp[i][j] + evt) << endl;  return 0; }
										loglike[i][j] += TMath::Log(*(out_amp[i][j] + evt));
								}
						}
				}
		}
		//	exit(1);
		double norm[4][2];
		for (int i = 0; i < years; i++){
				for (int j = 0; j < 2; j++){
						norm[i][j] = 0;
					//	angdis[i][j]->PrintInt();
						norm[i][j] = angdis[i][j]->CalcToIntegral();
				}
		}



		double llk = 0;
		double l1 = 0;
		double l2 = 0;

		for(int i = 0; i < years; i++){
	//			l1 = - loglike[i][0] + NBKG[i][0]*loglike[i][3]/Double_t(NN[i][3]) + (Double_t(NN[i][0]) - NBKG[i][0])*TMath::Log(loglike[i][2]/Double_t(NN[i][2]));
	//			l2 = - loglike[i][4] + NBKG[i][1]*loglike[i][7]/Double_t(NN[i][7]) + (Double_t(NN[i][4]) - NBKG[i][1])*TMath::Log(loglike[i][6]/Double_t(NN[i][6]));
				l1 = - loglike[i][0] + NBKG[i][0]*loglike[i][3]/Double_t(NN[i][3]) + (Double_t(NN[i][0]) - NBKG[i][0])*TMath::Log(norm[i][0]/Double_t(NN[i][2]));
				l2 = - loglike[i][4] + NBKG[i][1]*loglike[i][7]/Double_t(NN[i][7]) + (Double_t(NN[i][4]) - NBKG[i][1])*TMath::Log(norm[i][1]/Double_t(NN[i][6]));
				llk += (l1 + l2);
		}
/*
		if(fit_step%100 == 0){
				std::cout << "Loglike: " << llk << std::endl; 
				for( int i = 0; i<10 ; i++ ) cout<<pp[i]<<" ";
				cout << endl;
		}
		*/
		fit_step++;
		return llk;

}


//=====================================================================
void rootfile::IOReadData(const int index, const int MM)
{
		//	int years = 2;
		for(int i = 0; i < m_nyear; i++){ 		// read data
				for(int j = 0; j < nsample; j++){
						int l = -1;
						int n = i*nsample + j;
						if(!type(n).CompareTo("xixipm")){
								l = 0;
						}
						else if(!type(n).CompareTo("xixipp")){
								l = 1;
						}
						cout << i << "   "  << l  << endl;
						NN[i][j] = IOreadData(n, angdis[i][l], angdata[i][j], index, MM);
	//					if(sample(n).CompareTo("phsp")){
	//							idx.push_back(n);
	//					}
				}
		}
		for(int i = 0; i < m_nyear; i++){
				for(int j = 0; j < nsample; j ++){
						cout << "N[" << i << "][" << j << "] : " << NN[i][j] << endl;
				}
		}

		double **temp_angdata[m_nyear][20]; // define a temporary array 
		for(int i = 0; i < m_nyear; i++){
				for(int j = 0; j < nsample; j ++){
						temp_angdata[i][j] = new double * [9];
						for(int l = 0; l < 9; l++){
								*(temp_angdata[i][j] + l) =  new double [NN[i][j] + THREADS_PER_BLOCK];
								for(int k = 0; k< NN[i][j]; k ++){
										*(*(temp_angdata[i][j] + l) + k) = *(*(angdata[i][j] + l) + k);
								}
						}
				}
		}

		for(int i = 0; i < m_nyear; i++){    // copy data from cpu to gpu
				for(int j = 0; j < nsample; j ++){
						int size1 = (NN[i][j] + THREADS_PER_BLOCK) *sizeof(double);
						gpu_angdata[i][j] = new double * [9];
						for(int l = 0; l < 9; l++){
								hipMalloc( (void **) &(*(gpu_angdata[i][j] + l)), size1 );
								hipMemcpy( *(gpu_angdata[i][j] + l), *(temp_angdata[i][j] + l), size1, hipMemcpyHostToDevice );
								delete [] *(temp_angdata[i][j] + l);
						}
						hipMalloc( (void **) &gpu_Matrix[i][j], size1 * MATRIX_SIZE  );
						hipMalloc( (void **) &gpu_amp[i][j], size1 * MATRIX_SIZE  );
						out_amp[i][j] = new double [NN[i][j] + THREADS_PER_BLOCK];
				}
		}
}


int rootfile::IOreadData(const int n, AngDisXiXi *ang, double **para, const int  index, const int MM){
		cout << n << "   " << m_file[n] << " " << m_sample[n] << " " <<  m_year[n]<<  "  " << type(n) << endl;


		bool data = false;
		bool mdiy = false;
		bool phsp = false;
		bool bkg = false;
		bool inc = false;
		if(!m_sample[n].CompareTo("data")) data = true;
		if(!m_sample[n].CompareTo("mdiy")) mdiy = true;
		if(!m_sample[n].CompareTo("phsp")) phsp = true;
		if(!m_sample[n].CompareTo("bkg1")) bkg = true;
		if(!m_sample[n].CompareTo("inclusive")) inc = true;

		int runNo_low = 0;
		int runNo_high = 0;
		int iyear = -1;
		int itype = -1;
		if(!m_type[n].CompareTo("xixipm")){
				itype = 0;
		}
		else if(!m_type[n].CompareTo("xixipp")){
				itype = 1;
		}

		if(!m_year[n].CompareTo("2009")){
				iyear = 0;
				runNo_low = 9800;
				runNo_high =  11000;
		}
		else if(!m_year[n].CompareTo("2012")){
				iyear = 1;
				runNo_low = 27100;
				runNo_high = 28400;
		}
		else if(!m_year[n].CompareTo("2018")){
				iyear = 2;
				runNo_low = 52840;
				runNo_high = 56646;
		}
		else if(!m_year[n].CompareTo("2019")){
				iyear = 3;
				runNo_low = 56778 ;
				runNo_high = 59115;
		}
		Int_t NN1;
		std::vector<double> gD1Xithe;
		std::vector<double> gD1Lthe;
		std::vector<double> gD1Lphi;
		std::vector<double> gD1Lbthe;
		std::vector<double> gD1Lbphi;
		std::vector<double> gD1pthe;
		std::vector<double> gD1pphi;
		std::vector<double> gD1apthe;
		std::vector<double> gD1apphi;

		gD1Xithe.clear();
		gD1Lthe.clear();
		gD1Lphi.clear();
		gD1Lbthe.clear();
		gD1Lbphi.clear();
		gD1pthe.clear();
		gD1pphi.clear();
		gD1apthe.clear();
		gD1apphi.clear();

		Double_t the, Lthe, Lphi, Lbthe, Lbphi, pthe, pphi, apthe, apphi;
		Double_t m_LmdDL, m_XiDL, m_XiCosTheta, m_mXi2, m_mXi1, m_mLmd1, m_mn;
		Double_t m_chi2kmf, m_chi2Xi, m_chi2Lmd, m_angle_gam1, m_angle_gam2;

		int runNo;

		TFile *f1 = new TFile(m_file[n], "read");
		TTree *t1 = (TTree*)f1->Get("xixi");
		t1->SetBranchAddress("the", &the);
		t1->SetBranchAddress("Lthe", &Lthe);
		t1->SetBranchAddress("Lphi", &Lphi);
		t1->SetBranchAddress("Lbthe", &Lbthe);
		t1->SetBranchAddress("Lbphi", &Lbphi);
		t1->SetBranchAddress("pthe", &pthe);
		t1->SetBranchAddress("pphi", &pphi);
		t1->SetBranchAddress("apthe", &apthe);
		t1->SetBranchAddress("apphi", &apphi);
		t1->SetBranchAddress("runNo", &runNo);

		t1->SetBranchAddress("LmdDL", &m_LmdDL);
		t1->SetBranchAddress("XiDL", &m_XiDL);
		t1->SetBranchAddress("XiCosTheta", &m_XiCosTheta);
		t1->SetBranchAddress("mXi2", &m_mXi2);
		t1->SetBranchAddress("mXi1", &m_mXi1);
		t1->SetBranchAddress("mLmd1", &m_mLmd1);
		t1->SetBranchAddress("mn", &m_mn);
		t1->SetBranchAddress("chi2kmf", &m_chi2kmf);
		t1->SetBranchAddress("chi2Xi", &m_chi2Xi);
		t1->SetBranchAddress("chi2Lmd", &m_chi2Lmd);
		t1->SetBranchAddress("angle_gam1", &m_angle_gam1);
		t1->SetBranchAddress("angle_gam2", &m_angle_gam2);
		int nn = 0;
		int NEvt = t1->GetEntries();
		int low = 0;
		int high = NEvt;
		int iEvt = NEvt/30;
		if(mdiy){
				low = index * iEvt;
				high = (index+1) * iEvt;
		}
		//		NEvt = 10000;

		int count1 = 0;
		int count2 = 0;
		for(int i = low; i <  high; i++){
				t1->GetEntry(i);
				if(abs(runNo) < runNo_low || abs(runNo) > runNo_high) continue;
				if(m_LmdDL < cut_LmdDL) continue;
				if(m_XiDL < cut_XiDL) continue;
				if(fabs(m_XiCosTheta) >  cut_XiCosTheta) continue;
				if(fabs(m_mXi2 - 1.32171) > cut_mXi) continue;
				if(fabs(m_mXi1 - 1.32171) > cut_mXi) continue;
				if(fabs(m_mLmd1 - 1.1157) > cut_mLmd1) continue;
				if(m_chi2kmf > cut_chi2kmf) continue;
				if(m_chi2Xi > cut_chi2Xi) continue;
				if(m_chi2Lmd > cut_chi2Lmd) continue;


				if(data){
						ang->setDataMass(count1, m_mn);
						count1++;
						if(m_mn < cut_mn1 || m_mn > cut_mn2) continue;
				}
				if(mdiy){
						ang->setDataMass(count1, m_mn);
						count1++;
						if(m_angle_gam1 > 0.3) continue;
						if(m_angle_gam2 > 0.3) continue;
						if(m_angle_gam1 < 0.3 && m_angle_gam2 < 0.3){
								ang->setMCMass(count2, m_mn);
								count2++;
						}
						if(m_mn < cut_mn1 || m_mn > cut_mn2) continue;
				}
				if(phsp){
						if(m_angle_gam1 > 0.3) continue;
						if(m_angle_gam2 > 0.3) continue;
						if(m_mn < cut_mn1 || m_mn > cut_mn2) continue;
				}
				if(bkg){
						if(m_mn < cut_mn1 || m_mn > cut_mn2) continue;
				}
				if(inc){
						if(m_mn < cut_mn1 || m_mn > cut_mn2) continue;
				}

				gD1Xithe.push_back(the);
				gD1Lthe.push_back(Lthe);
				gD1Lphi.push_back(Lphi);
				gD1Lbthe.push_back(Lbthe);
				gD1Lbphi.push_back(Lbphi);
				gD1pthe.push_back(pthe);
				gD1pphi.push_back(pphi);
				gD1apthe.push_back(apthe);
				gD1apphi.push_back(apphi);
				nn++;
				if(phsp){
						ang->AddToIntegral(the, Lthe, Lphi, Lbthe, Lbphi, pthe, pphi, apthe, apphi);
				}
		}
		//	cout << infile << ", " << type << ", " << flag << ", number : " <<   nn << endl;

		NN1 = nn;

		for(int i = 0; i < NN1; i++){
				*(*(para+0)+i) = gD1Xithe[i];
				*(*(para+1)+i) = gD1Lthe[i];
				*(*(para+2)+i) = gD1Lphi[i];
				*(*(para+3)+i) = gD1Lbthe[i];
				*(*(para+4)+i) = gD1Lbphi[i];
				*(*(para+5)+i) = gD1pthe[i];
				*(*(para+6)+i) = gD1pphi[i];
				*(*(para+7)+i) = gD1apthe[i];
				*(*(para+8)+i) = gD1apphi[i];
		}
		f1->Close();
		cout << m_file[n] << endl;
		cout << "Finish" << endl;
		return NN1;
}




double rootfile::IOfcnmll(double *pp){
		double pp1[8], pp2[8];
		for(int i = 0; i < 6; i++){
				pp1[i] = pp[i];
				pp2[i] = pp[i];
		}
		pp1[6] = pp[6]; pp1[7] = pp[7];
		pp2[6] = pp[8]; pp2[7] = pp[9];

		AA_parameter aa_para;
		aa_para.alpha_jpsi = pp[0];
		aa_para.phi_jpsi = pp[1];
		aa_para.alpha_xi = pp[2];
		aa_para.phi_xi = pp[3];
		aa_para.alpha_xibar = pp[4];
		aa_para.phi_xibar = pp[5];
		aa_para.alpha1_lambda = pp[6];
		aa_para.alpha1_lambdabar = pp[7];
		aa_para.alpha2_lambda = pp[8];
		aa_para.alpha2_lambdabar = pp[9];


		hipError_t cudaStatus;
		//		clock_t start,end;
		double loglike[4][12];
		int years = Nyear();
		for(int i = 0; i < years; i ++){
				angdis[i][0]->SetParameter(pp1);
				angdis[i][1]->SetParameter(pp2);
				for (int j = 0; j < (nsample); j++){
						int flag = j / ((nsample) / 2);
						int n = i*nsample + j;
						if(!sample(n).CompareTo("phsp")) continue;
						//	if(!sample(n).CompareTo("mdiy")) continue;
						//	start = clock();
						gpu_Amp <<< (NN[i][j] * MATRIX_SIZE + MATRIX_SIZE * THREADS_PER_BLOCK ) / (MATRIX_SIZE * THREADS_PER_BLOCK), MATRIX_SIZE * THREADS_PER_BLOCK >>> ( 
										*(gpu_angdata[i][j] + 0), 
										*(gpu_angdata[i][j] + 1), 
										*(gpu_angdata[i][j] + 2), 
										*(gpu_angdata[i][j] + 3), 
										*(gpu_angdata[i][j] + 4), 
										*(gpu_angdata[i][j] + 5), 
										*(gpu_angdata[i][j] + 6), 
										*(gpu_angdata[i][j] + 7), 
										*(gpu_angdata[i][j] + 8),
										gpu_amp[i][j],
										(NN[i][j] + THREADS_PER_BLOCK)*80, 
										aa_para, flag, gpu_Matrix[i][j]);
						hipDeviceSynchronize(); // wait until prior kernel is finished
						//	end = clock();
						//	double time3 = ((double)(end-start))/CLOCKS_PER_SEC;
						//	cout << "GPU 3: running kernel " << time3 << " seconds" << endl;
						cudaStatus = hipGetLastError();
						if (cudaStatus != hipSuccess){
								cerr << "failure to call cuda kernel 004!" << endl;
								exit(1);
						}

						int mat_size = (NN[i][j] + THREADS_PER_BLOCK) *sizeof(double);
						hipMemcpy( out_amp[i][j], gpu_amp[i][j], mat_size, hipMemcpyDeviceToHost );
						cudaStatus = hipGetLastError();
						if (cudaStatus != hipSuccess){
								cerr << "failure to call cuda kernel 002!" << endl;
								exit(1);
						}
						loglike[i][j] = 0;
						if(!sample(n).CompareTo("phsp")){
								for(int evt = 0; evt < NN[i][j]; evt++){
										if(*(out_amp[i][j] + evt) <= 0){ cout << "data : " << *(out_amp[i][j] + evt) << endl;  return 0; }
										loglike[i][j] += *(out_amp[i][j] + evt);
								}

						}
						else{
								for(int evt = 0; evt < NN[i][j]; evt++){
										//	cout << "host C munu 0: " << 	*(host_eval + evt) << endl;
										if(*(out_amp[i][j] + evt) <= 0){ cout << "data : " << *(out_amp[i][j] + evt) << endl;  return 0; }
										loglike[i][j] += TMath::Log(*(out_amp[i][j] + evt));
								}
						}
				}
		}
		//	exit(1);
		double norm[4][2];
		for (int i = 0; i < years; i++){
				for (int j = 0; j < 2; j++){
						norm[i][j] = 0;
						//	angdis[i][j]->PrintInt();
						norm[i][j] = angdis[i][j]->CalcToIntegral();
				}
		}

		double llk = 0;
		double l1 = 0;
		double l2 = 0;

		for(int i = 0; i < years; i++){
				//	cout << loglike[i][0] << "		" << loglike[i][2] << "		" << loglike[i][3] << "		" << loglike[i][5] << endl;
				//	l1 = - loglike[i][0] + NBKG[i][0]*loglike[i][3]/Double_t(NN[i][3]) + (Double_t(NN[i][0]) - NBKG[i][0])*TMath::Log(loglike[i][2]/Double_t(NN[i][2]));
				//	l2 = - loglike[i][4] + NBKG[i][1]*loglike[i][7]/Double_t(NN[i][7]) + (Double_t(NN[i][4]) - NBKG[i][1])*TMath::Log(loglike[i][6]/Double_t(NN[i][6]));
				//	l1 = - loglike[i][0] + NBKG[i][0]*loglike[i][2]/Double_t(NN[i][2]) + (Double_t(NN[i][0]) - NBKG[i][0])*TMath::Log(norm[i][0]/Double_t(NN[i][1]));
				//	l2 = - loglike[i][3] + NBKG[i][1]*loglike[i][5]/Double_t(NN[i][5]) + (Double_t(NN[i][3]) - NBKG[i][1])*TMath::Log(norm[i][1]/Double_t(NN[i][4]));
				l1 = - loglike[i][0]  + (Double_t(NN[i][0]))*TMath::Log(norm[i][0]/Double_t(NN[i][1]));
				l2 = - loglike[i][3]  + (Double_t(NN[i][3]))*TMath::Log(norm[i][1]/Double_t(NN[i][4]));
				llk += (l1 + l2);
		}

		if(fit_step%100 == 0){
				std::cout << "Loglike: " << llk << std::endl; 
				for( int i = 0; i<10 ; i++ ) cout<<pp[i]<<" ";
				cout << endl;
		}
		fit_step++;
		return llk;

}

