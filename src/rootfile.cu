#include "hip/hip_runtime.h"
#include "rootfile.cuh"

void rootfile::InitialMemory(){
		nsample = size()/Nyear();
		for(int i = 0; i < Nyear(); i++){
				for(int j = 0; j < nsample; j ++){
						angdata[i][j] = new double * [9];
						for(int l = 0; l < 9; l++){
								*(angdata[i][j] + l) =  new double [NUM];
						}
				}
		}
		for(int i = 0; i < Nyear(); i++){
				for(int j = 0; j < 2; j ++){
								angdis[i][j] = new AngDisXiXi();
								angdis[i][j]->InitialInt();
								angdis[i][j]->InitialIntmDIY();
				}
		}
}


//=====================================================================
void rootfile::ReadData(const int index, const int MM)
{
		//	int years = 2;
		for(int i = 0; i < m_nyear; i++){ 		// read data
				double pp1[8], pp2[8];
				pp1[0] = 0.586;   pp2[0] = 0.586;
				pp1[1] = 1.213;   pp2[1] = 1.213; 
				pp1[2] = -0.375;  pp2[2] = -0.375;
				pp1[3] = 0.02;    pp2[3] = 0.02;
				pp1[4] = 0.375;   pp2[4] = 0.375;
				pp1[5] = -0.02;    pp2[5] = -0.02;
				pp1[6] = 0.692;   pp2[6] = 0.757;
				pp1[7] = -0.757;  pp2[7] = -0.692;
				angdis[i][0]->SetParameter(pp1);
				angdis[i][1]->SetParameter(pp2);
				angdis[i][0]->InitialInt();
				angdis[i][1]->InitialInt();
				angdis[i][0]->InitialIntmDIY();
				angdis[i][1]->InitialIntmDIY();

				for(int j = 0; j < nsample; j++){
						int l = -1;
						int n = i*nsample + j;
						if(!type(n).CompareTo("xixipm")){
								l = 0;
						}
						else if(!type(n).CompareTo("xixipp")){
								l = 1;
						}
						cout << i << "   "  << l  << endl;
						NN[i][j] = readData(n, angdis[i][l], angdata[i][j], index, MM);
						//					if(sample(n).CompareTo("phsp")){
						//							idx.push_back(n);
						//					}
				}
		}
		for(int i = 0; i < m_nyear; i++){
				for(int j = 0; j < nsample; j ++){
						cout << "N[" << i << "][" << j << "] : " << NN[i][j] << endl;
				}
		}

		double **temp_angdata[m_nyear][20]; // define a temporary array 
		for(int i = 0; i < m_nyear; i++){
				for(int j = 0; j < nsample; j ++){
						temp_angdata[i][j] = new double * [9];
						for(int l = 0; l < 9; l++){
								*(temp_angdata[i][j] + l) =  new double [NN[i][j] + THREADS_PER_BLOCK];
								for(int k = 0; k< NN[i][j]; k ++){
										*(*(temp_angdata[i][j] + l) + k) = *(*(angdata[i][j] + l) + k);
								}
						}
				}
		}

		for(int i = 0; i < m_nyear; i++){    // copy data from cpu to gpu
				for(int j = 0; j < nsample; j ++){
						int n = i*nsample + j;
						if(!sample(n).CompareTo("phsp")) continue;
						if(!sample(n).CompareTo("mdiy")) continue;
						int size1 = (NN[i][j] + THREADS_PER_BLOCK) *sizeof(double);
						gpu_angdata[i][j] = new double * [9];
						for(int l = 0; l < 9; l++){
								hipMalloc( (void **) &(*(gpu_angdata[i][j] + l)), size1 );
								hipMemcpy( *(gpu_angdata[i][j] + l), *(temp_angdata[i][j] + l), size1, hipMemcpyHostToDevice );
								delete [] *(temp_angdata[i][j] + l);
						}
						hipMalloc( (void **) &gpu_Matrix[i][j], size1 * MATRIX_SIZE  );
						hipMalloc( (void **) &gpu_amp[i][j], size1 * MATRIX_SIZE  );
						out_amp[i][j] = new double [NN[i][j] + THREADS_PER_BLOCK];
				}
		}
}

void rootfile::FreeMemory(){
		for(int i = 0; i < m_nyear; i++){
				for(int j = 0; j < nsample; j ++){
						int n = i*nsample + j;
						if(!sample(n).CompareTo("phsp")) continue;
						if(!sample(n).CompareTo("mdiy")) continue;
						for(int l = 0; l < 9; l++){
								hipFree((*(gpu_angdata[i][j] + l)));
						}
						hipFree(gpu_Matrix[i][j]);
						hipFree(gpu_amp[i][j]);
						delete [] out_amp[i][j];
				}
		}
}


int rootfile::readData(const int n, AngDisXiXi *ang, double **para, const int  index, const int MM){
		cout << n << "   " << m_file[n] << " " << m_sample[n] << " " <<  m_year[n]<<  "  " << type(n) << endl;


		bool data = false;
		bool mdiy = false;
		bool phsp = false;
		bool bkg = false;
		bool inc = false;
		if(!m_sample[n].CompareTo("data")) data = true;
		if(!m_sample[n].CompareTo("mdiy")) mdiy = true;
		if(!m_sample[n].CompareTo("phsp")) phsp = true;
		if(!m_sample[n].CompareTo("bkg1")) bkg = true;
		if(!m_sample[n].CompareTo("inclusive")) inc = true;

		int runNo_low = 0;
		int runNo_high = 0;
		int iyear = -1;
		int itype = -1;
		if(!m_type[n].CompareTo("xixipm")){
				itype = 0;
		}
		else if(!m_type[n].CompareTo("xixipp")){
				itype = 1;
		}

		if(!m_year[n].CompareTo("2009")){
				iyear = 0;
				runNo_low = 9800;
				runNo_high =  11000;
		}
		else if(!m_year[n].CompareTo("2012")){
				iyear = 1;
				runNo_low = 27100;
				runNo_high = 28400;
		}
		else if(!m_year[n].CompareTo("2018")){
				iyear = 2;
				runNo_low = 52840;
				runNo_high = 56646;
		}
		else if(!m_year[n].CompareTo("2019")){
				iyear = 3;
				runNo_low = 56778 ;
				runNo_high = 59115;
		}
		Int_t NN1;
		std::vector<double> gD1Xithe;
		std::vector<double> gD1Lthe;
		std::vector<double> gD1Lphi;
		std::vector<double> gD1Lbthe;
		std::vector<double> gD1Lbphi;
		std::vector<double> gD1pthe;
		std::vector<double> gD1pphi;
		std::vector<double> gD1apthe;
		std::vector<double> gD1apphi;

		gD1Xithe.clear();
		gD1Lthe.clear();
		gD1Lphi.clear();
		gD1Lbthe.clear();
		gD1Lbphi.clear();
		gD1pthe.clear();
		gD1pphi.clear();
		gD1apthe.clear();
		gD1apphi.clear();

		Double_t the, Lthe, Lphi, Lbthe, Lbphi, pthe, pphi, apthe, apphi;
		Double_t m_LmdDL, m_XiDL, m_XiCosTheta, m_mXi2, m_mXi1, m_mLmd1, m_mn;
		Double_t m_chi2kmf, m_chi2Xi, m_chi2Lmd, m_angle_gam1, m_angle_gam2;

		int runNo;

		TFile *f1 = new TFile(m_file[n], "read");
		TTree *t1 = (TTree*)f1->Get("xixi");
		t1->SetBranchAddress("the", &the);
		t1->SetBranchAddress("Lthe", &Lthe);
		t1->SetBranchAddress("Lphi", &Lphi);
		t1->SetBranchAddress("Lbthe", &Lbthe);
		t1->SetBranchAddress("Lbphi", &Lbphi);
		t1->SetBranchAddress("pthe", &pthe);
		t1->SetBranchAddress("pphi", &pphi);
		t1->SetBranchAddress("apthe", &apthe);
		t1->SetBranchAddress("apphi", &apphi);
		t1->SetBranchAddress("runNo", &runNo);

		t1->SetBranchAddress("LmdDL", &m_LmdDL);
		t1->SetBranchAddress("XiDL", &m_XiDL);
		t1->SetBranchAddress("XiCosTheta", &m_XiCosTheta);
		t1->SetBranchAddress("mXi2", &m_mXi2);
		t1->SetBranchAddress("mXi1", &m_mXi1);
		t1->SetBranchAddress("mLmd1", &m_mLmd1);
		t1->SetBranchAddress("mn", &m_mn);
		t1->SetBranchAddress("chi2kmf", &m_chi2kmf);
		t1->SetBranchAddress("chi2Xi", &m_chi2Xi);
		t1->SetBranchAddress("chi2Lmd", &m_chi2Lmd);
		t1->SetBranchAddress("angle_gam1", &m_angle_gam1);
		t1->SetBranchAddress("angle_gam2", &m_angle_gam2);
		int nn = 0;
		int NEvt = t1->GetEntries();
		int low = 0;
		int high = NEvt;
		if(mdiy)
				high = 90000;
		//		NEvt = 10000;

		int count = 0;
		for(int i = low; i <  high; i++){
				t1->GetEntry(i);
				if(!m_year[n].CompareTo("2018")){
						if(i%3 != 0) continue;
				}
				else if(!m_year[n].CompareTo("2019")){
						if(i%3 != 0) continue;
				}

				if(abs(runNo) < runNo_low || abs(runNo) > runNo_high) continue;
				if(m_LmdDL < cut_LmdDL) continue;
				if(m_XiDL < cut_XiDL) continue;
				if(fabs(m_XiCosTheta) >  cut_XiCosTheta) continue;
				if(fabs(m_mXi2 - 1.32171) > cut_mXi) continue;
				if(fabs(m_mXi1 - 1.32171) > cut_mXi) continue;
				if(fabs(m_mLmd1 - 1.1157) > cut_mLmd1) continue;
				if(m_chi2kmf > cut_chi2kmf) continue;
				if(m_chi2Xi > cut_chi2Xi) continue;
				if(m_chi2Lmd > cut_chi2Lmd) continue;


				if(data){
						ang->setDataMass(count, m_mn);
						count++;
						if(m_mn < cut_mn1 || m_mn > cut_mn2) continue;
				}
				if(mdiy){
						if(m_angle_gam1 > 0.3) continue;
						if(m_angle_gam2 > 0.3) continue;
						ang->setMCMass(count, m_mn);
						count++;
						if(m_mn < cut_mn1 || m_mn > cut_mn2) continue;
				}
				if(phsp){
						if(m_angle_gam1 > 0.3) continue;
						if(m_angle_gam2 > 0.3) continue;
						if(m_mn < cut_mn1 || m_mn > cut_mn2) continue;
				}
				if(bkg){
						if(m_mn < cut_mn1 || m_mn > cut_mn2) continue;
				}
				if(inc){
						if(m_mn < cut_mn1 || m_mn > cut_mn2) continue;
				}

				gD1Xithe.push_back(the);
				gD1Lthe.push_back(Lthe);
				gD1Lphi.push_back(Lphi);
				gD1Lbthe.push_back(Lbthe);
				gD1Lbphi.push_back(Lbphi);
				gD1pthe.push_back(pthe);
				gD1pphi.push_back(pphi);
				gD1apthe.push_back(apthe);
				gD1apphi.push_back(apphi);
				nn++;
				if(phsp){
					//	ang->AddToIntegral(the, Lthe, Lphi, Lbthe, Lbphi, pthe, pphi, apthe, apphi);
						if(!m_norm.CompareTo("phsp")){
								ang->AddToIntegral(the, Lthe, Lphi, Lbthe, Lbphi, pthe, pphi, apthe, apphi);
						}
						else if(!m_norm.CompareTo("mdiy")){
								ang->AddToIntegralmDIY(the, Lthe, Lphi, Lbthe, Lbphi, pthe, pphi, apthe, apphi);
						}
				}
		}
		//	cout << infile << ", " << type << ", " << flag << ", number : " <<   nn << endl;

		NN1 = nn;

		for(int i = 0; i < NN1; i++){
				*(*(para+0)+i) = gD1Xithe[i];
				*(*(para+1)+i) = gD1Lthe[i];
				*(*(para+2)+i) = gD1Lphi[i];
				*(*(para+3)+i) = gD1Lbthe[i];
				*(*(para+4)+i) = gD1Lbphi[i];
				*(*(para+5)+i) = gD1pthe[i];
				*(*(para+6)+i) = gD1pphi[i];
				*(*(para+7)+i) = gD1apthe[i];
				*(*(para+8)+i) = gD1apphi[i];
		}
		f1->Close();
		cout << m_file[n] << endl;
		cout << "Finish" << endl;
		return NN1;
}

void rootfile::MassFit(const int index, const TString m_outfile){
		ofstream out;
		TString fitout = m_outfile + ".massfit";
		out.open(fitout, ios::out | ios::app);
		int iyear =-1;
		int itype = -1;
		cout << "Mass Fit" << endl;
		if(nsample != 4){
				out << "t " << index << "   ";
				for(int i = 0; i < Nyear(); i++){
						for(int j =0; j < 2; j++){
								out << nBkg[i][j] << "  ";
						}
				}
				 out << endl;
		}

		if(nsample != 4){
				out << "c " << index << "	";
				for(int i = 0; i < Nyear(); i++){
						for(int j =0; j < 2; j++){
								NBKG[i][j] = 0;
								NBKG[i][j]  = massFit(i, j);
								out << NBKG[i][j] << "	";
						}
				}
				out << endl;
		}
		out.close();


}

double rootfile::massFit(const int iyear, const int itype){
		using namespace RooFit;
		RooRealVar mn("mn", "M(n#pi^{-}) (GeV/#font[52]{c}^{2})",0.90,0.99);
		RooDataSet signal("signal", "signal", mn);
		cout << angdis[iyear][itype]->getNmc() << endl;
		for(int i = 0; i < angdis[iyear][itype]->getNmc(); i++){
				if(i == 20000) break;
				mn = angdis[iyear][itype]->MCMass(i);
				signal.add(mn);
		}
		RooKeysPdf keysshape("keysshape", "keysshape", mn, signal, RooKeysPdf::MirrorBoth, 2);

		RooDataSet data("data", "data", mn);

		cout << angdis[iyear][itype]->getNdata() << endl;
		double dataNevt = angdis[iyear][itype]->getNdata();
		for(int i = 0; i < angdis[iyear][itype]->getNdata(); i++){
				mn = angdis[iyear][itype]->DataMass(i);
				data.add(mn);
		}


		RooRealVar mean2("mean2", "mean2", 0.005, -0.05, 0.05);	
		RooRealVar sigma2("sigma2", "sigma2",0.005, 0, 0.05);
		RooGaussian ga1("ga1", "ga1", mn, mean2, sigma2);
		mn.setBins(10000, "cache");
		RooFFTConvPdf shape("shape", "shape", mn, keysshape, ga1);


		RooRealVar m0("m0", "m0", 0.98, 0.96, 0.999);
		RooRealVar k("k", "k", -10, -50, -1.0);
		RooArgusBG argus("argus", "argus", mn, m0, k);

		double fitstatus  = 10;
		double nBKG = 0;
		TRandom rdm(3452);
		while(fitstatus > 0.8){
				double inisig = rdm.Rndm()*0.7 + 0.2;
				double inibkg = rdm.Rndm()*0.35 +0.1;
				RooRealVar fra1("fra1", "fra1", dataNevt*inisig, 0, 20000000);
				RooRealVar fra2("fra2", "fra2", dataNevt*inibkg, 0, 20000000);
				RooAddPdf sum("sum", "sig+bak", RooArgList(shape, argus), RooArgList(fra1, fra2));
				RooFitResult *res = sum.fitTo(data, NumCPU(32), Extended(), Save(1));
				fitstatus = res->edm();
				if(res->edm() > 0.8) continue;
				mn.setRange("bkg1", 0.9, 0.98);
				mn.setRange("bkg2", cut_mn1, cut_mn2);
				RooAbsReal *intpolyX10 = argus.createIntegral(mn,  NormSet(mn), Range("bkg1"));
				RooAbsReal *intpolyX11 = argus.createIntegral(mn,  NormSet(mn), Range("bkg2"));
				double back10 = intpolyX10->getVal();
				double back11 = intpolyX11->getVal();
				cout << back10 << endl;
				cout << back11 << endl;
				cout << fra2.getVal() * back11 << endl;
				cout << "Fit Status : " << res->status() << endl;
				res->printValue(cout);
				res->Print();
				nBKG = fra2.getVal() * back11;
		}
		return nBKG;
}

double rootfile::fcnmll(double *pp){
		double pp1[8], pp2[8];
		for(int i = 0; i < 6; i++){
				pp1[i] = pp[i];
				pp2[i] = pp[i];
		}
		pp1[6] = pp[6]; pp1[7] = pp[7];
		pp2[6] = pp[8]; pp2[7] = pp[9];

		AA_parameter aa_para;
		aa_para.alpha_jpsi = pp[0];
		aa_para.phi_jpsi = pp[1];
		aa_para.alpha_xi = pp[2];
		aa_para.phi_xi = pp[3];
		aa_para.alpha_xibar = pp[4];
		aa_para.phi_xibar = pp[5];
		aa_para.alpha1_lambda = pp[6];
		aa_para.alpha1_lambdabar = pp[7];
		aa_para.alpha2_lambda = pp[8];
		aa_para.alpha2_lambdabar = pp[9];


		hipError_t cudaStatus;
		//	clock_t start,end;
		double loglike[4][12];
		int years = Nyear();
		for(int i = 0; i < years; i ++){
				angdis[i][0]->SetParameter(pp1);
				angdis[i][1]->SetParameter(pp2);
				for (int j = 0; j < (nsample); j++){
						int flag = j / ((nsample) / 2);
						int n = i*nsample + j;
						if(!sample(n).CompareTo("phsp")) continue;
						if(!sample(n).CompareTo("mdiy")) continue;
						//	start = clock();
						gpu_Amp <<< (NN[i][j] * MATRIX_SIZE + MATRIX_SIZE * THREADS_PER_BLOCK ) / (MATRIX_SIZE * THREADS_PER_BLOCK), MATRIX_SIZE * THREADS_PER_BLOCK >>> ( 
										*(gpu_angdata[i][j] + 0), 
										*(gpu_angdata[i][j] + 1), 
										*(gpu_angdata[i][j] + 2), 
										*(gpu_angdata[i][j] + 3), 
										*(gpu_angdata[i][j] + 4), 
										*(gpu_angdata[i][j] + 5), 
										*(gpu_angdata[i][j] + 6), 
										*(gpu_angdata[i][j] + 7), 
										*(gpu_angdata[i][j] + 8),
										gpu_amp[i][j],
										(NN[i][j] + THREADS_PER_BLOCK)*80, 
										aa_para, flag, gpu_Matrix[i][j]);
						hipDeviceSynchronize(); // wait until prior kernel is finished
						//	end = clock();
						//	double time3 = ((double)(end-start))/CLOCKS_PER_SEC;
						//	cout << "GPU 3: running kernel " << time3 << " seconds" << endl;
						cudaStatus = hipGetLastError();
						if (cudaStatus != hipSuccess){
								cerr << "failure to call cuda kernel 004!" << endl;
								exit(1);
						}

						int mat_size = (NN[i][j] + THREADS_PER_BLOCK) *sizeof(double);
						hipMemcpy( out_amp[i][j], gpu_amp[i][j], mat_size, hipMemcpyDeviceToHost );
						cudaStatus = hipGetLastError();
						if (cudaStatus != hipSuccess){
								cerr << "failure to call cuda kernel 002!" << endl;
								exit(1);
						}
						loglike[i][j] = 0;
						if(!sample(n).CompareTo("phsp")){
								for(int evt = 0; evt < NN[i][j]; evt++){
										if(*(out_amp[i][j] + evt) <= 0){ cout << "data : " << *(out_amp[i][j] + evt) << endl;  return 0; }
										loglike[i][j] += *(out_amp[i][j] + evt);
								}

						}
						else{
								for(int evt = 0; evt < NN[i][j]; evt++){
										//	cout << "host C munu 0: " << 	*(host_eval + evt) << endl;
										if(*(out_amp[i][j] + evt) <= 0){ cout << "data : " << *(out_amp[i][j] + evt) << endl;  return 0; }
										loglike[i][j] += TMath::Log(*(out_amp[i][j] + evt));
								}
						}
				}
		}
		//	exit(1);
		double norm[4][2];
		for (int i = 0; i < years; i++){
				for (int j = 0; j < 2; j++){
						norm[i][j] = 0;
						//	angdis[i][j]->PrintInt();
					//	norm[i][j] = angdis[i][j]->CalcToIntegral();
						if(!m_norm.CompareTo("phsp")){
								norm[i][j] = angdis[i][j]->CalcToIntegral();
						}
						else if(!m_norm.CompareTo("mdiy")){
								norm[i][j] = angdis[i][j]->CalcToIntegralmDIY();
						}
				}
		}



		double llk = 0;
		double l1 = 0;
		double l2 = 0;

		for(int i = 0; i < years; i++){
				//			l1 = - loglike[i][0] + NBKG[i][0]*loglike[i][3]/Double_t(NN[i][3]) + (Double_t(NN[i][0]) - NBKG[i][0])*TMath::Log(loglike[i][2]/Double_t(NN[i][2]));
				//			l2 = - loglike[i][4] + NBKG[i][1]*loglike[i][7]/Double_t(NN[i][7]) + (Double_t(NN[i][4]) - NBKG[i][1])*TMath::Log(loglike[i][6]/Double_t(NN[i][6]));
			//	l1 = - loglike[i][0] + NBKG[i][0]*loglike[i][3]/Double_t(NN[i][3]) + (Double_t(NN[i][0]) - NBKG[i][0])*TMath::Log(norm[i][0]/Double_t(NN[i][2]));
			//	l2 = - loglike[i][4] + NBKG[i][1]*loglike[i][7]/Double_t(NN[i][7]) + (Double_t(NN[i][4]) - NBKG[i][1])*TMath::Log(norm[i][1]/Double_t(NN[i][6]));

				int iy = iyear(i * nsample );
	//			cout << i << " " << year(i * nsample) <<"  " << NBKG[iy][0] << "	" << NBKG[iy][1] << "	";
				for (int j = 0; j < (nsample)/2; j++){
						if(!sample(j).CompareTo("data")){
								l1 = - loglike[i][j];
								l2 = - loglike[i][j + (nsample)/2];
						}
						else if(!sample(j).CompareTo("phsp")){
								l1 += (Double_t(NN[i][0]) - NBKG[iy][0]) * TMath::Log(norm[i][0]/Double_t(NN[i][j]));
								l2 += (Double_t(NN[i][(nsample)/2]) - NBKG[iy][1]) * TMath::Log(norm[i][1]/Double_t(NN[i][j + (nsample)/2]));
						}
						else if(!sample(j).CompareTo("bkg1")){
								l1 += NBKG[iy][0]*loglike[i][j]/Double_t(NN[i][j]);
								l2 += NBKG[iy][1]*loglike[i][j + (nsample)/2]/Double_t(NN[i][j + (nsample)/2]);
						}
						else if(!sample(j).CompareTo("bkg2")){
								l1 += NBKG[iy][0]*loglike[i][j]/Double_t(NN[i][j]);
								l2 += NBKG[iy][1]*loglike[i][j + (nsample)/2]/Double_t(NN[i][j + (nsample)/2]);
						}
				}


				llk += (l1 + l2);
		}

		if(fit_step%100 == 0){

				for(int i = 0; i < years; i++){
						int iy = iyear(i * nsample );
						for (int j = 0; j < (nsample)/2; j++){
								if(!sample(j).CompareTo("data")){
										cout << "data " <<loglike[i][j] << "	" << loglike[i][j + (nsample)/2] << "	";
								}
								else if(!sample(j).CompareTo("phsp")){
										cout << "phsp "<< norm[i][0] << "	" << norm[i][1] << "	" ;
								}
								else if(!sample(j).CompareTo("bkg1")){
										cout << "bkg1 " << loglike[i][j] << "  " << loglike[i][j + (nsample)/2] << "   ";
								}
								else if(!sample(j).CompareTo("bkg2")){
								}
						}
						cout << endl;
				}





				//	   std::cout << "Loglike: " << llk << std::endl; 
				//	   for( int i = 0; i<10 ; i++ ) cout<<pp[i]<<" ";
				//	   cout << endl;
		}

		fit_step++;
		return llk;

}


//=====================================================================
void rootfile::IOReadData(const int index, const int MM)
{
		//	int years = 2;
		for(int i = 0; i < m_nyear; i++){ 		// read data
				double pp1[8], pp2[8];
				pp1[0] = 0.586;   pp2[0] = 0.586;
				pp1[1] = 1.213;   pp2[1] = 1.213; 
				pp1[2] = -0.375;  pp2[2] = -0.375;
				pp1[3] = 0.02;    pp2[3] = 0.02;
				pp1[4] = 0.375;   pp2[4] = 0.375;
				pp1[5] = -0.02;    pp2[5] = -0.02;
				pp1[6] = 0.692;   pp2[6] = 0.757;
				pp1[7] = -0.757;  pp2[7] = -0.692;
				angdis[i][0]->SetParameter(pp1);
				angdis[i][1]->SetParameter(pp2);

				angdis[i][0]->InitialInt();
				angdis[i][1]->InitialInt();
				angdis[i][0]->InitialIntmDIY();
				angdis[i][1]->InitialIntmDIY();

				for(int j = 0; j < nsample; j++){
						int l = -1;
						int n = i*nsample + j;
						if(!type(n).CompareTo("xixipm")){
								l = 0;
						}
						else if(!type(n).CompareTo("xixipp")){
								l = 1;
						}
						cout << i << "   "  << l  << endl;
						NN[i][j] = IOreadData(n, angdis[i][l], angdata[i][j], index, MM);
						//					if(sample(n).CompareTo("phsp")){
						//							idx.push_back(n);
						//					}
				}
		}
		for(int i = 0; i < m_nyear; i++){
				for(int j = 0; j < nsample; j ++){
						if(j == 0){
								cout << "N[" << i << "][" << j << "] : " << NN[i][j]  <<"  " << nBkg[i][j]<< endl;
						}
						else  if(j == nsample/2){
								cout << "N[" << i << "][" << j << "] : " << NN[i][j]  <<"  " << nBkg[i][1]<< endl;
						}
						else {
								cout << "N[" << i << "][" << j << "] : " << NN[i][j] << endl;
						}
				}
		}

		double **temp_angdata[m_nyear][20]; // define a temporary array 
		for(int i = 0; i < m_nyear; i++){
				for(int j = 0; j < nsample; j ++){
						temp_angdata[i][j] = new double * [9];
						for(int l = 0; l < 9; l++){
								*(temp_angdata[i][j] + l) =  new double [NN[i][j] + THREADS_PER_BLOCK];
								for(int k = 0; k< NN[i][j]; k ++){
										*(*(temp_angdata[i][j] + l) + k) = *(*(angdata[i][j] + l) + k);
								}
						}
				}
		}

		for(int i = 0; i < m_nyear; i++){    // copy data from cpu to gpu
				for(int j = 0; j < nsample; j ++){
						int size1 = (NN[i][j] + THREADS_PER_BLOCK) *sizeof(double);
						gpu_angdata[i][j] = new double * [9];
						for(int l = 0; l < 9; l++){
								hipMalloc( (void **) &(*(gpu_angdata[i][j] + l)), size1 );
								hipMemcpy( *(gpu_angdata[i][j] + l), *(temp_angdata[i][j] + l), size1, hipMemcpyHostToDevice );
								delete [] *(temp_angdata[i][j] + l);
						}
						hipMalloc( (void **) &gpu_Matrix[i][j], size1 * MATRIX_SIZE  );
						hipMalloc( (void **) &gpu_amp[i][j], size1 * MATRIX_SIZE  );
						out_amp[i][j] = new double [NN[i][j] + THREADS_PER_BLOCK];
				}
		}
}


int rootfile::IOreadData(const int n, AngDisXiXi *ang, double **para, const int  index, const int MM){
		cout << n << "   " << m_file[n] << " " << m_sample[n] << " " <<  m_year[n]<<  "  " << type(n) << endl;


		bool data = false;
		bool mdiy = false;
		bool phsp = false;
		bool bkg = false;
		bool inc = false;
		if(!m_sample[n].CompareTo("data")) data = true;
		if(!m_sample[n].CompareTo("mdiy")) mdiy = true;
		if(!m_sample[n].CompareTo("phsp")) phsp = true;
		if(!m_sample[n].CompareTo("bkg1")) bkg = true;
		if(!m_sample[n].CompareTo("inclusive")) inc = true;

		int runNo_low = 0;
		int runNo_high = 0;
		int iyear = -1;
		int itype = -1;
		if(!m_type[n].CompareTo("xixipm")){
				itype = 0;
		}
		else if(!m_type[n].CompareTo("xixipp")){
				itype = 1;
		}

		if(!m_year[n].CompareTo("2009")){
				iyear = 0;
				runNo_low = 9800;
				runNo_high =  11000;
		}
		else if(!m_year[n].CompareTo("2012")){
				iyear = 1;
				runNo_low = 27100;
				runNo_high = 28400;
		}
		else if(!m_year[n].CompareTo("2018")){
				iyear = 2;
				runNo_low = 52840;
				runNo_high = 56646;
		}
		else if(!m_year[n].CompareTo("2019")){
				iyear = 3;
				runNo_low = 56778 ;
				runNo_high = 59115;
		}
		Int_t NN1;
		std::vector<double> gD1Xithe;
		std::vector<double> gD1Lthe;
		std::vector<double> gD1Lphi;
		std::vector<double> gD1Lbthe;
		std::vector<double> gD1Lbphi;
		std::vector<double> gD1pthe;
		std::vector<double> gD1pphi;
		std::vector<double> gD1apthe;
		std::vector<double> gD1apphi;

		gD1Xithe.clear();
		gD1Lthe.clear();
		gD1Lphi.clear();
		gD1Lbthe.clear();
		gD1Lbphi.clear();
		gD1pthe.clear();
		gD1pphi.clear();
		gD1apthe.clear();
		gD1apphi.clear();

		Double_t the, Lthe, Lphi, Lbthe, Lbphi, pthe, pphi, apthe, apphi;
		Double_t m_LmdDL, m_XiDL, m_XiCosTheta, m_mXi2, m_mXi1, m_mLmd1, m_mn;
		Double_t m_chi2kmf, m_chi2Xi, m_chi2Lmd, m_angle_gam1, m_angle_gam2;

		int runNo;

		TFile *f1 = new TFile(m_file[n], "read");
		TTree *t1 = (TTree*)f1->Get("xixi");
		t1->SetBranchAddress("the", &the);
		t1->SetBranchAddress("Lthe", &Lthe);
		t1->SetBranchAddress("Lphi", &Lphi);
		t1->SetBranchAddress("Lbthe", &Lbthe);
		t1->SetBranchAddress("Lbphi", &Lbphi);
		t1->SetBranchAddress("pthe", &pthe);
		t1->SetBranchAddress("pphi", &pphi);
		t1->SetBranchAddress("apthe", &apthe);
		t1->SetBranchAddress("apphi", &apphi);
		t1->SetBranchAddress("runNo", &runNo);

		t1->SetBranchAddress("LmdDL", &m_LmdDL);
		t1->SetBranchAddress("XiDL", &m_XiDL);
		t1->SetBranchAddress("XiCosTheta", &m_XiCosTheta);
		t1->SetBranchAddress("mXi2", &m_mXi2);
		t1->SetBranchAddress("mXi1", &m_mXi1);
		t1->SetBranchAddress("mLmd1", &m_mLmd1);
		t1->SetBranchAddress("mn", &m_mn);
		t1->SetBranchAddress("chi2kmf", &m_chi2kmf);
		t1->SetBranchAddress("chi2Xi", &m_chi2Xi);
		t1->SetBranchAddress("chi2Lmd", &m_chi2Lmd);
		t1->SetBranchAddress("angle_gam1", &m_angle_gam1);
		t1->SetBranchAddress("angle_gam2", &m_angle_gam2);
		int nn = 0;
		int NEvt = t1->GetEntries();
		int low = 0;
		int high = NEvt;
		int iEvt = NEvt/30;
		int iy  = n/nsample;
		int isample  = n%nsample;
		int isam = isample/(nsample/2);
		if(mdiy){
				low = index * iEvt;
				high = (index+1) * iEvt;
				nBkg[iy][isam] = 0;
		}
		//		NEvt = 10000;

		int count1 = 0;
		int count2 = 0;
		for(int i = low; i < high; i++){
				t1->GetEntry(i);
				if(abs(runNo) < runNo_low || abs(runNo) > runNo_high) continue;
				if(m_LmdDL < cut_LmdDL) continue;
				if(m_XiDL < cut_XiDL) continue;
				if(fabs(m_XiCosTheta) >  cut_XiCosTheta) continue;
				if(fabs(m_mXi2 - 1.32171) > cut_mXi) continue;
				if(fabs(m_mXi1 - 1.32171) > cut_mXi) continue;
				if(fabs(m_mLmd1 - 1.1157) > cut_mLmd1) continue;
				if(m_chi2kmf > cut_chi2kmf) continue;
				if(m_chi2Xi > cut_chi2Xi) continue;
				if(m_chi2Lmd > cut_chi2Lmd) continue;


				if(data){
						ang->setDataMass(count1, m_mn);
						count1++;
						if(m_mn < cut_mn1 || m_mn > cut_mn2) continue;
				}
				if(mdiy){
						ang->setDataMass(count1, m_mn);
						count1++;
						if(nsample == 4){
								if(m_angle_gam1 > 0.3) continue;
								if(m_angle_gam2 > 0.3) continue;
						}
						if(m_angle_gam1 < 0.3 && m_angle_gam2 < 0.3){
								ang->setMCMass(count2, m_mn);
								count2++;
						}

						if(m_angle_gam1 > 0.3 || m_angle_gam2 > 0.3){
								if(m_mn > cut_mn1 && m_mn < cut_mn2){
										nBkg[iy][isam]++;
								}
						}
						if(m_mn < cut_mn1 || m_mn > cut_mn2) continue;
				}
				if(phsp){
						if(m_angle_gam1 > 0.3) continue;
						if(m_angle_gam2 > 0.3) continue;
						if(m_mn < cut_mn1 || m_mn > cut_mn2) continue;
				}
				if(bkg){
						if(m_mn < cut_mn1 || m_mn > cut_mn2) continue;
				}
				if(inc){
						if(m_mn < cut_mn1 || m_mn > cut_mn2) continue;
				}
				gD1Xithe.push_back(the);
				gD1Lthe.push_back(Lthe);
				gD1Lphi.push_back(Lphi);
				gD1Lbthe.push_back(Lbthe);
				gD1Lbphi.push_back(Lbphi);
				gD1pthe.push_back(pthe);
				gD1pphi.push_back(pphi);
				gD1apthe.push_back(apthe);
				gD1apphi.push_back(apphi);
				nn++;
				if(phsp){
						if(!m_norm.CompareTo("phsp")){
								ang->AddToIntegral(the, Lthe, Lphi, Lbthe, Lbphi, pthe, pphi, apthe, apphi);
						}
						else if(!m_norm.CompareTo("mdiy")){
								ang->AddToIntegralmDIY(the, Lthe, Lphi, Lbthe, Lbphi, pthe, pphi, apthe, apphi);
						}
				}
		}
		//	cout << infile << ", " << type << ", " << flag << ", number : " <<   nn << endl;

		NN1 = nn;

		for(int i = 0; i < NN1; i++){
				*(*(para+0)+i) = gD1Xithe[i];
				*(*(para+1)+i) = gD1Lthe[i];
				*(*(para+2)+i) = gD1Lphi[i];
				*(*(para+3)+i) = gD1Lbthe[i];
				*(*(para+4)+i) = gD1Lbphi[i];
				*(*(para+5)+i) = gD1pthe[i];
				*(*(para+6)+i) = gD1pphi[i];
				*(*(para+7)+i) = gD1apthe[i];
				*(*(para+8)+i) = gD1apphi[i];
		}
		f1->Close();
		cout << m_file[n] << endl;
		cout << "Finish " << nBkg[iy][isam]  << endl;
		return NN1;
}




double rootfile::IOfcnmll(double *pp){
		double pp1[8], pp2[8];
		for(int i = 0; i < 6; i++){
				pp1[i] = pp[i];
				pp2[i] = pp[i];
		}
		pp1[6] = pp[6]; pp1[7] = pp[7];
		pp2[6] = pp[8]; pp2[7] = pp[9];

		AA_parameter aa_para;
		aa_para.alpha_jpsi = pp[0];
		aa_para.phi_jpsi = pp[1];
		aa_para.alpha_xi = pp[2];
		aa_para.phi_xi = pp[3];
		aa_para.alpha_xibar = pp[4];
		aa_para.phi_xibar = pp[5];
		aa_para.alpha1_lambda = pp[6];
		aa_para.alpha1_lambdabar = pp[7];
		aa_para.alpha2_lambda = pp[8];
		aa_para.alpha2_lambdabar = pp[9];


		hipError_t cudaStatus;
		//		clock_t start,end;
		double loglike[4][12];
		int years = Nyear();
		for(int i = 0; i < years; i ++){
				angdis[i][0]->SetParameter(pp1);
				angdis[i][1]->SetParameter(pp2);
				for (int j = 0; j < (nsample); j++){
						int flag = j / ((nsample) / 2);
						int n = i*nsample + j;
						if(!sample(n).CompareTo("phsp")) continue;
						//	if(!sample(n).CompareTo("mdiy")) continue;
						//	start = clock();
						gpu_Amp <<< (NN[i][j] * MATRIX_SIZE + MATRIX_SIZE * THREADS_PER_BLOCK ) / (MATRIX_SIZE * THREADS_PER_BLOCK), MATRIX_SIZE * THREADS_PER_BLOCK >>> ( 
										*(gpu_angdata[i][j] + 0), 
										*(gpu_angdata[i][j] + 1), 
										*(gpu_angdata[i][j] + 2), 
										*(gpu_angdata[i][j] + 3), 
										*(gpu_angdata[i][j] + 4), 
										*(gpu_angdata[i][j] + 5), 
										*(gpu_angdata[i][j] + 6), 
										*(gpu_angdata[i][j] + 7), 
										*(gpu_angdata[i][j] + 8),
										gpu_amp[i][j],
										(NN[i][j] + THREADS_PER_BLOCK)*80, 
										aa_para, flag, gpu_Matrix[i][j]);
						hipDeviceSynchronize(); // wait until prior kernel is finished
						//	end = clock();
						//	double time3 = ((double)(end-start))/CLOCKS_PER_SEC;
						//	cout << "GPU 3: running kernel " << time3 << " seconds" << endl;
						cudaStatus = hipGetLastError();
						if (cudaStatus != hipSuccess){
								cerr << "failure to call cuda kernel 004!" << endl;
								exit(1);
						}

						int mat_size = (NN[i][j] + THREADS_PER_BLOCK) *sizeof(double);
						hipMemcpy( out_amp[i][j], gpu_amp[i][j], mat_size, hipMemcpyDeviceToHost );
						cudaStatus = hipGetLastError();
						if (cudaStatus != hipSuccess){
								cerr << "failure to call cuda kernel 002!" << endl;
								exit(1);
						}
						loglike[i][j] = 0;
						if(!sample(n).CompareTo("phsp")){
								for(int evt = 0; evt < NN[i][j]; evt++){
										if(*(out_amp[i][j] + evt) <= 0){ cout << "data : " << *(out_amp[i][j] + evt) << endl;  return 0; }
										loglike[i][j] += *(out_amp[i][j] + evt);
								}

						}
						else{
								for(int evt = 0; evt < NN[i][j]; evt++){
										//	cout << "host C munu 0: " << 	*(host_eval + evt) << endl;
										if(*(out_amp[i][j] + evt) <= 0){ cout << "data : " << *(out_amp[i][j] + evt) << endl;  return 0; }
										loglike[i][j] += TMath::Log(*(out_amp[i][j] + evt));
								}
						}
				}
		}
		//	exit(1);
		double norm[4][2];
		for (int i = 0; i < years; i++){
				for (int j = 0; j < 2; j++){
						norm[i][j] = 0;
						//	angdis[i][j]->PrintInt();
						if(!m_norm.CompareTo("phsp")){
								norm[i][j] = angdis[i][j]->CalcToIntegral();
						}
						else if(!m_norm.CompareTo("mdiy")){
								norm[i][j] = angdis[i][j]->CalcToIntegralmDIY();
						}
				}
		}

		double llk = 0;
		double l1 = 0;
		double l2 = 0;

		for(int i = 0; i < years; i++){
				//	cout << loglike[i][0] << "		" << loglike[i][2] << "		" << loglike[i][3] << "		" << loglike[i][5] << endl;
				//	l1 = - loglike[i][0] + NBKG[i][0]*loglike[i][3]/Double_t(NN[i][3]) + (Double_t(NN[i][0]) - NBKG[i][0])*TMath::Log(loglike[i][2]/Double_t(NN[i][2]));
				//	l2 = - loglike[i][4] + NBKG[i][1]*loglike[i][7]/Double_t(NN[i][7]) + (Double_t(NN[i][4]) - NBKG[i][1])*TMath::Log(loglike[i][6]/Double_t(NN[i][6]));
				//	l1 = - loglike[i][0] + NBKG[i][0]*loglike[i][2]/Double_t(NN[i][2]) + (Double_t(NN[i][0]) - NBKG[i][0])*TMath::Log(norm[i][0]/Double_t(NN[i][1]));
				//	l2 = - loglike[i][3] + NBKG[i][1]*loglike[i][5]/Double_t(NN[i][5]) + (Double_t(NN[i][3]) - NBKG[i][1])*TMath::Log(norm[i][1]/Double_t(NN[i][4]));

				int iy = iyear(i * nsample );
				//			cout << i << " " << year(i * nsample) <<"  " << NBKG[iy][0] << "	" << NBKG[iy][1] << "	";
				for (int j = 0; j < (nsample)/2; j++){
						if(!sample(j).CompareTo("mdiy")){
								l1 = - loglike[i][j];
								l2 = - loglike[i][j + (nsample)/2];
						}
						else if(!sample(j).CompareTo("phsp")){
								l1 += (Double_t(NN[i][0]) - NBKG[iy][0]) * TMath::Log(norm[i][0]/Double_t(NN[i][j]));
								l2 += (Double_t(NN[i][(nsample)/2]) - NBKG[iy][1]) * TMath::Log(norm[i][1]/Double_t(NN[i][j + (nsample)/2]));
						}
						else if(!sample(j).CompareTo("bkg1")){
								l1 += NBKG[iy][0]*loglike[i][j]/Double_t(NN[i][j]);
								l2 += NBKG[iy][1]*loglike[i][j + (nsample)/2]/Double_t(NN[i][j + (nsample)/2]);
						}
						else if(!sample(j).CompareTo("bkg2")){
								l1 += NBKG[iy][0]*loglike[i][j]/Double_t(NN[i][j]);
								l2 += NBKG[iy][1]*loglike[i][j + (nsample)/2]/Double_t(NN[i][j + (nsample)/2]);
						}
				}
				//l1 = - loglike[i][0]  + (Double_t(NN[i][0]))*TMath::Log(norm[i][0]/Double_t(NN[i][1]));
				//l2 = - loglike[i][2]  + (Double_t(NN[i][2]))*TMath::Log(norm[i][1]/Double_t(NN[i][3]));
				llk += (l1 + l2);
		}
		//	cout << endl;

		if(fit_step%100 == 0){
				cout << norm[0][0] << "	" << norm[0][1] << "		" << loglike[0][0] << "		" << loglike[0][2] << endl;
				std::cout << "Loglike: " << llk << "   " <<  norm[0][0] << std::endl; 
				for( int i = 0; i<10 ; i++ ) cout<<pp[i]<<" ";
				cout << endl;
		}
		fit_step++;
		return llk;

}

void rootfile::readBKG(const int index){
		ifstream infile("/home/liul/workarea/XiXi/MLL/XIXIRUN/iocheckv5/2018/bkg.txt", ios::in);
		if(!infile){
				cerr << "open error!" << endl;
				exit(2);
		}
		TString str;
		int idx;
		double ibkg;
		while(!infile.eof()){
				infile >> str;
				if(!str.CompareTo("c")){
						infile >> idx;
						if(idx == index)
								for(int i = 0; i < 4; i++){
										for(int j =0; j < 2; j++){
												infile >> ibkg;
												NBKG[i][j] = ibkg;
										}
								}
				}
		}
}

