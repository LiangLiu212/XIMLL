#include "hip/hip_runtime.h"
#include "Amplitude.cuh"

__device__ double device_AADecay12(const double alpha, const double xiphi,const double theta, const double phi, const int index){

		double gamma = sqrt(1-alpha*alpha)*cos(xiphi);
		double beta = sqrt(1-alpha*alpha)*sin(xiphi);
		double c=0;
		switch(index){
				case 0: c=1; break;
				case 3: c=alpha; break;
				case 4: c=alpha*cos(phi)*sin(theta); break;
				case 5: c=gamma*cos(phi)*cos(theta) - beta*sin(phi); break;
				case 6: c=-(beta*cos(phi)*cos(theta)) - gamma*sin(phi); break;
				case 7: c=cos(phi)*sin(theta); break;
				case 8: c=alpha*sin(phi)*sin(theta); break;
				case 9: c=beta*cos(phi) + gamma*cos(theta)*sin(phi); break;
				case 10: c=gamma*cos(phi) - beta*cos(theta)*sin(phi); break;
				case 11: c=sin(phi)*sin(theta); break;
				case 12: c=alpha*cos(theta); break;
				case 13: c=-(gamma*sin(theta)); break;
				case 14: c=beta*sin(theta); break;
				case 15: c=cos(theta); break;
		}

		//printf("device_AADecay12 %d ~ %f\n", index, c);
		return c;
}


__device__ double  device_AAProd1212(double alpha_Jpsi, double delta_phi, double Jpsi_the, const int index){
		double v  = sqrt(1-alpha_Jpsi*alpha_Jpsi);
		double c=0;
		switch (index){
				case 0: c=(1+alpha_Jpsi*cos(Jpsi_the)*cos(Jpsi_the)); break;
				case 2: c=v*sin(Jpsi_the)*cos(Jpsi_the)*sin(delta_phi); break;
				case 5: c=sin(Jpsi_the)*sin(Jpsi_the); break;
				case 7: c=v*sin(Jpsi_the)*cos(Jpsi_the)* cos(delta_phi); break;
				case 8: c=-v*sin(Jpsi_the)*cos(Jpsi_the)*sin(delta_phi); break;
				case 10: c=alpha_Jpsi*sin(Jpsi_the)*sin(Jpsi_the); break;
				case 13: c=-v*sin(Jpsi_the)*cos(Jpsi_the)* cos(delta_phi); break;
				case 15: c=-alpha_Jpsi - cos(Jpsi_the)*cos(Jpsi_the); break;
		}
		//printf("device_AADecay1212 %d ~ %f\n", index, c);
		return c;
}


__device__ double Amp(double *Matrix, const int Index){
		double amp = 0;
		int i1 = 0;
		int i2 = 16;
		int i3 = 32;
		int i4 = 48;
		int i5 = 64;
		for(int mu=0; mu<4;mu++){// Xi loop
				for(int nu=0;nu<4;nu++){// Xibar loop
						for(int k=0;k<4;k++){
								for(int j=0;j<4;j++){
										amp += Matrix[Index*80 + i1 + mu*4 + nu]*  // G_numu
												Matrix[Index*80 + i2 + mu*4 + k]*  // A_numu for Xi decay
												Matrix[Index*80 + i3 + nu*4 + j]*  // A_numu for Xibar decay
												Matrix[Index*80 + i4 + k*4 + 0]*	// A_numu for lambda decay
												Matrix[Index*80 + i5 + j*4 + 0];  	// A_numu for lambdabar decay
								}
						}
				}
		}
		return amp;
}

__global__ void gpu_Amp(
				double *g_xithe,
				double *g_lthe,
				double *g_lphi,
				double *g_lbthe,
				double *g_lbphi,
				double *g_pthe,
				double *g_pphi,
				double *g_apthe,
				double *g_apphi,
				double *amp,
				const int g_NN,
				const AA_parameter g_para,
				const int g_flag, double *Matrix){

		int index = blockIdx.x * blockDim.x + threadIdx.x;
		if( index <  g_NN){
				int iEvt = index / 80;

				int IMat = index % 80;
				int iMat = IMat / 16;
				int jMat = IMat % 16;
				if(g_flag  == 0){

						switch (iMat) {
								case 0: Matrix[index] = device_AAProd1212(g_para.alpha_jpsi, g_para.phi_jpsi, g_xithe[iEvt], jMat); break;
								case 1: Matrix[index] = device_AADecay12(g_para.alpha_xi, g_para.phi_xi, g_lthe[iEvt], g_lphi[iEvt], jMat); break;
								case 2: Matrix[index] = device_AADecay12(g_para.alpha_xibar, g_para.phi_xibar, g_lbthe[iEvt], g_lbphi[iEvt], jMat); break;
								case 3: Matrix[index] = device_AADecay12(g_para.alpha1_lambda, 0, g_pthe[iEvt], g_pphi[iEvt], jMat); break;
								case 4: Matrix[index] = device_AADecay12(g_para.alpha1_lambdabar, 0, g_apthe[iEvt], g_apphi[iEvt], jMat); break;
						}

				}
				else if(g_flag  == 1){

						switch (iMat) {
								case 0: Matrix[index] = device_AAProd1212(g_para.alpha_jpsi, g_para.phi_jpsi, g_xithe[iEvt], jMat); break;
								case 1: Matrix[index] = device_AADecay12(g_para.alpha_xi, g_para.phi_xi, g_lthe[iEvt], g_lphi[iEvt], jMat); break;
								case 2: Matrix[index] = device_AADecay12(g_para.alpha_xibar, g_para.phi_xibar, g_lbthe[iEvt], g_lbphi[iEvt], jMat); break;
								case 3: Matrix[index] = device_AADecay12(g_para.alpha2_lambda, 0, g_pthe[iEvt], g_pphi[iEvt], jMat); break;
								case 4: Matrix[index] = device_AADecay12(g_para.alpha2_lambdabar, 0, g_apthe[iEvt], g_apphi[iEvt], jMat); break;
						}
				}
				__syncthreads();

				if(IMat == 0){
						amp[iEvt] = Amp(Matrix, iEvt);
						//printf("%d ~ %f\n", iEvt, amp[iEvt]);
				}
		}
		__syncthreads();
}

