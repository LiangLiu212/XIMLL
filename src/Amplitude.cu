#include "hip/hip_runtime.h"
#include "Amplitude.cuh"

__device__ double device_AADecay12(const double alpha, const double xiphi,const double theta, const double phi, const int index){

		double gamma = sqrt(1-alpha*alpha)*cos(xiphi);
		double beta = sqrt(1-alpha*alpha)*sin(xiphi);
		double c=0;
		switch(index){
				case 0: c=1; break;
				case 3: c=alpha; break;
				case 4: c=alpha*cos(phi)*sin(theta); break;
				case 5: c=gamma*cos(phi)*cos(theta) - beta*sin(phi); break;
				case 6: c=-(beta*cos(phi)*cos(theta)) - gamma*sin(phi); break;
				case 7: c=cos(phi)*sin(theta); break;
				case 8: c=alpha*sin(phi)*sin(theta); break;
				case 9: c=beta*cos(phi) + gamma*cos(theta)*sin(phi); break;
				case 10: c=gamma*cos(phi) - beta*cos(theta)*sin(phi); break;
				case 11: c=sin(phi)*sin(theta); break;
				case 12: c=alpha*cos(theta); break;
				case 13: c=-(gamma*sin(theta)); break;
				case 14: c=beta*sin(theta); break;
				case 15: c=cos(theta); break;
		}

		//printf("device_AADecay12 %d ~ %f\n", index, c);
		return c;
}


__device__ double  device_AAProd1212(double alpha_Jpsi, double delta_phi, double Jpsi_the, const int index){
		double v  = sqrt(1-alpha_Jpsi*alpha_Jpsi);
		double c=0;
		switch (index){
				case 0: c=(1+alpha_Jpsi*cos(Jpsi_the)*cos(Jpsi_the)); break;
				case 2: c=v*sin(Jpsi_the)*cos(Jpsi_the)*sin(delta_phi); break;
				case 5: c=sin(Jpsi_the)*sin(Jpsi_the); break;
				case 7: c=v*sin(Jpsi_the)*cos(Jpsi_the)* cos(delta_phi); break;
				case 8: c=-v*sin(Jpsi_the)*cos(Jpsi_the)*sin(delta_phi); break;
				case 10: c=alpha_Jpsi*sin(Jpsi_the)*sin(Jpsi_the); break;
				case 13: c=-v*sin(Jpsi_the)*cos(Jpsi_the)* cos(delta_phi); break;
				case 15: c=-alpha_Jpsi - cos(Jpsi_the)*cos(Jpsi_the); break;
		}
		//printf("device_AADecay1212 %d ~ %f\n", index, c);
		return c;
}


__device__ double Amp(double *Matrix, const int Index){
		double amp = 0;
		int i1 = 0;
		int i2 = 16;
		int i3 = 32;
		int i4 = 48;
		int i5 = 64;
		for(int mu=0; mu<4;mu++){// Xi loop
				for(int nu=0;nu<4;nu++){// Xibar loop
						for(int k=0;k<4;k++){
								for(int j=0;j<4;j++){
										amp += Matrix[Index*80 + i1 + mu*4 + nu]*  // G_numu
												Matrix[Index*80 + i2 + mu*4 + k]*  // A_numu for Xi decay
												Matrix[Index*80 + i3 + nu*4 + j]*  // A_numu for Xibar decay
												Matrix[Index*80 + i4 + k*4 + 0]*	// A_numu for lambda decay
												Matrix[Index*80 + i5 + j*4 + 0];  	// A_numu for lambdabar decay
								}
						}
				}
		}
		return amp;
}

__global__ void gpu_Amp(
				double *g_xithe,
				double *g_lthe,
				double *g_lphi,
				double *g_lbthe,
				double *g_lbphi,
				double *g_pthe,
				double *g_pphi,
				double *g_apthe,
				double *g_apphi,
				double *amp,
				const int g_NN,
				const AA_parameter g_para,
				const int g_flag, double *Matrix, const double norm){

		int index = blockIdx.x * blockDim.x + threadIdx.x;
		if( index <  g_NN){
				int iEvt = index / 80;

				int IMat = index % 80;
				int iMat = IMat / 16;
				int jMat = IMat % 16;
				if(g_flag  == 0){

						switch (iMat) {
								case 0: Matrix[index] = device_AAProd1212(g_para.alpha_jpsi, g_para.phi_jpsi, g_xithe[iEvt], jMat); break;
								case 1: Matrix[index] = device_AADecay12(g_para.alpha_xi, g_para.phi_xi, g_lthe[iEvt], g_lphi[iEvt], jMat); break;
								case 2: Matrix[index] = device_AADecay12(g_para.alpha_xibar, g_para.phi_xibar, g_lbthe[iEvt], g_lbphi[iEvt], jMat); break;
								case 3: Matrix[index] = device_AADecay12(g_para.alpha1_lambda, 0, g_pthe[iEvt], g_pphi[iEvt], jMat); break;
								case 4: Matrix[index] = device_AADecay12(g_para.alpha1_lambdabar, 0, g_apthe[iEvt], g_apphi[iEvt], jMat); break;
						}

				}
				else if(g_flag  == 1){

						switch (iMat) {
								case 0: Matrix[index] = device_AAProd1212(g_para.alpha_jpsi, g_para.phi_jpsi, g_xithe[iEvt], jMat); break;
								case 1: Matrix[index] = device_AADecay12(g_para.alpha_xi, g_para.phi_xi, g_lthe[iEvt], g_lphi[iEvt], jMat); break;
								case 2: Matrix[index] = device_AADecay12(g_para.alpha_xibar, g_para.phi_xibar, g_lbthe[iEvt], g_lbphi[iEvt], jMat); break;
								case 3: Matrix[index] = device_AADecay12(g_para.alpha2_lambda, 0, g_pthe[iEvt], g_pphi[iEvt], jMat); break;
								case 4: Matrix[index] = device_AADecay12(g_para.alpha2_lambdabar, 0, g_apthe[iEvt], g_apphi[iEvt], jMat); break;
						}
				}
				__syncthreads();

				if(IMat == 0){
						amp[iEvt] = log( Amp(Matrix, iEvt) / norm);
						//printf("%d ~ %f\n", iEvt, amp[iEvt]);
				}
		}
		__syncthreads();
}


// Bandwidth: (((2^27) + 1) unsigned ints * 4 bytes/unsigned int)/(38.716 * 10^-3 s)
//  13.867 GB/s = 96.297% -> excellent memory bandwidth
// Reasonable point to stop working on this implementation's optimization
// Algorithm is not compute-intensive, so acheiving >75% of theoretical bandwidth is goal
// Main strategies used:
// - Process as much data as possible (in terms of algorithm correctness) in shared memory
// - Use sequential addressing to get rid of bank conflicts
__global__
void block_sum_reduce(double* const d_block_sums, 
	double* const d_in,
	const unsigned int d_in_len)
{
	extern __shared__ unsigned int s_out[];

	unsigned int max_elems_per_block = blockDim.x * 2;
	unsigned int glbl_tid = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int tid = threadIdx.x;
	
	// Zero out shared memory
	// Especially important when padding shmem for
	//  non-power of 2 sized input
	s_out[threadIdx.x] = 0;
	s_out[threadIdx.x + blockDim.x] = 0;

	__syncthreads();

	// Copy d_in to shared memory per block
	if (glbl_tid < d_in_len)
	{
		s_out[threadIdx.x] = d_in[glbl_tid];
		if (glbl_tid + blockDim.x < d_in_len)
			s_out[threadIdx.x + blockDim.x] = d_in[glbl_tid + blockDim.x];
	}
	__syncthreads();

	// Actually do the reduction
	for (unsigned int s = blockDim.x; s > 0; s >>= 1) {
		if (tid < s) {
			s_out[tid] += s_out[tid + s];
		}
		__syncthreads();
	}

	// write result for this block to global mem
	if (tid == 0)
		d_block_sums[blockIdx.x] = s_out[0];
}







